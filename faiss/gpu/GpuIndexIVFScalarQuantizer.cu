#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <faiss/gpu/GpuIndexFlat.h>
#include <faiss/gpu/GpuIndexIVFScalarQuantizer.h>
#include <faiss/gpu/GpuResources.h>
#include <faiss/gpu/utils/DeviceUtils.h>
#include <faiss/gpu/impl/GpuScalarQuantizer.cuh>
#include <faiss/gpu/impl/IVFFlat.cuh>
#include <faiss/gpu/utils/CopyUtils.cuh>
#include <limits>

namespace faiss {
namespace gpu {

GpuIndexIVFScalarQuantizer::GpuIndexIVFScalarQuantizer(
        GpuResourcesProvider* provider,
        const faiss::IndexIVFScalarQuantizer* index,
        GpuIndexIVFScalarQuantizerConfig config)
        : GpuIndexIVF(
                  provider,
                  index->d,
                  index->metric_type,
                  index->metric_arg,
                  index->nlist,
                  config),
          sq(index->sq),
          by_residual(index->by_residual),
          ivfSQConfig_(config),
          reserveMemoryVecs_(0) {
    copyFrom(index);

    FAISS_THROW_IF_NOT_MSG(
            isSQSupported(sq.qtype), "Unsupported QuantizerType on GPU");
}

GpuIndexIVFScalarQuantizer::GpuIndexIVFScalarQuantizer(
        GpuResourcesProvider* provider,
        int dims,
        int nlist,
        faiss::ScalarQuantizer::QuantizerType qtype,
        faiss::MetricType metric,
        bool encodeResidual,
        GpuIndexIVFScalarQuantizerConfig config)
        : GpuIndexIVF(provider, dims, metric, 0, nlist, config),
          sq(dims, qtype),
          by_residual(encodeResidual),
          ivfSQConfig_(config),
          reserveMemoryVecs_(0) {
    // faiss::Index params
    this->is_trained = false;

    // We haven't trained ourselves, so don't construct the IVFFlat
    // index yet
    FAISS_THROW_IF_NOT_MSG(
            isSQSupported(sq.qtype), "Unsupported QuantizerType on GPU");
}

GpuIndexIVFScalarQuantizer::~GpuIndexIVFScalarQuantizer() {}

void GpuIndexIVFScalarQuantizer::reserveMemory(size_t numVecs) {
    reserveMemoryVecs_ = numVecs;
    if (index_) {
        DeviceScope scope(config_.device);
        index_->reserveMemory(numVecs);
    }
}

void GpuIndexIVFScalarQuantizer::copyFrom(
        const faiss::IndexIVFScalarQuantizer* index) {
    DeviceScope scope(config_.device);

    // Clear out our old data
    index_.reset();

    // Copy what we need from the CPU index
    GpuIndexIVF::copyFrom(index);
    sq = index->sq;
    by_residual = index->by_residual;

    // The other index might not be trained, in which case we don't need to copy
    // over the lists
    if (!index->is_trained) {
        return;
    }

    // Otherwise, we can populate ourselves from the other index
    this->is_trained = true;

    // Copy our lists as well
    index_.reset(new IVFFlat(
            resources_.get(),
            quantizer->getGpuData(),
            index->metric_type,
            index->metric_arg,
            by_residual,
            &sq,
            ivfSQConfig_.interleavedLayout,
            ivfSQConfig_.indicesOptions,
            config_.memorySpace));

    // Copy all of the IVF data
    index_->copyInvertedListsFrom(index->invlists);
}

void GpuIndexIVFScalarQuantizer::copyTo(
        faiss::IndexIVFScalarQuantizer* index) const {
    DeviceScope scope(config_.device);

    // We must have the indices in order to copy to ourselves
    FAISS_THROW_IF_NOT_MSG(
            ivfSQConfig_.indicesOptions != INDICES_IVF,
            "Cannot copy to CPU as GPU index doesn't retain "
            "indices (INDICES_IVF)");

    GpuIndexIVF::copyTo(index);
    index->sq = sq;
    index->code_size = sq.code_size;
    index->by_residual = by_residual;

    auto ivf = new ArrayInvertedLists(nlist, index->code_size);
    index->replace_invlists(ivf, true);

    if (index_) {
        // Copy IVF lists
        index_->copyInvertedListsTo(ivf);
    }
}

size_t GpuIndexIVFScalarQuantizer::reclaimMemory() {
    if (index_) {
        DeviceScope scope(config_.device);

        return index_->reclaimMemory();
    }

    return 0;
}

void GpuIndexIVFScalarQuantizer::reset() {
    if (index_) {
        DeviceScope scope(config_.device);

        index_->reset();
        this->ntotal = 0;
    } else {
        FAISS_ASSERT(this->ntotal == 0);
    }
}

int GpuIndexIVFScalarQuantizer::getListLength(int listId) const {
    FAISS_ASSERT(index_);
    DeviceScope scope(config_.device);

    return index_->getListLength(listId);
}

std::vector<uint8_t> GpuIndexIVFScalarQuantizer::getListVectorData(
        int listId,
        bool gpuFormat) const {
    FAISS_ASSERT(index_);
    DeviceScope scope(config_.device);

    return index_->getListVectorData(listId, gpuFormat);
}

std::vector<Index::idx_t> GpuIndexIVFScalarQuantizer::getListIndices(
        int listId) const {
    FAISS_ASSERT(index_);
    DeviceScope scope(config_.device);

    return index_->getListIndices(listId);
}

void GpuIndexIVFScalarQuantizer::trainResiduals_(
        Index::idx_t n,
        const float* x) {
    // The input is already guaranteed to be on the CPU
    sq.train_residual(n, x, quantizer, by_residual, verbose);
}

void GpuIndexIVFScalarQuantizer::train(Index::idx_t n, const float* x) {
    // For now, only support <= max int results
    FAISS_THROW_IF_NOT_FMT(
            n <= (Index::idx_t)std::numeric_limits<int>::max(),
            "GPU index only supports up to %d indices",
            std::numeric_limits<int>::max());

    DeviceScope scope(config_.device);

    if (this->is_trained) {
        FAISS_ASSERT(quantizer->is_trained);
        FAISS_ASSERT(quantizer->ntotal == nlist);
        FAISS_ASSERT(index_);
        return;
    }

    FAISS_ASSERT(!index_);

    // FIXME: GPUize more of this
    // First, make sure that the data is resident on the CPU, if it is not on
    // the CPU, as we depend upon parts of the CPU code
    auto hostData = toHost<float, 2>(
            (float*)x,
            resources_->getDefaultStream(config_.device),
            {(int)n, (int)this->d});

    trainQuantizer_(n, hostData.data());
    trainResiduals_(n, hostData.data());

    // The quantizer is now trained; construct the IVF index
    index_.reset(new IVFFlat(
            resources_.get(),
            quantizer->getGpuData(),
            this->metric_type,
            this->metric_arg,
            by_residual,
            &sq,
            ivfSQConfig_.interleavedLayout,
            ivfSQConfig_.indicesOptions,
            config_.memorySpace));

    if (reserveMemoryVecs_) {
        index_->reserveMemory(reserveMemoryVecs_);
    }

    this->is_trained = true;
}

void GpuIndexIVFScalarQuantizer::addImpl_(
        int n,
        const float* x,
        const Index::idx_t* xids) {
    // Device is already set in GpuIndex::add
    FAISS_ASSERT(index_);
    FAISS_ASSERT(n > 0);

    // Data is already resident on the GPU
    Tensor<float, 2, true> data(const_cast<float*>(x), {n, (int)this->d});
    Tensor<Index::idx_t, 1, true> labels(const_cast<Index::idx_t*>(xids), {n});

    // Not all vectors may be able to be added (some may contain NaNs etc)
    index_->addVectors(data, labels);

    // but keep the ntotal based on the total number of vectors that we
    // attempted to add
    ntotal += n;
}

void GpuIndexIVFScalarQuantizer::searchImpl_(
        int n,
        const float* x,
        int k,
        float* distances,
        Index::idx_t* labels) const {
    // Device is already set in GpuIndex::search
    FAISS_ASSERT(index_);
    FAISS_ASSERT(n > 0);
    FAISS_THROW_IF_NOT(nprobe > 0 && nprobe <= nlist);

    // Data is already resident on the GPU
    Tensor<float, 2, true> queries(const_cast<float*>(x), {n, (int)this->d});
    Tensor<float, 2, true> outDistances(distances, {n, k});
    Tensor<Index::idx_t, 2, true> outLabels(
            const_cast<Index::idx_t*>(labels), {n, k});

    index_->query(queries, nprobe, k, outDistances, outLabels);
}

} // namespace gpu
} // namespace faiss
