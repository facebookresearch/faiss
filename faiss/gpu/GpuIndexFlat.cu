#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <faiss/IndexFlat.h>
#include <faiss/gpu/GpuIndexFlat.h>
#include <faiss/gpu/GpuResources.h>
#include <faiss/gpu/impl/IndexUtils.h>
#include <faiss/gpu/utils/DeviceUtils.h>
#include <faiss/gpu/utils/StaticUtils.h>
#include <faiss/gpu/impl/FlatIndex.cuh>
#include <faiss/gpu/utils/ConversionOperators.cuh>
#include <faiss/gpu/utils/CopyUtils.cuh>
#include <faiss/gpu/utils/Float16.cuh>
#include <limits>

namespace faiss {
namespace gpu {

GpuIndexFlat::GpuIndexFlat(
        GpuResourcesProvider* provider,
        const faiss::IndexFlat* index,
        GpuIndexFlatConfig config)
        : GpuIndex(
                  provider->getResources(),
                  index->d,
                  index->metric_type,
                  index->metric_arg,
                  config),
          flatConfig_(config) {
    // Flat index doesn't need training
    this->is_trained = true;

    copyFrom(index);
}

GpuIndexFlat::GpuIndexFlat(
        std::shared_ptr<GpuResources> resources,
        const faiss::IndexFlat* index,
        GpuIndexFlatConfig config)
        : GpuIndex(
                  resources,
                  index->d,
                  index->metric_type,
                  index->metric_arg,
                  config),
          flatConfig_(config) {
    // Flat index doesn't need training
    this->is_trained = true;

    copyFrom(index);
}

GpuIndexFlat::GpuIndexFlat(
        GpuResourcesProvider* provider,
        int dims,
        faiss::MetricType metric,
        GpuIndexFlatConfig config)
        : GpuIndex(provider->getResources(), dims, metric, 0, config),
          flatConfig_(config) {
    DeviceScope scope(config_.device);

    // Flat index doesn't need training
    this->is_trained = true;

    // Construct index
    data_.reset(new FlatIndex(
            resources_.get(),
            dims,
            flatConfig_.useFloat16,
            config_.memorySpace));
}

GpuIndexFlat::GpuIndexFlat(
        std::shared_ptr<GpuResources> resources,
        int dims,
        faiss::MetricType metric,
        GpuIndexFlatConfig config)
        : GpuIndex(resources, dims, metric, 0, config), flatConfig_(config) {
    DeviceScope scope(config_.device);

    // Flat index doesn't need training
    this->is_trained = true;

    // Construct index
    data_.reset(new FlatIndex(
            resources_.get(),
            dims,
            flatConfig_.useFloat16,
            config_.memorySpace));
}

GpuIndexFlat::~GpuIndexFlat() {}

void GpuIndexFlat::copyFrom(const faiss::IndexFlat* index) {
    DeviceScope scope(config_.device);

    GpuIndex::copyFrom(index);

    data_.reset();
    data_.reset(new FlatIndex(
            resources_.get(),
            this->d,
            flatConfig_.useFloat16,
            config_.memorySpace));

    // The index could be empty
    if (index->ntotal > 0) {
        data_->add(
                index->get_xb(),
                index->ntotal,
                resources_->getDefaultStream(config_.device));
    }
}

void GpuIndexFlat::copyTo(faiss::IndexFlat* index) const {
    DeviceScope scope(config_.device);

    GpuIndex::copyTo(index);
    index->code_size = sizeof(float) * this->d;

    FAISS_ASSERT(data_);
    FAISS_ASSERT(data_->getSize() == this->ntotal);
    index->codes.resize(this->ntotal * index->code_size);

    if (this->ntotal > 0) {
        // FIXME: there is an extra GPU allocation here and copy if the flat
        // index is already float32
        reconstruct_n(0, this->ntotal, index->get_xb());
    }
}

size_t GpuIndexFlat::getNumVecs() const {
    return this->ntotal;
}

void GpuIndexFlat::reset() {
    DeviceScope scope(config_.device);

    // Free the underlying memory
    data_->reset();
    this->ntotal = 0;
}

void GpuIndexFlat::train(idx_t n, const float* x) {
    // nothing to do
}

void GpuIndexFlat::add(idx_t n, const float* x) {
    DeviceScope scope(config_.device);

    FAISS_THROW_IF_NOT_MSG(this->is_trained, "Index not trained");

    if (n == 0) {
        // nothing to add
        return;
    }

    // To avoid multiple re-allocations, ensure we have enough storage
    // available
    data_->reserve(n, resources_->getDefaultStream(config_.device));

    // If we're not operating in float16 mode, we don't need the input
    // data to be resident on our device; we can add directly.
    if (!flatConfig_.useFloat16) {
        addImpl_(n, x, nullptr);
    } else {
        // Otherwise, perform the paging
        GpuIndex::add(n, x);
    }
}

bool GpuIndexFlat::addImplRequiresIDs_() const {
    return false;
}

void GpuIndexFlat::addImpl_(idx_t n, const float* x, const idx_t* ids) {
    // current device already set
    // n already validated
    FAISS_ASSERT(data_);
    FAISS_ASSERT(n > 0);

    // We do not support add_with_ids
    FAISS_THROW_IF_NOT_MSG(!ids, "add_with_ids not supported");

    data_->add(x, n, resources_->getDefaultStream(config_.device));
    this->ntotal += n;
}

void GpuIndexFlat::searchImpl_(
        idx_t n,
        const float* x,
        int k,
        float* distances,
        idx_t* labels,
        const SearchParameters* params) const {
    // current device already set
    // n/k already validated
    auto stream = resources_->getDefaultStream(config_.device);

    // Input and output data are already resident on the GPU
    Tensor<float, 2, true> queries(const_cast<float*>(x), {n, this->d});
    Tensor<float, 2, true> outDistances(distances, {n, k});
    Tensor<idx_t, 2, true> outLabels(labels, {n, k});

    data_->query(
            queries, k, metric_type, metric_arg, outDistances, outLabels, true);
}

void GpuIndexFlat::reconstruct(idx_t key, float* out) const {
    DeviceScope scope(config_.device);

    FAISS_THROW_IF_NOT_FMT(
            key < this->ntotal,
            "index %zu out of bounds (ntotal %zu)",
            key,
            this->ntotal);
    auto stream = resources_->getDefaultStream(config_.device);

    // FIXME: `out` may already be on the device, in which case this is an
    // unneeded allocation
    DeviceTensor<float, 2, true> vec(
            resources_.get(),
            makeTempAlloc(AllocType::Other, stream),
            {1, this->d});

    FAISS_ASSERT(data_);
    data_->reconstruct(key, 1, vec);

    fromDevice(vec.data(), out, this->d, stream);
}

void GpuIndexFlat::reconstruct_n(idx_t i0, idx_t n, float* out) const {
    DeviceScope scope(config_.device);

    if (n == 0) {
        // nothing to do
        return;
    }

    FAISS_THROW_IF_NOT_FMT(
            i0 < this->ntotal,
            "start index (%zu) out of bounds (ntotal %zu)",
            i0,
            this->ntotal);
    FAISS_THROW_IF_NOT_FMT(
            i0 + n - 1 < this->ntotal,
            "max index requested (%zu) out of bounds (ntotal %zu)",
            i0 + n - 1,
            this->ntotal);
    auto stream = resources_->getDefaultStream(config_.device);

    auto outDevice = toDeviceTemporary<float, 2>(
            resources_.get(), config_.device, out, stream, {n, this->d});

    FAISS_ASSERT(data_);
    data_->reconstruct(i0, n, outDevice);

    fromDevice<float, 2>(outDevice, out, stream);
}

void GpuIndexFlat::reconstruct_batch(idx_t n, const idx_t* keys, float* out)
        const {
    DeviceScope scope(config_.device);
    auto stream = resources_->getDefaultStream(config_.device);

    if (n == 0) {
        // nothing to do
        return;
    }

    auto keysDevice = toDeviceTemporary<faiss::idx_t, 1>(
            resources_.get(),
            config_.device,
            const_cast<idx_t*>(keys),
            stream,
            {n});

    auto outDevice = toDeviceTemporary<float, 2>(
            resources_.get(), config_.device, out, stream, {n, this->d});

    FAISS_ASSERT(data_);
    data_->reconstruct(keysDevice, outDevice);

    // If the output is on the host, copy back if needed
    fromDevice<float, 2>(outDevice, out, stream);
}

void GpuIndexFlat::compute_residual(const float* x, float* residual, idx_t key)
        const {
    compute_residual_n(1, x, residual, &key);
}

void GpuIndexFlat::compute_residual_n(
        idx_t n,
        const float* xs,
        float* residuals,
        const idx_t* keys) const {
    DeviceScope scope(config_.device);
    auto stream = resources_->getDefaultStream(config_.device);

    if (n == 0) {
        // nothing to do
        return;
    }

    auto vecsDevice = toDeviceTemporary<float, 2>(
            resources_.get(),
            config_.device,
            const_cast<float*>(xs),
            stream,
            {n, this->d});
    auto idsDevice = toDeviceTemporary<idx_t, 1>(
            resources_.get(),
            config_.device,
            const_cast<idx_t*>(keys),
            stream,
            {n});
    auto residualDevice = toDeviceTemporary<float, 2>(
            resources_.get(), config_.device, residuals, stream, {n, this->d});

    FAISS_ASSERT(data_);
    data_->computeResidual(vecsDevice, idsDevice, residualDevice);

    // If the output is on the host, copy back if needed
    fromDevice<float, 2>(residualDevice, residuals, stream);
}

//
// GpuIndexFlatL2
//

GpuIndexFlatL2::GpuIndexFlatL2(
        GpuResourcesProvider* provider,
        faiss::IndexFlatL2* index,
        GpuIndexFlatConfig config)
        : GpuIndexFlat(provider, index, config) {}

GpuIndexFlatL2::GpuIndexFlatL2(
        std::shared_ptr<GpuResources> resources,
        faiss::IndexFlatL2* index,
        GpuIndexFlatConfig config)
        : GpuIndexFlat(resources, index, config) {}

GpuIndexFlatL2::GpuIndexFlatL2(
        GpuResourcesProvider* provider,
        int dims,
        GpuIndexFlatConfig config)
        : GpuIndexFlat(provider, dims, faiss::METRIC_L2, config) {}

GpuIndexFlatL2::GpuIndexFlatL2(
        std::shared_ptr<GpuResources> resources,
        int dims,
        GpuIndexFlatConfig config)
        : GpuIndexFlat(resources, dims, faiss::METRIC_L2, config) {}

void GpuIndexFlatL2::copyFrom(faiss::IndexFlat* index) {
    FAISS_THROW_IF_NOT_MSG(
            index->metric_type == metric_type,
            "Cannot copy a GpuIndexFlatL2 from an index of "
            "different metric_type");

    GpuIndexFlat::copyFrom(index);
}

void GpuIndexFlatL2::copyTo(faiss::IndexFlat* index) {
    FAISS_THROW_IF_NOT_MSG(
            index->metric_type == metric_type,
            "Cannot copy a GpuIndexFlatL2 to an index of "
            "different metric_type");

    GpuIndexFlat::copyTo(index);
}

//
// GpuIndexFlatIP
//

GpuIndexFlatIP::GpuIndexFlatIP(
        GpuResourcesProvider* provider,
        faiss::IndexFlatIP* index,
        GpuIndexFlatConfig config)
        : GpuIndexFlat(provider, index, config) {}

GpuIndexFlatIP::GpuIndexFlatIP(
        std::shared_ptr<GpuResources> resources,
        faiss::IndexFlatIP* index,
        GpuIndexFlatConfig config)
        : GpuIndexFlat(resources, index, config) {}

GpuIndexFlatIP::GpuIndexFlatIP(
        GpuResourcesProvider* provider,
        int dims,
        GpuIndexFlatConfig config)
        : GpuIndexFlat(provider, dims, faiss::METRIC_INNER_PRODUCT, config) {}

GpuIndexFlatIP::GpuIndexFlatIP(
        std::shared_ptr<GpuResources> resources,
        int dims,
        GpuIndexFlatConfig config)
        : GpuIndexFlat(resources, dims, faiss::METRIC_INNER_PRODUCT, config) {}

void GpuIndexFlatIP::copyFrom(faiss::IndexFlat* index) {
    FAISS_THROW_IF_NOT_MSG(
            index->metric_type == metric_type,
            "Cannot copy a GpuIndexFlatIP from an index of "
            "different metric_type");

    GpuIndexFlat::copyFrom(index);
}

void GpuIndexFlatIP::copyTo(faiss::IndexFlat* index) {
    // The passed in index must be IP
    FAISS_THROW_IF_NOT_MSG(
            index->metric_type == metric_type,
            "Cannot copy a GpuIndexFlatIP to an index of "
            "different metric_type");

    GpuIndexFlat::copyTo(index);
}

} // namespace gpu
} // namespace faiss
