#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <faiss/gpu/GpuIndexBinaryFlat.h>

#include <faiss/gpu/GpuResources.h>
#include <faiss/gpu/utils/DeviceUtils.h>
#include <faiss/gpu/impl/BinaryFlatIndex.cuh>
#include <faiss/gpu/utils/ConversionOperators.cuh>
#include <faiss/gpu/utils/CopyUtils.cuh>

namespace faiss {
namespace gpu {

/// Default CPU search size for which we use paged copies
constexpr size_t kMinPageSize = (size_t)256 * 1024 * 1024;

GpuIndexBinaryFlat::GpuIndexBinaryFlat(
        GpuResourcesProvider* provider,
        const faiss::IndexBinaryFlat* index,
        GpuIndexBinaryFlatConfig config)
        : IndexBinary(index->d),
          resources_(provider->getResources()),
          binaryFlatConfig_(config) {
    FAISS_THROW_IF_NOT_FMT(
            this->d % 8 == 0,
            "vector dimension (number of bits) "
            "must be divisible by 8 (passed %d)",
            this->d);

    // Flat index doesn't need training
    this->is_trained = true;

    copyFrom(index);
}

GpuIndexBinaryFlat::GpuIndexBinaryFlat(
        GpuResourcesProvider* provider,
        int dims,
        GpuIndexBinaryFlatConfig config)
        : IndexBinary(dims),
          resources_(provider->getResources()),
          binaryFlatConfig_(std::move(config)) {
    FAISS_THROW_IF_NOT_FMT(
            this->d % 8 == 0,
            "vector dimension (number of bits) "
            "must be divisible by 8 (passed %d)",
            this->d);

    // Flat index doesn't need training
    this->is_trained = true;

    // Construct index
    DeviceScope scope(binaryFlatConfig_.device);
    data_.reset(new BinaryFlatIndex(
            resources_.get(), this->d, binaryFlatConfig_.memorySpace));
}

GpuIndexBinaryFlat::~GpuIndexBinaryFlat() {}

int GpuIndexBinaryFlat::getDevice() const {
    return binaryFlatConfig_.device;
}

std::shared_ptr<GpuResources> GpuIndexBinaryFlat::getResources() {
    return resources_;
}

void GpuIndexBinaryFlat::copyFrom(const faiss::IndexBinaryFlat* index) {
    DeviceScope scope(binaryFlatConfig_.device);

    this->d = index->d;

    // GPU code has 32 bit indices
    FAISS_THROW_IF_NOT_FMT(
            index->ntotal <= (Index::idx_t)std::numeric_limits<int>::max(),
            "GPU index only supports up to %zu indices; "
            "attempting to copy CPU index with %zu parameters",
            (size_t)std::numeric_limits<int>::max(),
            (size_t)index->ntotal);
    this->ntotal = index->ntotal;

    // destroy old first before allocating new
    data_.reset();
    data_.reset(new BinaryFlatIndex(
            resources_.get(), this->d, binaryFlatConfig_.memorySpace));

    // The index could be empty
    if (index->ntotal > 0) {
        data_->add(
                index->xb.data(),
                index->ntotal,
                resources_->getDefaultStream(binaryFlatConfig_.device));
    }
}

void GpuIndexBinaryFlat::copyTo(faiss::IndexBinaryFlat* index) const {
    DeviceScope scope(binaryFlatConfig_.device);

    index->d = this->d;
    index->ntotal = this->ntotal;

    FAISS_ASSERT(data_);
    FAISS_ASSERT(data_->getSize() == this->ntotal);
    index->xb.resize(this->ntotal * (this->d / 8));

    if (this->ntotal > 0) {
        fromDevice(
                data_->getVectorsRef(),
                index->xb.data(),
                resources_->getDefaultStream(binaryFlatConfig_.device));
    }
}

void GpuIndexBinaryFlat::add(faiss::IndexBinary::idx_t n, const uint8_t* x) {
    DeviceScope scope(binaryFlatConfig_.device);

    // To avoid multiple re-allocations, ensure we have enough storage
    // available
    data_->reserve(n, resources_->getDefaultStream(binaryFlatConfig_.device));

    // Due to GPU indexing in int32, we can't store more than this
    // number of vectors on a GPU
    FAISS_THROW_IF_NOT_FMT(
            this->ntotal + n <= (Index::idx_t)std::numeric_limits<int>::max(),
            "GPU index only supports up to %zu indices",
            (size_t)std::numeric_limits<int>::max());

    data_->add(
            (const unsigned char*)x,
            n,
            resources_->getDefaultStream(binaryFlatConfig_.device));
    this->ntotal += n;
}

void GpuIndexBinaryFlat::reset() {
    DeviceScope scope(binaryFlatConfig_.device);

    // Free the underlying memory
    data_->reset();
    this->ntotal = 0;
}

void GpuIndexBinaryFlat::search(
        faiss::IndexBinary::idx_t n,
        const uint8_t* x,
        faiss::IndexBinary::idx_t k,
        int32_t* distances,
        faiss::IndexBinary::idx_t* labels) const {
    if (n == 0) {
        return;
    }

    FAISS_THROW_IF_NOT(k > 0);

    // For now, only support <= max int results
    FAISS_THROW_IF_NOT_FMT(
            n <= (Index::idx_t)std::numeric_limits<int>::max(),
            "GPU index only supports up to %zu indices",
            (size_t)std::numeric_limits<int>::max());
    FAISS_THROW_IF_NOT_FMT(
            k <= (Index::idx_t)getMaxKSelection(),
            "GPU only supports k <= %d (requested %d)",
            getMaxKSelection(),
            (int)k); // select limitation

    DeviceScope scope(binaryFlatConfig_.device);
    auto stream = resources_->getDefaultStream(binaryFlatConfig_.device);

    // The input vectors may be too large for the GPU, but we still
    // assume that the output distances and labels are not.
    // Go ahead and make space for output distances and labels on the
    // GPU.
    // If we reach a point where all inputs are too big, we can add
    // another level of tiling.
    auto outDistances = toDeviceTemporary<int32_t, 2>(
            resources_.get(),
            binaryFlatConfig_.device,
            distances,
            stream,
            {(int)n, (int)k});

    // FlatIndex only supports an interface returning int indices
    DeviceTensor<int, 2, true> outIntIndices(
            resources_.get(),
            makeTempAlloc(AllocType::Other, stream),
            {(int)n, (int)k});

    bool usePaged = false;

    if (getDeviceForAddress(x) == -1) {
        // It is possible that the user is querying for a vector set size
        // `x` that won't fit on the GPU.
        // In this case, we will have to handle paging of the data from CPU
        // -> GPU.
        // Currently, we don't handle the case where the output data won't
        // fit on the GPU (e.g., n * k is too large for the GPU memory).
        size_t dataSize = (size_t)n * (this->d / 8) * sizeof(uint8_t);

        if (dataSize >= kMinPageSize) {
            searchFromCpuPaged_(
                    n, x, k, outDistances.data(), outIntIndices.data());
            usePaged = true;
        }
    }

    if (!usePaged) {
        searchNonPaged_(n, x, k, outDistances.data(), outIntIndices.data());
    }

    // Convert and copy int indices out
    auto outIndices = toDeviceTemporary<Index::idx_t, 2>(
            resources_.get(),
            binaryFlatConfig_.device,
            labels,
            stream,
            {(int)n, (int)k});

    // Convert int to idx_t
    convertTensor<int, Index::idx_t, 2>(stream, outIntIndices, outIndices);

    // Copy back if necessary
    fromDevice<int32_t, 2>(outDistances, distances, stream);
    fromDevice<Index::idx_t, 2>(outIndices, labels, stream);
}

void GpuIndexBinaryFlat::searchNonPaged_(
        int n,
        const uint8_t* x,
        int k,
        int32_t* outDistancesData,
        int* outIndicesData) const {
    Tensor<int32_t, 2, true> outDistances(outDistancesData, {n, k});
    Tensor<int, 2, true> outIndices(outIndicesData, {n, k});

    auto stream = resources_->getDefaultStream(binaryFlatConfig_.device);

    // Make sure arguments are on the device we desire; use temporary
    // memory allocations to move it if necessary
    auto vecs = toDeviceTemporary<uint8_t, 2>(
            resources_.get(),
            binaryFlatConfig_.device,
            const_cast<uint8_t*>(x),
            stream,
            {n, (int)(this->d / 8)});

    data_->query(vecs, k, outDistances, outIndices);
}

void GpuIndexBinaryFlat::searchFromCpuPaged_(
        int n,
        const uint8_t* x,
        int k,
        int32_t* outDistancesData,
        int* outIndicesData) const {
    Tensor<int32_t, 2, true> outDistances(outDistancesData, {n, k});
    Tensor<int, 2, true> outIndices(outIndicesData, {n, k});

    auto vectorSize = sizeof(uint8_t) * (this->d / 8);

    // Just page without overlapping copy with compute (as GpuIndexFlat does)
    int batchSize = utils::nextHighestPowerOf2(
            (int)((size_t)kMinPageSize / vectorSize));

    for (int cur = 0; cur < n; cur += batchSize) {
        int num = std::min(batchSize, n - cur);

        auto outDistancesSlice = outDistances.narrowOutermost(cur, num);
        auto outIndicesSlice = outIndices.narrowOutermost(cur, num);

        searchNonPaged_(
                num,
                x + (size_t)cur * (this->d / 8),
                k,
                outDistancesSlice.data(),
                outIndicesSlice.data());
    }
}

void GpuIndexBinaryFlat::reconstruct(
        faiss::IndexBinary::idx_t key,
        uint8_t* out) const {
    DeviceScope scope(binaryFlatConfig_.device);

    FAISS_THROW_IF_NOT_MSG(key < this->ntotal, "index out of bounds");
    auto stream = resources_->getDefaultStream(binaryFlatConfig_.device);

    auto& vecs = data_->getVectorsRef();
    auto vec = vecs[key];

    fromDevice(vec.data(), out, vecs.getSize(1), stream);
}

} // namespace gpu
} // namespace faiss
