#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <faiss/gpu/GpuIndexBinaryFlat.h>

#include <faiss/gpu/GpuResources.h>
#include <faiss/gpu/impl/IndexUtils.h>
#include <faiss/gpu/utils/DeviceUtils.h>
#include <faiss/gpu/impl/BinaryFlatIndex.cuh>
#include <faiss/gpu/utils/ConversionOperators.cuh>
#include <faiss/gpu/utils/CopyUtils.cuh>

namespace faiss {
namespace gpu {

/// Default CPU search size for which we use paged copies
constexpr size_t kMinPageSize = (size_t)256 * 1024 * 1024;

GpuIndexBinaryFlat::GpuIndexBinaryFlat(
        GpuResourcesProvider* provider,
        const faiss::IndexBinaryFlat* index,
        GpuIndexBinaryFlatConfig config)
        : IndexBinary(index->d),
          resources_(provider->getResources()),
          binaryFlatConfig_(config) {
    FAISS_THROW_IF_NOT_FMT(
            this->d % 8 == 0,
            "vector dimension (number of bits) "
            "must be divisible by 8 (passed %d)",
            this->d);

    // Flat index doesn't need training
    this->is_trained = true;

    copyFrom(index);
}

GpuIndexBinaryFlat::GpuIndexBinaryFlat(
        GpuResourcesProvider* provider,
        int dims,
        GpuIndexBinaryFlatConfig config)
        : IndexBinary(dims),
          resources_(provider->getResources()),
          binaryFlatConfig_(std::move(config)) {
    DeviceScope scope(binaryFlatConfig_.device);
    FAISS_THROW_IF_NOT_FMT(
            this->d % 8 == 0,
            "vector dimension (number of bits) "
            "must be divisible by 8 (passed %d)",
            this->d);

    // Flat index doesn't need training
    this->is_trained = true;

    // Construct index
    data_.reset(new BinaryFlatIndex(
            resources_.get(), this->d, binaryFlatConfig_.memorySpace));
}

GpuIndexBinaryFlat::~GpuIndexBinaryFlat() {}

int GpuIndexBinaryFlat::getDevice() const {
    return binaryFlatConfig_.device;
}

std::shared_ptr<GpuResources> GpuIndexBinaryFlat::getResources() {
    return resources_;
}

void GpuIndexBinaryFlat::copyFrom(const faiss::IndexBinaryFlat* index) {
    DeviceScope scope(binaryFlatConfig_.device);

    this->d = index->d;

    this->ntotal = index->ntotal;

    // destroy old first before allocating new
    data_.reset();
    data_.reset(new BinaryFlatIndex(
            resources_.get(), this->d, binaryFlatConfig_.memorySpace));

    // The index could be empty
    if (index->ntotal > 0) {
        data_->add(
                index->xb.data(),
                index->ntotal,
                resources_->getDefaultStream(binaryFlatConfig_.device));
    }
}

void GpuIndexBinaryFlat::copyTo(faiss::IndexBinaryFlat* index) const {
    DeviceScope scope(binaryFlatConfig_.device);

    index->d = this->d;
    index->ntotal = this->ntotal;

    FAISS_ASSERT(data_);
    FAISS_ASSERT(data_->getSize() == this->ntotal);
    index->xb.resize(this->ntotal * (this->d / 8));

    if (this->ntotal > 0) {
        fromDevice(
                data_->getVectorsRef(),
                index->xb.data(),
                resources_->getDefaultStream(binaryFlatConfig_.device));
    }
}

void GpuIndexBinaryFlat::add(idx_t n, const uint8_t* x) {
    DeviceScope scope(binaryFlatConfig_.device);

    // To avoid multiple re-allocations, ensure we have enough storage
    // available
    data_->reserve(n, resources_->getDefaultStream(binaryFlatConfig_.device));

    data_->add(
            (const unsigned char*)x,
            n,
            resources_->getDefaultStream(binaryFlatConfig_.device));
    this->ntotal += n;
}

void GpuIndexBinaryFlat::reset() {
    DeviceScope scope(binaryFlatConfig_.device);

    // Free the underlying memory
    data_->reset();
    this->ntotal = 0;
}

void GpuIndexBinaryFlat::search(
        idx_t n,
        const uint8_t* x,
        idx_t k,
        int32_t* distances,
        faiss::idx_t* labels,
        const SearchParameters* params) const {
    DeviceScope scope(binaryFlatConfig_.device);
    auto stream = resources_->getDefaultStream(binaryFlatConfig_.device);

    if (n == 0) {
        return;
    }

    FAISS_THROW_IF_NOT_MSG(!params, "params not implemented");

    validateKSelect(k);

    // The input vectors may be too large for the GPU, but we still
    // assume that the output distances and labels are not.
    // Go ahead and make space for output distances and labels on the
    // GPU.
    // If we reach a point where all inputs are too big, we can add
    // another level of tiling.
    auto outDistances = toDeviceTemporary<int32_t, 2>(
            resources_.get(),
            binaryFlatConfig_.device,
            distances,
            stream,
            {n, k});

    auto outIndices = toDeviceTemporary<idx_t, 2>(
            resources_.get(), binaryFlatConfig_.device, labels, stream, {n, k});

    bool usePaged = false;

    if (getDeviceForAddress(x) == -1) {
        // It is possible that the user is querying for a vector set size
        // `x` that won't fit on the GPU.
        // In this case, we will have to handle paging of the data from CPU
        // -> GPU.
        // Currently, we don't handle the case where the output data won't
        // fit on the GPU (e.g., n * k is too large for the GPU memory).
        size_t dataSize = n * (this->d / 8) * sizeof(uint8_t);

        if (dataSize >= kMinPageSize) {
            searchFromCpuPaged_(
                    n, x, k, outDistances.data(), outIndices.data());
            usePaged = true;
        }
    }

    if (!usePaged) {
        searchNonPaged_(n, x, k, outDistances.data(), outIndices.data());
    }

    // Copy back if necessary
    fromDevice<int32_t, 2>(outDistances, distances, stream);
    fromDevice<idx_t, 2>(outIndices, labels, stream);
}

void GpuIndexBinaryFlat::searchNonPaged_(
        idx_t n,
        const uint8_t* x,
        int k,
        int32_t* outDistancesData,
        idx_t* outIndicesData) const {
    Tensor<int32_t, 2, true> outDistances(outDistancesData, {n, k});
    Tensor<idx_t, 2, true> outIndices(outIndicesData, {n, k});

    auto stream = resources_->getDefaultStream(binaryFlatConfig_.device);

    // Make sure arguments are on the device we desire; use temporary
    // memory allocations to move it if necessary
    auto vecs = toDeviceTemporary<uint8_t, 2>(
            resources_.get(),
            binaryFlatConfig_.device,
            const_cast<uint8_t*>(x),
            stream,
            {n, (this->d / 8)});

    data_->query(vecs, k, outDistances, outIndices);
}

void GpuIndexBinaryFlat::searchFromCpuPaged_(
        idx_t n,
        const uint8_t* x,
        int k,
        int32_t* outDistancesData,
        idx_t* outIndicesData) const {
    Tensor<int32_t, 2, true> outDistances(outDistancesData, {n, k});
    Tensor<idx_t, 2, true> outIndices(outIndicesData, {n, k});

    idx_t vectorSize = sizeof(uint8_t) * (this->d / 8);

    // Just page without overlapping copy with compute (as GpuIndexFlat does)
    auto batchSize =
            utils::nextHighestPowerOf2(((idx_t)kMinPageSize / vectorSize));

    for (idx_t cur = 0; cur < n; cur += batchSize) {
        auto num = std::min(batchSize, n - cur);

        auto outDistancesSlice = outDistances.narrowOutermost(cur, num);
        auto outIndicesSlice = outIndices.narrowOutermost(cur, num);

        searchNonPaged_(
                num,
                x + cur * (this->d / 8),
                k,
                outDistancesSlice.data(),
                outIndicesSlice.data());
    }
}

void GpuIndexBinaryFlat::reconstruct(faiss::idx_t key, uint8_t* out) const {
    DeviceScope scope(binaryFlatConfig_.device);

    FAISS_THROW_IF_NOT_MSG(key < this->ntotal, "index out of bounds");
    auto stream = resources_->getDefaultStream(binaryFlatConfig_.device);

    auto& vecs = data_->getVectorsRef();
    auto vec = vecs[key];

    fromDevice(vec.data(), out, vecs.getSize(1), stream);
}

} // namespace gpu
} // namespace faiss
