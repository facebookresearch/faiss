#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <faiss/gpu/impl/IndexUtils.h>
#include <faiss/impl/FaissAssert.h>
#include <faiss/gpu/utils/DeviceDefs.cuh>
#include <limits>

namespace faiss {
namespace gpu {

/// A collection of various utility functions for index implementation

/// Returns the maximum k-selection value supported based on the CUDA SDK that
/// we were compiled with. .cu files can use DeviceDefs.cuh, but this is for
/// non-CUDA files
int getMaxKSelection() {
    return GPU_MAX_SELECTION_K;
}

void validateKSelect(int k) {
    FAISS_THROW_IF_NOT_FMT(
            k > 0 && k <= getMaxKSelection(),
            "GPU index only supports min/max-K selection up to %d (requested %d)",
            getMaxKSelection(),
            k);
}

void validateNProbe(size_t nprobe) {
    FAISS_THROW_IF_NOT_FMT(
            nprobe > 0 && nprobe <= (size_t)getMaxKSelection(),
            "GPU IVF index only supports nprobe selection up to %d (requested %zu)",
            getMaxKSelection(),
            nprobe);
}

} // namespace gpu
} // namespace faiss
