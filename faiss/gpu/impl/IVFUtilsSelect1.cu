#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */


#include <faiss/gpu/impl/IVFUtils.cuh>
#include <faiss/gpu/utils/DeviceDefs.cuh>
#include <faiss/gpu/utils/DeviceUtils.h>
#include <faiss/gpu/utils/Limits.cuh>
#include <faiss/gpu/utils/Select.cuh>
#include <faiss/gpu/utils/StaticUtils.h>
#include <faiss/gpu/utils/Tensor.cuh>

//
// This kernel is split into a separate compilation unit to cut down
// on compile time
//

namespace faiss { namespace gpu {

template <int ThreadsPerBlock, int NumWarpQ, int NumThreadQ, bool Dir>
__global__ void
pass1SelectLists(Tensor<int, 2, true> prefixSumOffsets,
                 Tensor<float, 1, true> distance,
                 int nprobe,
                 int k,
                 Tensor<float, 3, true> heapDistances,
                 Tensor<int, 3, true> heapIndices) {
  constexpr int kNumWarps = ThreadsPerBlock / kWarpSize;

  __shared__ float smemK[kNumWarps * NumWarpQ];
  __shared__ int smemV[kNumWarps * NumWarpQ];

  constexpr auto kInit = Dir ? kFloatMin : kFloatMax;
  BlockSelect<float, int, Dir, Comparator<float>,
              NumWarpQ, NumThreadQ, ThreadsPerBlock>
    heap(kInit, -1, smemK, smemV, k);

  auto queryId = blockIdx.y;
  auto sliceId = blockIdx.x;
  auto numSlices = gridDim.x;

  int sliceSize = (nprobe / numSlices);
  int sliceStart = sliceSize * sliceId;
  int sliceEnd = sliceId == (numSlices - 1) ? nprobe :
    sliceStart + sliceSize;
  auto offsets = prefixSumOffsets[queryId].data();

  // We ensure that before the array (at offset -1), there is a 0 value
  int start = *(&offsets[sliceStart] - 1);
  int end = offsets[sliceEnd - 1];

  int num = end - start;
  int limit = utils::roundDown(num, kWarpSize);

  int i = threadIdx.x;
  auto distanceStart = distance[start].data();

  // BlockSelect add cannot be used in a warp divergent circumstance; we
  // handle the remainder warp below
  for (; i < limit; i += blockDim.x) {
    heap.add(distanceStart[i], start + i);
  }

  // Handle warp divergence separately
  if (i < num) {
    heap.addThreadQ(distanceStart[i], start + i);
  }

  // Merge all final results
  heap.reduce();

  // Write out the final k-selected values; they should be all
  // together
  for (int i = threadIdx.x; i < k; i += blockDim.x) {
    heapDistances[queryId][sliceId][i] = smemK[i];
    heapIndices[queryId][sliceId][i] = smemV[i];
  }
}

void
runPass1SelectLists(Tensor<int, 2, true>& prefixSumOffsets,
                    Tensor<float, 1, true>& distance,
                    int nprobe,
                    int k,
                    bool chooseLargest,
                    Tensor<float, 3, true>& heapDistances,
                    Tensor<int, 3, true>& heapIndices,
                    hipStream_t stream) {
  // This is caught at a higher level
  FAISS_ASSERT(k <= GPU_MAX_SELECTION_K);

  auto grid = dim3(heapDistances.getSize(1), prefixSumOffsets.getSize(0));

#define RUN_PASS(BLOCK, NUM_WARP_Q, NUM_THREAD_Q, DIR)                  \
  do {                                                                  \
    pass1SelectLists<BLOCK, NUM_WARP_Q, NUM_THREAD_Q, DIR>              \
      <<<grid, BLOCK, 0, stream>>>(prefixSumOffsets,                    \
                                   distance,                            \
                                   nprobe,                              \
                                   k,                                   \
                                   heapDistances,                       \
                                   heapIndices);                        \
    CUDA_TEST_ERROR();                                                  \
    return; /* success */                                               \
  } while (0)

#if GPU_MAX_SELECTION_K >= 2048

  // block size 128 for k <= 1024, 64 for k = 2048
#define RUN_PASS_DIR(DIR)                                 \
  do {                                                    \
    if (k == 1) {                                         \
      RUN_PASS(128, 1, 1, DIR);                           \
    } else if (k <= 32) {                                 \
      RUN_PASS(128, 32, 2, DIR);                          \
    } else if (k <= 64) {                                 \
      RUN_PASS(128, 64, 3, DIR);                          \
    } else if (k <= 128) {                                \
      RUN_PASS(128, 128, 3, DIR);                         \
    } else if (k <= 256) {                                \
      RUN_PASS(128, 256, 4, DIR);                         \
    } else if (k <= 512) {                                \
      RUN_PASS(128, 512, 8, DIR);                         \
    } else if (k <= 1024) {                               \
      RUN_PASS(128, 1024, 8, DIR);                        \
    } else if (k <= 2048) {                               \
      RUN_PASS(64, 2048, 8, DIR);                         \
    }                                                     \
  } while (0)

#else

#define RUN_PASS_DIR(DIR)                                 \
  do {                                                    \
    if (k == 1) {                                         \
      RUN_PASS(128, 1, 1, DIR);                           \
    } else if (k <= 32) {                                 \
      RUN_PASS(128, 32, 2, DIR);                          \
    } else if (k <= 64) {                                 \
      RUN_PASS(128, 64, 3, DIR);                          \
    } else if (k <= 128) {                                \
      RUN_PASS(128, 128, 3, DIR);                         \
    } else if (k <= 256) {                                \
      RUN_PASS(128, 256, 4, DIR);                         \
    } else if (k <= 512) {                                \
      RUN_PASS(128, 512, 8, DIR);                         \
    } else if (k <= 1024) {                               \
      RUN_PASS(128, 1024, 8, DIR);                        \
    }                                                     \
  } while (0)

#endif // GPU_MAX_SELECTION_K

  if (chooseLargest) {
    RUN_PASS_DIR(true);
  } else {
    RUN_PASS_DIR(false);
  }

#undef RUN_PASS_DIR
#undef RUN_PASS
}

} } // namespace
