#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <faiss/gpu/GpuIndex.h>
#include <faiss/gpu/GpuResources.h>
#include <faiss/gpu/impl/InterleavedCodes.h>
#include <faiss/gpu/impl/RemapIndices.h>
#include <faiss/gpu/utils/DeviceUtils.h>
#include <thrust/host_vector.h>
#include <faiss/gpu/impl/FlatIndex.cuh>
#include <faiss/gpu/impl/IVFAppend.cuh>
#include <faiss/gpu/impl/IVFFlat.cuh>
#include <faiss/gpu/impl/IVFFlatScan.cuh>
#include <faiss/gpu/impl/IVFInterleaved.cuh>
#include <faiss/gpu/utils/ConversionOperators.cuh>
#include <faiss/gpu/utils/CopyUtils.cuh>
#include <faiss/gpu/utils/DeviceDefs.cuh>
#include <faiss/gpu/utils/Float16.cuh>
#include <faiss/gpu/utils/HostTensor.cuh>
#include <faiss/gpu/utils/Transpose.cuh>
#include <limits>
#include <unordered_map>

namespace faiss {
namespace gpu {

IVFFlat::IVFFlat(
        GpuResources* res,
        int dim,
        idx_t nlist,
        faiss::MetricType metric,
        float metricArg,
        bool useResidual,
        faiss::ScalarQuantizer* scalarQ,
        bool interleavedLayout,
        IndicesOptions indicesOptions,
        MemorySpace space)
        : IVFBase(res,
                  dim,
                  nlist,
                  metric,
                  metricArg,
                  useResidual,
                  interleavedLayout,
                  indicesOptions,
                  space),
          scalarQ_(scalarQ ? new GpuScalarQuantizer(res, *scalarQ) : nullptr) {}

IVFFlat::~IVFFlat() {}

size_t IVFFlat::getGpuVectorsEncodingSize_(idx_t numVecs) const {
    if (interleavedLayout_) {
        // bits per scalar code
        idx_t bits = scalarQ_ ? scalarQ_->bits : 32 /* float */;

        // bytes to encode a block of 32 vectors (single dimension)
        idx_t bytesPerDimBlock = bits * 32 / 8;

        // bytes to fully encode 32 vectors
        idx_t bytesPerBlock = bytesPerDimBlock * dim_;

        // number of blocks of 32 vectors we have
        idx_t numBlocks = utils::divUp(numVecs, 32);

        // total size to encode numVecs
        return bytesPerBlock * numBlocks;
    } else {
        size_t sizePerVector =
                (scalarQ_ ? scalarQ_->code_size : sizeof(float) * dim_);

        return (size_t)numVecs * sizePerVector;
    }
}

size_t IVFFlat::getCpuVectorsEncodingSize_(idx_t numVecs) const {
    size_t sizePerVector =
            (scalarQ_ ? scalarQ_->code_size : sizeof(float) * dim_);

    return (size_t)numVecs * sizePerVector;
}

std::vector<uint8_t> IVFFlat::translateCodesToGpu_(
        std::vector<uint8_t> codes,
        idx_t numVecs) const {
    if (!interleavedLayout_) {
        // same format
        return codes;
    }

    int bitsPerCode = scalarQ_ ? scalarQ_->bits : 32;

    auto up =
            unpackNonInterleaved(std::move(codes), numVecs, dim_, bitsPerCode);
    return packInterleaved(std::move(up), numVecs, dim_, bitsPerCode);
}

std::vector<uint8_t> IVFFlat::translateCodesFromGpu_(
        std::vector<uint8_t> codes,
        idx_t numVecs) const {
    if (!interleavedLayout_) {
        // same format
        return codes;
    }

    int bitsPerCode = scalarQ_ ? scalarQ_->bits : 32;

    auto up = unpackInterleaved(std::move(codes), numVecs, dim_, bitsPerCode);
    return packNonInterleaved(std::move(up), numVecs, dim_, bitsPerCode);
}

void IVFFlat::appendVectors_(
        Tensor<float, 2, true>& vecs,
        Tensor<float, 2, true>& ivfCentroidResiduals,
        Tensor<idx_t, 1, true>& indices,
        Tensor<idx_t, 1, true>& uniqueLists,
        Tensor<idx_t, 1, true>& vectorsByUniqueList,
        Tensor<idx_t, 1, true>& uniqueListVectorStart,
        Tensor<idx_t, 1, true>& uniqueListStartOffset,
        Tensor<idx_t, 1, true>& listIds,
        Tensor<idx_t, 1, true>& listOffset,
        hipStream_t stream) {
    //
    // Append the new encodings
    //

    // Append indices to the IVF lists
    runIVFIndicesAppend(
            listIds,
            listOffset,
            indices,
            indicesOptions_,
            deviceListIndexPointers_,
            stream);

    // Append the encoded vectors to the IVF lists
    if (interleavedLayout_) {
        runIVFFlatInterleavedAppend(
                listIds,
                listOffset,
                uniqueLists,
                vectorsByUniqueList,
                uniqueListVectorStart,
                uniqueListStartOffset,
                useResidual_ ? ivfCentroidResiduals : vecs,
                scalarQ_.get(),
                deviceListDataPointers_,
                resources_,
                stream);
    } else {
        runIVFFlatAppend(
                listIds,
                listOffset,
                useResidual_ ? ivfCentroidResiduals : vecs,
                scalarQ_.get(),
                deviceListDataPointers_,
                stream);
    }
}

void IVFFlat::search(
        Index* coarseQuantizer,
        Tensor<float, 2, true>& queries,
        int nprobe,
        int k,
        Tensor<float, 2, true>& outDistances,
        Tensor<idx_t, 2, true>& outIndices) {
    auto stream = resources_->getDefaultStreamCurrentDevice();

    // These are caught at a higher level
    FAISS_ASSERT(nprobe <= GPU_MAX_SELECTION_K);
    FAISS_ASSERT(k <= GPU_MAX_SELECTION_K);
    nprobe = int(std::min(idx_t(nprobe), getNumLists()));

    FAISS_ASSERT(queries.getSize(1) == dim_);

    FAISS_ASSERT(outDistances.getSize(0) == queries.getSize(0));
    FAISS_ASSERT(outIndices.getSize(0) == queries.getSize(0));

    // Reserve space for the quantized information
    DeviceTensor<float, 2, true> coarseDistances(
            resources_,
            makeTempAlloc(AllocType::Other, stream),
            {queries.getSize(0), nprobe});
    DeviceTensor<idx_t, 2, true> coarseIndices(
            resources_,
            makeTempAlloc(AllocType::Other, stream),
            {queries.getSize(0), nprobe});
    // in case we also want/need residuals, we need the original centroids as
    // well
    // FIXME: why centroids instead of calculating residuals in one go?
    DeviceTensor<float, 3, true> residualBase(
            resources_,
            makeTempAlloc(AllocType::Other, stream),
            {queries.getSize(0), nprobe, dim_});

    searchCoarseQuantizer_(
            coarseQuantizer,
            nprobe,
            queries,
            coarseDistances,
            coarseIndices,
            nullptr,
            // we need the IVF centroids to which vectors were assigned if
            // vectors are encoded using the residual
            useResidual_ ? &residualBase : nullptr);

    searchImpl_(
            queries,
            coarseDistances,
            coarseIndices,
            residualBase,
            k,
            outDistances,
            outIndices,
            false);
}

void IVFFlat::searchPreassigned(
        Index* coarseQuantizer,
        Tensor<float, 2, true>& vecs,
        Tensor<float, 2, true>& ivfDistances,
        Tensor<idx_t, 2, true>& ivfAssignments,
        int k,
        Tensor<float, 2, true>& outDistances,
        Tensor<idx_t, 2, true>& outIndices,
        bool storePairs) {
    FAISS_ASSERT(ivfDistances.getSize(0) == vecs.getSize(0));
    FAISS_ASSERT(ivfAssignments.getSize(0) == vecs.getSize(0));
    FAISS_ASSERT(outDistances.getSize(0) == vecs.getSize(0));
    FAISS_ASSERT(outIndices.getSize(0) == vecs.getSize(0));
    FAISS_ASSERT(vecs.getSize(1) == dim_);

    auto stream = resources_->getDefaultStreamCurrentDevice();
    auto nprobe = ivfAssignments.getSize(1);
    FAISS_ASSERT(nprobe <= numLists_);

    // Based on the IVF assignments, we need the IVF centroids to which vectors
    // were assigned
    // FIXME: IVFPQ doesn't need this information as it has direct reference to
    // all IVF centroids and within the various kernels can look it up by index
    // as needed. Can we convert IVFFlat to do the same thing?
    DeviceTensor<float, 3, true> ivfCentroids(
            resources_,
            makeTempAlloc(AllocType::Other, stream),
            {vecs.getSize(0), nprobe, dim_});

    auto gpuQuantizer = tryCastGpuIndex(coarseQuantizer);
    if (gpuQuantizer) {
        // We can pass device pointers directly
        gpuQuantizer->reconstruct_batch(
                vecs.getSize(0) * nprobe,
                ivfAssignments.data(),
                ivfCentroids.data());
    } else {
        // CPU coarse quantizer
        auto cpuIVFCentroids =
                std::vector<float>(vecs.getSize(0) * nprobe * dim_);

        // We need to copy `ivfAssignments` to the CPU, in order to pass to a
        // CPU index
        auto cpuIVFAssignments = ivfAssignments.copyToVector(stream);

        coarseQuantizer->reconstruct_batch(
                vecs.getSize(0) * nprobe,
                cpuIVFAssignments.data(),
                cpuIVFCentroids.data());

        ivfCentroids.copyFrom(cpuIVFCentroids, stream);
    }

    searchImpl_(
            vecs,
            ivfDistances,
            ivfAssignments,
            ivfCentroids,
            k,
            outDistances,
            outIndices,
            storePairs);
}

void IVFFlat::searchImpl_(
        Tensor<float, 2, true>& queries,
        Tensor<float, 2, true>& coarseDistances,
        Tensor<idx_t, 2, true>& coarseIndices,
        Tensor<float, 3, true>& ivfCentroids,
        int k,
        Tensor<float, 2, true>& outDistances,
        Tensor<idx_t, 2, true>& outIndices,
        bool storePairs) {
    FAISS_ASSERT(storePairs == false);

    auto stream = resources_->getDefaultStreamCurrentDevice();

    if (interleavedLayout_) {
        runIVFInterleavedScan(
                queries,
                coarseIndices,
                deviceListDataPointers_,
                deviceListIndexPointers_,
                indicesOptions_,
                deviceListLengths_,
                k,
                metric_,
                useResidual_,
                ivfCentroids,
                scalarQ_.get(),
                outDistances,
                outIndices,
                resources_);
    } else {
        runIVFFlatScan(
                queries,
                coarseIndices,
                deviceListDataPointers_,
                deviceListIndexPointers_,
                indicesOptions_,
                deviceListLengths_,
                maxListLength_,
                k,
                metric_,
                useResidual_,
                ivfCentroids,
                scalarQ_.get(),
                outDistances,
                outIndices,
                resources_);
    }

    // If the GPU isn't storing indices (they are on the CPU side), we
    // need to perform the re-mapping here
    // FIXME: we might ultimately be calling this function with inputs
    // from the CPU, these are unnecessary copies
    if (indicesOptions_ == INDICES_CPU) {
        HostTensor<idx_t, 2, true> hostOutIndices(outIndices, stream);

        ivfOffsetToUserIndex(
                hostOutIndices.data(),
                numLists_,
                hostOutIndices.getSize(0),
                hostOutIndices.getSize(1),
                listOffsetToUserIndex_);

        // Copy back to GPU, since the input to this function is on the
        // GPU
        outIndices.copyFrom(hostOutIndices, stream);
    }
}

} // namespace gpu
} // namespace faiss
