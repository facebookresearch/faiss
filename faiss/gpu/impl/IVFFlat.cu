#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <faiss/gpu/GpuResources.h>
#include <faiss/gpu/impl/InterleavedCodes.h>
#include <faiss/gpu/impl/RemapIndices.h>
#include <faiss/gpu/utils/DeviceUtils.h>
#include <thrust/host_vector.h>
#include <faiss/gpu/impl/FlatIndex.cuh>
#include <faiss/gpu/impl/IVFAppend.cuh>
#include <faiss/gpu/impl/IVFFlat.cuh>
#include <faiss/gpu/impl/IVFFlatScan.cuh>
#include <faiss/gpu/impl/IVFInterleaved.cuh>
#include <faiss/gpu/utils/ConversionOperators.cuh>
#include <faiss/gpu/utils/CopyUtils.cuh>
#include <faiss/gpu/utils/DeviceDefs.cuh>
#include <faiss/gpu/utils/Float16.cuh>
#include <faiss/gpu/utils/HostTensor.cuh>
#include <faiss/gpu/utils/Transpose.cuh>
#include <limits>
#include <unordered_map>

namespace faiss {
namespace gpu {

IVFFlat::IVFFlat(
        GpuResources* res,
        FlatIndex* quantizer,
        faiss::MetricType metric,
        float metricArg,
        bool useResidual,
        faiss::ScalarQuantizer* scalarQ,
        bool interleavedLayout,
        IndicesOptions indicesOptions,
        MemorySpace space)
        : IVFBase(res,
                  metric,
                  metricArg,
                  quantizer,
                  interleavedLayout,
                  indicesOptions,
                  space),
          useResidual_(useResidual),
          scalarQ_(scalarQ ? new GpuScalarQuantizer(res, *scalarQ) : nullptr) {}

IVFFlat::~IVFFlat() {}

size_t IVFFlat::getGpuVectorsEncodingSize_(int numVecs) const {
    if (interleavedLayout_) {
        // bits per scalar code
        int bits = scalarQ_ ? scalarQ_->bits : 32 /* float */;

        // bytes to encode a block of 32 vectors (single dimension)
        int bytesPerDimBlock = bits * 32 / 8;

        // bytes to fully encode 32 vectors
        int bytesPerBlock = bytesPerDimBlock * dim_;

        // number of blocks of 32 vectors we have
        int numBlocks = utils::divUp(numVecs, 32);

        // total size to encode numVecs
        return bytesPerBlock * numBlocks;
    } else {
        size_t sizePerVector =
                (scalarQ_ ? scalarQ_->code_size : sizeof(float) * dim_);

        return (size_t)numVecs * sizePerVector;
    }
}

size_t IVFFlat::getCpuVectorsEncodingSize_(int numVecs) const {
    size_t sizePerVector =
            (scalarQ_ ? scalarQ_->code_size : sizeof(float) * dim_);

    return (size_t)numVecs * sizePerVector;
}

std::vector<uint8_t> IVFFlat::translateCodesToGpu_(
        std::vector<uint8_t> codes,
        size_t numVecs) const {
    if (!interleavedLayout_) {
        // same format
        return codes;
    }

    int bitsPerCode = scalarQ_ ? scalarQ_->bits : 32;

    auto up =
            unpackNonInterleaved(std::move(codes), numVecs, dim_, bitsPerCode);
    return packInterleaved(std::move(up), numVecs, dim_, bitsPerCode);
}

std::vector<uint8_t> IVFFlat::translateCodesFromGpu_(
        std::vector<uint8_t> codes,
        size_t numVecs) const {
    if (!interleavedLayout_) {
        // same format
        return codes;
    }

    int bitsPerCode = scalarQ_ ? scalarQ_->bits : 32;

    auto up = unpackInterleaved(std::move(codes), numVecs, dim_, bitsPerCode);
    return packNonInterleaved(std::move(up), numVecs, dim_, bitsPerCode);
}

void IVFFlat::appendVectors_(
        Tensor<float, 2, true>& vecs,
        Tensor<Index::idx_t, 1, true>& indices,
        Tensor<int, 1, true>& uniqueLists,
        Tensor<int, 1, true>& vectorsByUniqueList,
        Tensor<int, 1, true>& uniqueListVectorStart,
        Tensor<int, 1, true>& uniqueListStartOffset,
        Tensor<int, 1, true>& listIds,
        Tensor<int, 1, true>& listOffset,
        hipStream_t stream) {
    //
    // Append the new encodings
    //

    // Calculate residuals for these vectors, if needed
    DeviceTensor<float, 2, true> residuals(
            resources_,
            makeTempAlloc(AllocType::Other, stream),
            {vecs.getSize(0), dim_});

    if (useResidual_) {
        quantizer_->computeResidual(vecs, listIds, residuals);
    }

    // Append indices to the IVF lists
    runIVFIndicesAppend(
            listIds,
            listOffset,
            indices,
            indicesOptions_,
            deviceListIndexPointers_,
            stream);

    // Append the encoded vectors to the IVF lists
    if (interleavedLayout_) {
        runIVFFlatInterleavedAppend(
                listIds,
                listOffset,
                uniqueLists,
                vectorsByUniqueList,
                uniqueListVectorStart,
                uniqueListStartOffset,
                useResidual_ ? residuals : vecs,
                scalarQ_.get(),
                deviceListDataPointers_,
                resources_,
                stream);
    } else {
        runIVFFlatAppend(
                listIds,
                listOffset,
                useResidual_ ? residuals : vecs,
                scalarQ_.get(),
                deviceListDataPointers_,
                stream);
    }
}

void IVFFlat::query(
        Tensor<float, 2, true>& queries,
        int nprobe,
        int k,
        Tensor<float, 2, true>& outDistances,
        Tensor<Index::idx_t, 2, true>& outIndices) {
    auto stream = resources_->getDefaultStreamCurrentDevice();

    // These are caught at a higher level
    FAISS_ASSERT(nprobe <= GPU_MAX_SELECTION_K);
    FAISS_ASSERT(k <= GPU_MAX_SELECTION_K);
    nprobe = std::min(nprobe, quantizer_->getSize());

    FAISS_ASSERT(queries.getSize(1) == dim_);

    FAISS_ASSERT(outDistances.getSize(0) == queries.getSize(0));
    FAISS_ASSERT(outIndices.getSize(0) == queries.getSize(0));

    // Reserve space for the quantized information
    DeviceTensor<float, 2, true> coarseDistances(
            resources_,
            makeTempAlloc(AllocType::Other, stream),
            {queries.getSize(0), nprobe});
    DeviceTensor<int, 2, true> coarseIndices(
            resources_,
            makeTempAlloc(AllocType::Other, stream),
            {queries.getSize(0), nprobe});

    // Find the `nprobe` closest lists; we can use int indices both
    // internally and externally
    quantizer_->query(
            queries,
            nprobe,
            metric_,
            metricArg_,
            coarseDistances,
            coarseIndices,
            false);

    DeviceTensor<float, 3, true> residualBase(
            resources_,
            makeTempAlloc(AllocType::Other, stream),
            {queries.getSize(0), nprobe, dim_});

    if (useResidual_) {
        // Reconstruct vectors from the quantizer
        quantizer_->reconstruct(coarseIndices, residualBase);
    }

    if (interleavedLayout_) {
        runIVFInterleavedScan(
                queries,
                coarseIndices,
                deviceListDataPointers_,
                deviceListIndexPointers_,
                indicesOptions_,
                deviceListLengths_,
                k,
                metric_,
                useResidual_,
                residualBase,
                scalarQ_.get(),
                outDistances,
                outIndices,
                resources_);
    } else {
        runIVFFlatScan(
                queries,
                coarseIndices,
                deviceListDataPointers_,
                deviceListIndexPointers_,
                indicesOptions_,
                deviceListLengths_,
                maxListLength_,
                k,
                metric_,
                useResidual_,
                residualBase,
                scalarQ_.get(),
                outDistances,
                outIndices,
                resources_);
    }

    // If the GPU isn't storing indices (they are on the CPU side), we
    // need to perform the re-mapping here
    // FIXME: we might ultimately be calling this function with inputs
    // from the CPU, these are unnecessary copies
    if (indicesOptions_ == INDICES_CPU) {
        HostTensor<Index::idx_t, 2, true> hostOutIndices(outIndices, stream);

        ivfOffsetToUserIndex(
                hostOutIndices.data(),
                numLists_,
                hostOutIndices.getSize(0),
                hostOutIndices.getSize(1),
                listOffsetToUserIndex_);

        // Copy back to GPU, since the input to this function is on the
        // GPU
        outIndices.copyFrom(hostOutIndices, stream);
    }
}

} // namespace gpu
} // namespace faiss
