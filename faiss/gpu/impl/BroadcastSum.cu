#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */


#include <algorithm>
#include <faiss/impl/FaissAssert.h>

#include <faiss/gpu/utils/DeviceUtils.h>
#include <faiss/gpu/utils/MathOperators.cuh>
#include <faiss/gpu/utils/Tensor.cuh>
#include <faiss/gpu/utils/StaticUtils.h>

namespace faiss { namespace gpu {

template <typename T, int kRowsPerBlock, int kRowUnroll, int kColLoad>
__global__ void sumAlongColumns(Tensor<T, 1, true> input,
                                Tensor<T, 2, true> output) {
  static_assert(kRowsPerBlock % kRowUnroll == 0, "must fit rows");

  // blockIdx.x: which chunk of rows we are responsible for updating
  // blockIdx.y: which chunk of columns we are responsible for
  // updating
  int rowStart = blockIdx.x * kRowsPerBlock;
  int rowEnd = rowStart + kRowsPerBlock;
  int colStart = blockIdx.y * blockDim.x * kColLoad;

  // FIXME: if we have exact multiples, don't need this
  bool endRow = (blockIdx.x == gridDim.x - 1);
  bool endCol = (blockIdx.y == gridDim.y - 1);

  if (endRow) {
    if (output.getSize(0) % kRowsPerBlock == 0) {
      endRow = false;
    }
  }

  if (endCol) {
    for (int col = colStart + threadIdx.x;
         col < input.getSize(0); col += blockDim.x) {
      T val = input[col];

      if (endRow) {
        for (int row = rowStart; row < output.getSize(0); ++row) {
          T out = output[row][col];
          out = Math<T>::add(out, val);
          output[row][col] = out;
        }
      } else {
        T rows[kRowUnroll];

        for (int row = rowStart; row < rowEnd; row += kRowUnroll) {
#pragma unroll
          for (int i = 0; i < kRowUnroll; ++i) {
            rows[i] = output[row + i][col];
          }

#pragma unroll
          for (int i = 0; i < kRowUnroll; ++i) {
            rows[i] = Math<T>::add(rows[i], val);
          }

#pragma unroll
          for (int i = 0; i < kRowUnroll; ++i) {
            output[row + i][col] = rows[i];
          }
        }
      }
    }
  } else {
    int col = colStart + threadIdx.x;

    T val[kColLoad];

#pragma unroll
    for (int i = 0; i < kColLoad; ++i) {
      val[i] = input[col + i * blockDim.x];
    }

    if (endRow) {
      for (int row = rowStart; row < output.getSize(0); ++row) {
#pragma unroll
        for (int i = 0; i < kColLoad; ++i) {
          T out = output[row][col + i * blockDim.x];
          out = Math<T>::add(out, val[i]);
          output[row][col + i * blockDim.x] = out;
        }
      }
    } else {
      T rows[kRowUnroll * kColLoad];

      for (int row = rowStart; row < rowEnd; row += kRowUnroll) {
#pragma unroll
        for (int i = 0; i < kRowUnroll; ++i) {
#pragma unroll
          for (int j = 0; j < kColLoad; ++j) {
            rows[i * kColLoad + j] =
              output[row + i][col + j * blockDim.x];
          }
        }

#pragma unroll
        for (int i = 0; i < kRowUnroll; ++i) {
#pragma unroll
          for (int j = 0; j < kColLoad; ++j) {
            rows[i * kColLoad + j] =
              Math<T>::add(rows[i * kColLoad + j], val[j]);
          }
        }

#pragma unroll
        for (int i = 0; i < kRowUnroll; ++i) {
#pragma unroll
          for (int j = 0; j < kColLoad; ++j) {
            output[row + i][col + j * blockDim.x] =
              rows[i * kColLoad + j];
          }
        }
      }
    }
  }
}

template <typename T, int kRowsPerBlock, int kRowUnroll, int kColLoad>
__global__ void assignAlongColumns(Tensor<T, 1, true> input,
                                   Tensor<T, 2, true> output) {
  static_assert(kRowsPerBlock % kRowUnroll == 0, "must fit rows");

  // blockIdx.x: which chunk of rows we are responsible for updating
  // blockIdx.y: which chunk of columns we are responsible for
  // updating
  int rowStart = blockIdx.x * kRowsPerBlock;
  int rowEnd = rowStart + kRowsPerBlock;
  int colStart = blockIdx.y * blockDim.x * kColLoad;

  // FIXME: if we have exact multiples, don't need this
  bool endRow = (blockIdx.x == gridDim.x - 1);
  bool endCol = (blockIdx.y == gridDim.y - 1);

  if (endRow) {
    if (output.getSize(0) % kRowsPerBlock == 0) {
      endRow = false;
    }
  }

  if (endCol) {
    for (int col = colStart + threadIdx.x;
         col < input.getSize(0); col += blockDim.x) {
      T val = input[col];

      if (endRow) {
        for (int row = rowStart; row < output.getSize(0); ++row) {
          output[row][col] = val;
        }
      } else {
        for (int row = rowStart; row < rowEnd; row += kRowUnroll) {
#pragma unroll
          for (int i = 0; i < kRowUnroll; ++i) {
            output[row + i][col] = val;
          }
        }
      }
    }
  } else {
    int col = colStart + threadIdx.x;

    T val[kColLoad];

#pragma unroll
    for (int i = 0; i < kColLoad; ++i) {
      val[i] = input[col + i * blockDim.x];
    }

    if (endRow) {
      for (int row = rowStart; row < output.getSize(0); ++row) {
#pragma unroll
        for (int i = 0; i < kColLoad; ++i) {
          output[row][col + i * blockDim.x] = val[i];
        }
      }
    } else {
      for (int row = rowStart; row < rowEnd; row += kRowUnroll) {
#pragma unroll
        for (int i = 0; i < kRowUnroll; ++i) {
#pragma unroll
          for (int j = 0; j < kColLoad; ++j) {
            output[row + i][col + j * blockDim.x] = val[j];
          }
        }
      }
    }
  }
}

template <typename T, bool ZeroClamp>
__global__ void sumAlongRows(Tensor<T, 1, true> input,
                             Tensor<T, 2, true> output) {
  __shared__ T sval;

  int row = blockIdx.x;

  if (threadIdx.x == 0) {
    sval = input[row];
  }

  __syncthreads();

  T val = sval;

  // FIXME: speed up
  for (int i = threadIdx.x; i < output.getSize(1); i += blockDim.x) {
    T out = output[row][i];
    out = Math<T>::add(out, val);
    out = Math<T>::lt(out, Math<T>::zero()) ? Math<T>::zero() : out;

    output[row][i] = out;
  }
}

template <typename T, typename TVec>
void runSumAlongColumns(Tensor<T, 1, true>& input,
                        Tensor<T, 2, true>& output,
                        hipStream_t stream) {
  FAISS_ASSERT(input.getSize(0) == output.getSize(1));

  int threadsPerBlock = 256;
  constexpr int kRowUnroll = 4;
  constexpr int kRowsPerBlock = kRowUnroll * 4;
  constexpr int kColLoad = 4;

  auto block = dim3(threadsPerBlock);

  if (input.template canCastResize<TVec>() &&
      output.template canCastResize<TVec>()) {
    auto inputV = input.template castResize<TVec>();
    auto outputV = output.template castResize<TVec>();

    auto grid =
      dim3(utils::divUp(outputV.getSize(0), kRowsPerBlock),
           utils::divUp(outputV.getSize(1), threadsPerBlock * kColLoad));

    sumAlongColumns<TVec, kRowsPerBlock, kRowUnroll, kColLoad>
      <<<grid, block, 0, stream>>>(inputV, outputV);
  } else {
    auto grid =
      dim3(utils::divUp(output.getSize(0), kRowsPerBlock),
           utils::divUp(output.getSize(1), threadsPerBlock * kColLoad));

    sumAlongColumns<T, kRowsPerBlock, kRowUnroll, kColLoad>
      <<<grid, block, 0, stream>>>(input, output);
  }

  CUDA_TEST_ERROR();
}

void runSumAlongColumns(Tensor<float, 1, true>& input,
                        Tensor<float, 2, true>& output,
                        hipStream_t stream) {
  runSumAlongColumns<float, float4>(input, output, stream);
}

void runSumAlongColumns(Tensor<half, 1, true>& input,
                        Tensor<half, 2, true>& output,
                        hipStream_t stream) {
  runSumAlongColumns<half, half2>(input, output, stream);
}

template <typename T, typename TVec>
void runAssignAlongColumns(Tensor<T, 1, true>& input,
                           Tensor<T, 2, true>& output,
                           hipStream_t stream) {
  FAISS_ASSERT(input.getSize(0) == output.getSize(1));

  int threadsPerBlock = 256;
  constexpr int kRowUnroll = 4;
  constexpr int kRowsPerBlock = kRowUnroll * 4;
  constexpr int kColLoad = 4;

  auto block = dim3(threadsPerBlock);

  if (input.template canCastResize<TVec>() &&
      output.template canCastResize<TVec>()) {
    auto inputV = input.template castResize<TVec>();
    auto outputV = output.template castResize<TVec>();

    auto grid =
      dim3(utils::divUp(outputV.getSize(0), kRowsPerBlock),
           utils::divUp(outputV.getSize(1), threadsPerBlock * kColLoad));

    assignAlongColumns<TVec, kRowsPerBlock, kRowUnroll, kColLoad>
      <<<grid, block, 0, stream>>>(inputV, outputV);
  } else {
    auto grid =
      dim3(utils::divUp(output.getSize(0), kRowsPerBlock),
           utils::divUp(output.getSize(1), threadsPerBlock * kColLoad));

    assignAlongColumns<T, kRowsPerBlock, kRowUnroll, kColLoad>
      <<<grid, block, 0, stream>>>(input, output);
  }

  CUDA_TEST_ERROR();
}

void runAssignAlongColumns(Tensor<float, 1, true>& input,
                           Tensor<float, 2, true>& output,
                           hipStream_t stream) {
  runAssignAlongColumns<float, float4>(input, output, stream);
}

void runAssignAlongColumns(Tensor<half, 1, true>& input,
                           Tensor<half, 2, true>& output,
                           hipStream_t stream) {
  runAssignAlongColumns<half, half2>(input, output, stream);
}

template <typename T>
void runSumAlongRows(Tensor<T, 1, true>& input,
                     Tensor<T, 2, true>& output,
                     bool zeroClamp,
                     hipStream_t stream) {
  FAISS_ASSERT(input.getSize(0) == output.getSize(0));

  int threadsPerBlock =
    std::min(output.getSize(1), getMaxThreadsCurrentDevice());
  auto grid = dim3(output.getSize(0));
  auto block = dim3(threadsPerBlock);

  if (zeroClamp) {
    sumAlongRows<T, true><<<grid, block, 0, stream>>>(input, output);
  } else {
    sumAlongRows<T, false><<<grid, block, 0, stream>>>(input, output);
  }

  CUDA_TEST_ERROR();
}

void runSumAlongRows(Tensor<float, 1, true>& input,
                     Tensor<float, 2, true>& output,
                     bool zeroClamp,
                     hipStream_t stream) {
  runSumAlongRows<float>(input, output, zeroClamp, stream);
}

void runSumAlongRows(Tensor<half, 1, true>& input,
                     Tensor<half, 2, true>& output,
                     bool zeroClamp,
                     hipStream_t stream) {
  runSumAlongRows<half>(input, output, zeroClamp, stream);
}

} } // namespace
