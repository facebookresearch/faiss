#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <faiss/gpu/GpuResources.h>
#include <faiss/gpu/impl/RemapIndices.h>
#include <faiss/gpu/utils/DeviceUtils.h>
#include <faiss/invlists/InvertedLists.h>
#include <thrust/host_vector.h>
#include <faiss/gpu/impl/FlatIndex.cuh>
#include <faiss/gpu/impl/IVFAppend.cuh>
#include <faiss/gpu/impl/IVFBase.cuh>
#include <faiss/gpu/utils/CopyUtils.cuh>
#include <faiss/gpu/utils/DeviceDefs.cuh>
#include <faiss/gpu/utils/HostTensor.cuh>
#include <limits>
#include <unordered_map>

namespace faiss {
namespace gpu {

IVFBase::DeviceIVFList::DeviceIVFList(GpuResources* res, const AllocInfo& info)
        : data(res, info), numVecs(0) {}

IVFBase::IVFBase(
        GpuResources* resources,
        faiss::MetricType metric,
        float metricArg,
        FlatIndex* quantizer,
        bool interleavedLayout,
        IndicesOptions indicesOptions,
        MemorySpace space)
        : resources_(resources),
          metric_(metric),
          metricArg_(metricArg),
          quantizer_(quantizer),
          dim_(quantizer->getDim()),
          numLists_(quantizer->getSize()),
          interleavedLayout_(interleavedLayout),
          indicesOptions_(indicesOptions),
          space_(space),
          maxListLength_(0) {
    reset();
}

IVFBase::~IVFBase() {}

void IVFBase::reserveMemory(size_t numVecs) {
    auto stream = resources_->getDefaultStreamCurrentDevice();

    auto vecsPerList = numVecs / deviceListData_.size();
    if (vecsPerList < 1) {
        return;
    }

    auto bytesPerDataList = getGpuVectorsEncodingSize_(vecsPerList);

    for (auto& list : deviceListData_) {
        list->data.reserve(bytesPerDataList, stream);
    }

    if ((indicesOptions_ == INDICES_32_BIT) ||
        (indicesOptions_ == INDICES_64_BIT)) {
        // Reserve for index lists as well
        size_t bytesPerIndexList = vecsPerList *
                (indicesOptions_ == INDICES_32_BIT ? sizeof(int)
                                                   : sizeof(Index::idx_t));

        for (auto& list : deviceListIndices_) {
            list->data.reserve(bytesPerIndexList, stream);
        }
    }

    // Update device info for all lists, since the base pointers may
    // have changed
    updateDeviceListInfo_(stream);
}

void IVFBase::reset() {
    deviceListData_.clear();
    deviceListIndices_.clear();
    deviceListDataPointers_.clear();
    deviceListIndexPointers_.clear();
    deviceListLengths_.clear();
    listOffsetToUserIndex_.clear();

    auto info = AllocInfo(
            AllocType::IVFLists,
            getCurrentDevice(),
            space_,
            resources_->getDefaultStreamCurrentDevice());

    for (size_t i = 0; i < numLists_; ++i) {
        deviceListData_.emplace_back(std::unique_ptr<DeviceIVFList>(
                new DeviceIVFList(resources_, info)));

        deviceListIndices_.emplace_back(std::unique_ptr<DeviceIVFList>(
                new DeviceIVFList(resources_, info)));

        listOffsetToUserIndex_.emplace_back(std::vector<Index::idx_t>());
    }

    deviceListDataPointers_.resize(numLists_, nullptr);
    deviceListIndexPointers_.resize(numLists_, nullptr);
    deviceListLengths_.resize(numLists_, 0);
    maxListLength_ = 0;
}

int IVFBase::getDim() const {
    return dim_;
}

size_t IVFBase::reclaimMemory() {
    // Reclaim all unused memory exactly
    return reclaimMemory_(true);
}

size_t IVFBase::reclaimMemory_(bool exact) {
    auto stream = resources_->getDefaultStreamCurrentDevice();

    size_t totalReclaimed = 0;

    for (int i = 0; i < deviceListData_.size(); ++i) {
        auto& data = deviceListData_[i]->data;
        totalReclaimed += data.reclaim(exact, stream);

        deviceListDataPointers_[i] = data.data();
    }

    for (int i = 0; i < deviceListIndices_.size(); ++i) {
        auto& indices = deviceListIndices_[i]->data;
        totalReclaimed += indices.reclaim(exact, stream);

        deviceListIndexPointers_[i] = indices.data();
    }

    // Update device info for all lists, since the base pointers may
    // have changed
    updateDeviceListInfo_(stream);

    return totalReclaimed;
}

void IVFBase::updateDeviceListInfo_(hipStream_t stream) {
    std::vector<int> listIds(deviceListData_.size());
    for (int i = 0; i < deviceListData_.size(); ++i) {
        listIds[i] = i;
    }

    updateDeviceListInfo_(listIds, stream);
}

void IVFBase::updateDeviceListInfo_(
        const std::vector<int>& listIds,
        hipStream_t stream) {
    HostTensor<int, 1, true> hostListsToUpdate({(int)listIds.size()});
    HostTensor<int, 1, true> hostNewListLength({(int)listIds.size()});
    HostTensor<void*, 1, true> hostNewDataPointers({(int)listIds.size()});
    HostTensor<void*, 1, true> hostNewIndexPointers({(int)listIds.size()});

    for (int i = 0; i < listIds.size(); ++i) {
        auto listId = listIds[i];
        auto& data = deviceListData_[listId];
        auto& indices = deviceListIndices_[listId];

        hostListsToUpdate[i] = listId;
        hostNewListLength[i] = data->numVecs;
        hostNewDataPointers[i] = data->data.data();
        hostNewIndexPointers[i] = indices->data.data();
    }

    // Copy the above update sets to the GPU
    DeviceTensor<int, 1, true> listsToUpdate(
            resources_,
            makeTempAlloc(AllocType::Other, stream),
            hostListsToUpdate);
    DeviceTensor<int, 1, true> newListLength(
            resources_,
            makeTempAlloc(AllocType::Other, stream),
            hostNewListLength);
    DeviceTensor<void*, 1, true> newDataPointers(
            resources_,
            makeTempAlloc(AllocType::Other, stream),
            hostNewDataPointers);
    DeviceTensor<void*, 1, true> newIndexPointers(
            resources_,
            makeTempAlloc(AllocType::Other, stream),
            hostNewIndexPointers);

    // Update all pointers to the lists on the device that may have
    // changed
    runUpdateListPointers(
            listsToUpdate,
            newListLength,
            newDataPointers,
            newIndexPointers,
            deviceListLengths_,
            deviceListDataPointers_,
            deviceListIndexPointers_,
            stream);
}

size_t IVFBase::getNumLists() const {
    return numLists_;
}

int IVFBase::getListLength(int listId) const {
    FAISS_THROW_IF_NOT_FMT(
            listId < numLists_,
            "IVF list %d is out of bounds (%d lists total)",
            listId,
            numLists_);
    FAISS_ASSERT(listId < deviceListLengths_.size());
    FAISS_ASSERT(listId < deviceListData_.size());

    // LHS is the GPU resident value, RHS is the CPU resident value
    FAISS_ASSERT(
            deviceListLengths_[listId] == deviceListData_[listId]->numVecs);

    return deviceListData_[listId]->numVecs;
}

std::vector<Index::idx_t> IVFBase::getListIndices(int listId) const {
    FAISS_THROW_IF_NOT_FMT(
            listId < numLists_,
            "IVF list %d is out of bounds (%d lists total)",
            listId,
            numLists_);
    FAISS_ASSERT(listId < deviceListData_.size());
    FAISS_ASSERT(listId < deviceListLengths_.size());

    auto stream = resources_->getDefaultStreamCurrentDevice();

    if (indicesOptions_ == INDICES_32_BIT) {
        // The data is stored as int32 on the GPU
        FAISS_ASSERT(listId < deviceListIndices_.size());

        auto intInd = deviceListIndices_[listId]->data.copyToHost<int>(stream);

        std::vector<Index::idx_t> out(intInd.size());
        for (size_t i = 0; i < intInd.size(); ++i) {
            out[i] = (Index::idx_t)intInd[i];
        }

        return out;
    } else if (indicesOptions_ == INDICES_64_BIT) {
        // The data is stored as int64 on the GPU
        FAISS_ASSERT(listId < deviceListIndices_.size());

        return deviceListIndices_[listId]->data.copyToHost<Index::idx_t>(
                stream);
    } else if (indicesOptions_ == INDICES_CPU) {
        // The data is not stored on the GPU
        FAISS_ASSERT(listId < listOffsetToUserIndex_.size());

        auto& userIds = listOffsetToUserIndex_[listId];

        // We should have the same number of indices on the CPU as we do vectors
        // encoded on the GPU
        FAISS_ASSERT(userIds.size() == deviceListData_[listId]->numVecs);

        // this will return a copy
        return userIds;
    } else {
        // unhandled indices type (includes INDICES_IVF)
        FAISS_ASSERT(false);
        return std::vector<Index::idx_t>();
    }
}

std::vector<uint8_t> IVFBase::getListVectorData(int listId, bool gpuFormat)
        const {
    FAISS_THROW_IF_NOT_FMT(
            listId < numLists_,
            "IVF list %d is out of bounds (%d lists total)",
            listId,
            numLists_);
    FAISS_ASSERT(listId < deviceListData_.size());
    FAISS_ASSERT(listId < deviceListLengths_.size());

    auto stream = resources_->getDefaultStreamCurrentDevice();

    auto& list = deviceListData_[listId];
    auto gpuCodes = list->data.copyToHost<uint8_t>(stream);

    if (gpuFormat) {
        return gpuCodes;
    } else {
        // The GPU layout may be different than the CPU layout (e.g., vectors
        // rather than dimensions interleaved), translate back if necessary
        return translateCodesFromGpu_(std::move(gpuCodes), list->numVecs);
    }
}

void IVFBase::copyInvertedListsFrom(const InvertedLists* ivf) {
    size_t nlist = ivf ? ivf->nlist : 0;
    for (size_t i = 0; i < nlist; ++i) {
        size_t listSize = ivf->list_size(i);

        // GPU index can only support max int entries per list
        FAISS_THROW_IF_NOT_FMT(
                listSize <= (size_t)std::numeric_limits<int>::max(),
                "GPU inverted list can only support "
                "%zu entries; %zu found",
                (size_t)std::numeric_limits<int>::max(),
                listSize);

        addEncodedVectorsToList_(
                i, ivf->get_codes(i), ivf->get_ids(i), listSize);
    }
}

void IVFBase::copyInvertedListsTo(InvertedLists* ivf) {
    for (int i = 0; i < numLists_; ++i) {
        auto listIndices = getListIndices(i);
        auto listData = getListVectorData(i, false);

        ivf->add_entries(
                i, listIndices.size(), listIndices.data(), listData.data());
    }
}

void IVFBase::addEncodedVectorsToList_(
        int listId,
        const void* codes,
        const Index::idx_t* indices,
        size_t numVecs) {
    auto stream = resources_->getDefaultStreamCurrentDevice();

    // This list must already exist
    FAISS_ASSERT(listId < deviceListData_.size());

    // This list must currently be empty
    auto& listCodes = deviceListData_[listId];
    FAISS_ASSERT(listCodes->data.size() == 0);
    FAISS_ASSERT(listCodes->numVecs == 0);

    // If there's nothing to add, then there's nothing we have to do
    if (numVecs == 0) {
        return;
    }

    // The GPU might have a different layout of the memory
    auto gpuListSizeInBytes = getGpuVectorsEncodingSize_(numVecs);
    auto cpuListSizeInBytes = getCpuVectorsEncodingSize_(numVecs);

    // We only have int32 length representaz3tions on the GPU per each
    // list; the length is in sizeof(char)
    FAISS_ASSERT(gpuListSizeInBytes <= (size_t)std::numeric_limits<int>::max());

    // Translate the codes as needed to our preferred form
    std::vector<uint8_t> codesV(cpuListSizeInBytes);
    std::memcpy(codesV.data(), codes, cpuListSizeInBytes);
    auto translatedCodes = translateCodesToGpu_(std::move(codesV), numVecs);

    listCodes->data.append(
            translatedCodes.data(),
            gpuListSizeInBytes,
            stream,
            true /* exact reserved size */);
    listCodes->numVecs = numVecs;

    // Handle the indices as well
    addIndicesFromCpu_(listId, indices, numVecs);

    deviceListDataPointers_[listId] = listCodes->data.data();
    deviceListLengths_[listId] = numVecs;

    // We update this as well, since the multi-pass algorithm uses it
    maxListLength_ = std::max(maxListLength_, (int)numVecs);

    // device_vector add is potentially happening on a different stream
    // than our default stream
    if (resources_->getDefaultStreamCurrentDevice() != 0) {
        streamWait({stream}, {0});
    }
}

void IVFBase::addIndicesFromCpu_(
        int listId,
        const Index::idx_t* indices,
        size_t numVecs) {
    auto stream = resources_->getDefaultStreamCurrentDevice();

    // This list must currently be empty
    auto& listIndices = deviceListIndices_[listId];
    FAISS_ASSERT(listIndices->data.size() == 0);
    FAISS_ASSERT(listIndices->numVecs == 0);

    if (indicesOptions_ == INDICES_32_BIT) {
        // Make sure that all indices are in bounds
        std::vector<int> indices32(numVecs);
        for (size_t i = 0; i < numVecs; ++i) {
            auto ind = indices[i];
            FAISS_ASSERT(ind <= (Index::idx_t)std::numeric_limits<int>::max());
            indices32[i] = (int)ind;
        }

        static_assert(sizeof(int) == 4, "");

        listIndices->data.append(
                (uint8_t*)indices32.data(),
                numVecs * sizeof(int),
                stream,
                true /* exact reserved size */);

    } else if (indicesOptions_ == INDICES_64_BIT) {
        listIndices->data.append(
                (uint8_t*)indices,
                numVecs * sizeof(Index::idx_t),
                stream,
                true /* exact reserved size */);
    } else if (indicesOptions_ == INDICES_CPU) {
        // indices are stored on the CPU
        FAISS_ASSERT(listId < listOffsetToUserIndex_.size());

        auto& userIndices = listOffsetToUserIndex_[listId];
        userIndices.insert(userIndices.begin(), indices, indices + numVecs);
    } else {
        // indices are not stored
        FAISS_ASSERT(indicesOptions_ == INDICES_IVF);
    }

    deviceListIndexPointers_[listId] = listIndices->data.data();
}

int IVFBase::addVectors(
        Tensor<float, 2, true>& vecs,
        Tensor<Index::idx_t, 1, true>& indices) {
    FAISS_ASSERT(vecs.getSize(0) == indices.getSize(0));
    FAISS_ASSERT(vecs.getSize(1) == dim_);

    auto stream = resources_->getDefaultStreamCurrentDevice();

    // Determine which IVF lists we need to append to

    // We don't actually need this
    DeviceTensor<float, 2, true> listDistance(
            resources_,
            makeTempAlloc(AllocType::Other, stream),
            {vecs.getSize(0), 1});
    // We use this
    DeviceTensor<int, 2, true> listIds2d(
            resources_,
            makeTempAlloc(AllocType::Other, stream),
            {vecs.getSize(0), 1});

    quantizer_->query(
            vecs, 1, metric_, metricArg_, listDistance, listIds2d, false);

    // Copy the lists that we wish to append to back to the CPU
    // FIXME: really this can be into pinned memory and a true async
    // copy on a different stream; we can start the copy early, but it's
    // tiny
    auto listIdsHost = listIds2d.copyToVector(stream);

    // Now we add the encoded vectors to the individual lists
    // First, make sure that there is space available for adding the new
    // encoded vectors and indices

    // list id -> vectors being added
    std::unordered_map<int, std::vector<int>> listToVectorIds;

    // vector id -> which list it is being appended to
    std::vector<int> vectorIdToList(vecs.getSize(0));

    // vector id -> offset in list
    // (we already have vector id -> list id in listIds)
    std::vector<int> listOffsetHost(listIdsHost.size());

    // Number of valid vectors that we actually add; we return this
    int numAdded = 0;

    for (int i = 0; i < listIdsHost.size(); ++i) {
        int listId = listIdsHost[i];

        // Add vector could be invalid (contains NaNs etc)
        if (listId < 0) {
            listOffsetHost[i] = -1;
            vectorIdToList[i] = -1;
            continue;
        }

        FAISS_ASSERT(listId < numLists_);
        ++numAdded;
        vectorIdToList[i] = listId;

        int offset = deviceListData_[listId]->numVecs;

        auto it = listToVectorIds.find(listId);
        if (it != listToVectorIds.end()) {
            offset += it->second.size();
            it->second.push_back(i);
        } else {
            listToVectorIds[listId] = std::vector<int>{i};
        }

        listOffsetHost[i] = offset;
    }

    // If we didn't add anything (all invalid vectors that didn't map to IVF
    // clusters), no need to continue
    if (numAdded == 0) {
        return 0;
    }

    // unique lists being added to
    std::vector<int> uniqueLists;

    for (auto& vecs : listToVectorIds) {
        uniqueLists.push_back(vecs.first);
    }

    std::sort(uniqueLists.begin(), uniqueLists.end());

    // In the same order as uniqueLists, list the vectors being added to that
    // list contiguously (unique list 0 vectors ...)(unique list 1 vectors ...)
    // ...
    std::vector<int> vectorsByUniqueList;

    // For each of the unique lists, the start offset in vectorsByUniqueList
    std::vector<int> uniqueListVectorStart;

    // For each of the unique lists, where we start appending in that list by
    // the vector offset
    std::vector<int> uniqueListStartOffset;

    // For each of the unique lists, find the vectors which should be appended
    // to that list
    for (auto ul : uniqueLists) {
        uniqueListVectorStart.push_back(vectorsByUniqueList.size());

        FAISS_ASSERT(listToVectorIds.count(ul) != 0);

        // The vectors we are adding to this list
        auto& vecs = listToVectorIds[ul];
        vectorsByUniqueList.insert(
                vectorsByUniqueList.end(), vecs.begin(), vecs.end());

        // How many vectors we previously had (which is where we start appending
        // on the device)
        uniqueListStartOffset.push_back(deviceListData_[ul]->numVecs);
    }

    // We terminate uniqueListVectorStart with the overall number of vectors
    // being added, which could be different than vecs.getSize(0) as some
    // vectors could be invalid
    uniqueListVectorStart.push_back(vectorsByUniqueList.size());

    // We need to resize the data structures for the inverted lists on
    // the GPUs, which means that they might need reallocation, which
    // means that their base address may change. Figure out the new base
    // addresses, and update those in a batch on the device
    {
        // Resize all of the lists that we are appending to
        for (auto& counts : listToVectorIds) {
            auto listId = counts.first;
            int numVecsToAdd = counts.second.size();

            auto& codes = deviceListData_[listId];
            int oldNumVecs = codes->numVecs;
            int newNumVecs = codes->numVecs + numVecsToAdd;

            auto newSizeBytes = getGpuVectorsEncodingSize_(newNumVecs);
            codes->data.resize(newSizeBytes, stream);
            codes->numVecs = newNumVecs;

            auto& indices = deviceListIndices_[listId];
            if ((indicesOptions_ == INDICES_32_BIT) ||
                (indicesOptions_ == INDICES_64_BIT)) {
                size_t indexSize = (indicesOptions_ == INDICES_32_BIT)
                        ? sizeof(int)
                        : sizeof(Index::idx_t);

                indices->data.resize(
                        indices->data.size() + numVecsToAdd * indexSize,
                        stream);
                FAISS_ASSERT(indices->numVecs == oldNumVecs);
                indices->numVecs = newNumVecs;

            } else if (indicesOptions_ == INDICES_CPU) {
                // indices are stored on the CPU side
                FAISS_ASSERT(listId < listOffsetToUserIndex_.size());

                auto& userIndices = listOffsetToUserIndex_[listId];
                userIndices.resize(newNumVecs);
            } else {
                // indices are not stored on the GPU or CPU side
                FAISS_ASSERT(indicesOptions_ == INDICES_IVF);
            }

            // This is used by the multi-pass query to decide how much scratch
            // space to allocate for intermediate results
            maxListLength_ = std::max(maxListLength_, newNumVecs);
        }

        // Update all pointers and sizes on the device for lists that we
        // appended to
        updateDeviceListInfo_(uniqueLists, stream);
    }

    // If we're maintaining the indices on the CPU side, update our
    // map. We already resized our map above.
    if (indicesOptions_ == INDICES_CPU) {
        // We need to maintain the indices on the CPU side
        HostTensor<Index::idx_t, 1, true> hostIndices(indices, stream);

        for (int i = 0; i < hostIndices.getSize(0); ++i) {
            int listId = listIdsHost[i];

            // Add vector could be invalid (contains NaNs etc)
            if (listId < 0) {
                continue;
            }

            int offset = listOffsetHost[i];
            FAISS_ASSERT(offset >= 0);

            FAISS_ASSERT(listId < listOffsetToUserIndex_.size());
            auto& userIndices = listOffsetToUserIndex_[listId];

            FAISS_ASSERT(offset < userIndices.size());
            userIndices[offset] = hostIndices[i];
        }
    }

    // Copy the offsets to the GPU
    auto listIdsDevice = listIds2d.downcastOuter<1>();
    auto listOffsetDevice =
            toDeviceTemporary(resources_, listOffsetHost, stream);
    auto uniqueListsDevice = toDeviceTemporary(resources_, uniqueLists, stream);
    auto vectorsByUniqueListDevice =
            toDeviceTemporary(resources_, vectorsByUniqueList, stream);
    auto uniqueListVectorStartDevice =
            toDeviceTemporary(resources_, uniqueListVectorStart, stream);
    auto uniqueListStartOffsetDevice =
            toDeviceTemporary(resources_, uniqueListStartOffset, stream);

    // Actually encode and append the vectors
    appendVectors_(
            vecs,
            indices,
            uniqueListsDevice,
            vectorsByUniqueListDevice,
            uniqueListVectorStartDevice,
            uniqueListStartOffsetDevice,
            listIdsDevice,
            listOffsetDevice,
            stream);

    // We added this number
    return numAdded;
}

} // namespace gpu
} // namespace faiss
