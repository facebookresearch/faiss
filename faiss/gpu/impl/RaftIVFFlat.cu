#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <raft/core/device_mdspan.hpp>
#include <raft/core/handle.hpp>
#include <cstdint>
#include <raft/neighbors/ivf_flat.cuh>

#include <faiss/gpu/GpuIndex.h>
#include <faiss/gpu/GpuResources.h>
#include <faiss/gpu/impl/InterleavedCodes.h>
#include <faiss/gpu/impl/RemapIndices.h>
#include <faiss/gpu/utils/DeviceUtils.h>
#include <thrust/host_vector.h>
#include <faiss/gpu/impl/FlatIndex.cuh>
#include <faiss/gpu/impl/IVFAppend.cuh>
#include <faiss/gpu/impl/IVFFlat.cuh>
#include <faiss/gpu/impl/IVFFlatScan.cuh>
#include <faiss/gpu/impl/IVFInterleaved.cuh>
#include <faiss/gpu/impl/RaftIVFFlat.cuh>
#include <faiss/gpu/utils/ConversionOperators.cuh>
#include <faiss/gpu/utils/CopyUtils.cuh>
#include <faiss/gpu/utils/DeviceDefs.cuh>
#include <faiss/gpu/utils/Float16.cuh>
#include <faiss/gpu/utils/HostTensor.cuh>
#include <faiss/gpu/utils/Transpose.cuh>
#include <limits>
#include <unordered_map>


#include <raft/neighbors/ivf_flat_types.hpp>
#include <raft/neighbors/ivf_flat_helpers.cuh>

#include <raft/core/logger.hpp>

namespace faiss {
namespace gpu {

RaftIVFFlat::RaftIVFFlat(
        GpuResources* res,
        int dim,
        int nlist,
        faiss::MetricType metric,
        float metricArg,
        bool useResidual,
        faiss::ScalarQuantizer* scalarQ,
        bool interleavedLayout,
        IndicesOptions indicesOptions,
        MemorySpace space)
        : IVFFlat(res,
                  dim,
                  nlist,
                  metric,
                  metricArg,
                  useResidual,
                  scalarQ,
                  interleavedLayout,
                  indicesOptions,
                  space) {}

RaftIVFFlat::~RaftIVFFlat() {}

/// Find the approximate k nearest neighbors for `queries` against
/// our database
void RaftIVFFlat::search(
        Index* coarseQuantizer,
        Tensor<float, 2, true>& queries,
        int nprobe,
        int k,
        Tensor<float, 2, true>& outDistances,
        Tensor<idx_t, 2, true>& outIndices) {

    // TODO: We probably don't want to ignore the coarse quantizer here...

    std::uint32_t n = queries.getSize(0);
    std::uint32_t cols = queries.getSize(1);
    std::uint32_t k_ = k;

    // Device is already set in GpuIndex::search
    FAISS_ASSERT(raft_knn_index.has_value());
    FAISS_ASSERT(n > 0);
    FAISS_THROW_IF_NOT(nprobe > 0 && nprobe <= numLists_);

    const raft::device_resources& raft_handle =
            resources_->getRaftHandleCurrentDevice();
    raft::neighbors::ivf_flat::search_params pams;
    pams.n_probes = nprobe;

    auto queries_view =
            raft::make_device_matrix_view<const float>(queries.data(), n, cols);
    auto out_inds_view =
            raft::make_device_matrix_view<idx_t>(outIndices.data(), n, k_);
    auto out_dists_view =
            raft::make_device_matrix_view<float>(outDistances.data(), n, k_);
    raft::neighbors::ivf_flat::search<float, idx_t>(
            raft_handle,
	    pams,
            raft_knn_index.value(),
            queries_view,
            out_inds_view,
            out_dists_view);

    raft_handle.sync_stream();
}

/// Classify and encode/add vectors to our IVF lists.
/// The input data must be on our current device.
/// Returns the number of vectors successfully added. Vectors may
/// not be able to be added because they contain NaNs.
idx_t RaftIVFFlat::addVectors(
        Index* coarseQuantizer,
        Tensor<float, 2, true>& vecs,
        Tensor<idx_t, 1, true>& indices) {

    auto vecs_view = raft::make_device_matrix_view<const float, idx_t>(
            vecs.data(), vecs.getSize(0), dim_);
    auto inds_view = raft::make_device_vector_view<const idx_t, idx_t>(
            indices.data(), (idx_t)indices.getSize(0));

    const raft::device_resources& raft_handle =
            resources_->getRaftHandleCurrentDevice();

    // TODO: We probably don't want to ignore the coarse quantizer here

    if (raft_knn_index.has_value()) {
        raft_knn_index.emplace(raft::neighbors::ivf_flat::extend(
                raft_handle,
                vecs_view,
                std::make_optional<
                        raft::device_vector_view<const idx_t, idx_t>>(
                        inds_view),
	        raft_knn_index.value()));

    }
    return vecs.getSize(0);
}

void RaftIVFFlat::reset() {
    raft_knn_index.reset();
}

idx_t RaftIVFFlat::getListLength(idx_t listId) const {

    FAISS_ASSERT(raft_knn_index.has_value());
    const raft::device_resources& raft_handle =
            resources_->getRaftHandleCurrentDevice();

    uint32_t size;
    raft::copy(
            &size,
            raft_knn_index.value().list_sizes().data_handle() + listId,
            1,
            raft_handle.get_stream());
    raft_handle.sync_stream();
    return int(size);
}

/// Return the list indices of a particular list back to the CPU
std::vector<idx_t> RaftIVFFlat::getListIndices(idx_t listId) const {

    FAISS_ASSERT(raft_knn_index.has_value());
    const raft::device_resources& raft_handle =
            resources_->getRaftHandleCurrentDevice();
    auto stream = raft_handle.get_stream();

    idx_t listSize = getListLength(listId);

    std::vector<idx_t> vec(listSize);

    idx_t* list_indices_ptr;

    // fetch the list indices ptr on host
    raft::update_host(&list_indices_ptr, raft_knn_index.value().inds_ptrs().data_handle()+listId, 1, stream);
    raft_handle.sync_stream();

    raft::update_host(vec.data(), list_indices_ptr, listSize, stream);
    raft_handle.sync_stream();
    return vec;
}

/// Return the encoded vectors of a particular list back to the CPU
std::vector<uint8_t> RaftIVFFlat::getListVectorData(idx_t listId, bool gpuFormat)
        const {

    FAISS_ASSERT(raft_knn_index.has_value());

    const raft::device_resources& raft_handle = resources_->getRaftHandleCurrentDevice();
    auto stream = raft_handle.get_stream();

    idx_t listSize = getListLength(listId);

    // the interleaved block can be slightly larger than the list size (it's
    // rounded up)
    auto gpuListSizeInBytes = getGpuVectorsEncodingSize_(listSize);
    auto cpuListSizeInBytes = getCpuVectorsEncodingSize_(listSize);

    std::vector<uint8_t> interleaved_codes(gpuListSizeInBytes);
    std::vector<uint8_t> flat_codes(cpuListSizeInBytes);

    float* list_data_ptr;

   // fetch the list data ptr on host
    raft::update_host(&list_data_ptr, raft_knn_index.value().data_ptrs().data_handle()+listId, 1, stream);
    raft_handle.sync_stream();

    raft::update_host(interleaved_codes.data(), reinterpret_cast<uint8_t*>(list_data_ptr), gpuListSizeInBytes, stream);
    raft_handle.sync_stream();

    RaftIVFFlatCodePackerInterleaved packer((size_t)listSize, dim_, raft_knn_index.value().veclen());
    packer.unpack_all(interleaved_codes.data(), flat_codes.data());
    return flat_codes;
}

/// Performs search when we are already given the IVF cells to look at
/// (GpuIndexIVF::search_preassigned implementation)
void RaftIVFFlat::searchPreassigned(
        Index* coarseQuantizer,
        Tensor<float, 2, true>& vecs,
        Tensor<float, 2, true>& ivfDistances,
        Tensor<idx_t, 2, true>& ivfAssignments,
        int k,
        Tensor<float, 2, true>& outDistances,
        Tensor<idx_t, 2, true>& outIndices,
        bool storePairs) {
    // TODO: Fill this in!
}

void RaftIVFFlat::updateQuantizer(Index* quantizer) {
    idx_t quantizer_ntotal = quantizer->ntotal;

    const raft::device_resources& handle = resources_->getRaftHandleCurrentDevice();
    auto stream = handle.get_stream();

    auto total_elems = size_t(quantizer_ntotal) * size_t(quantizer->d);

    raft::logger::get().set_level(RAFT_LEVEL_TRACE);

    raft::neighbors::ivf_flat::index_params pams;
    pams.add_data_on_build = false;

    pams.n_lists = this->numLists_;

    switch (this->metric_) {
        case faiss::METRIC_L2:
            pams.metric = raft::distance::DistanceType::L2Expanded;
            break;
        case faiss::METRIC_INNER_PRODUCT:
            pams.metric = raft::distance::DistanceType::InnerProduct;
            break;
        default:
            FAISS_THROW_MSG("Metric is not supported.");
    }

    raft_knn_index.emplace(
            handle,
            pams,
            (uint32_t)this->dim_);

    /// Copy (reconstructed) centroids over, rather than re-training
    std::vector<float> buf_host(total_elems);
    quantizer->reconstruct_n(0, quantizer_ntotal, buf_host.data());

    raft::update_device(
            raft_knn_index.value().centers().data_handle(),
            buf_host.data(),
            total_elems,
            stream);
}

//
//
void RaftIVFFlat::copyInvertedListsFrom(const InvertedLists* ivf) {
   size_t nlist = ivf ? ivf->nlist : 0;
   size_t ntotal = ivf ? ivf->compute_ntotal() : 0;

   raft::device_resources &raft_handle = resources_->getRaftHandleCurrentDevice();

   std::vector<std::uint32_t> list_sizes_(nlist);
   std::vector<idx_t> indices_(ntotal);

   // the index must already exist
   FAISS_ASSERT(raft_knn_index.has_value());

  auto& raft_lists = raft_knn_index.value().lists();

  // conservative memory alloc for cloning cpu inverted lists
  raft::neighbors::ivf_flat::list_spec<uint32_t, float, idx_t> raft_list_spec{static_cast<uint32_t>(dim_), true};

   for (size_t i = 0; i < nlist; ++i) {

        size_t listSize = ivf->list_size(i);

        // GPU index can only support max int entries per list
       FAISS_THROW_IF_NOT_FMT(
               listSize <= (size_t)std::numeric_limits<int>::max(),
               "GPU inverted list can only support "
               "%zu entries; %zu found",
               (size_t)std::numeric_limits<int>::max(),
               listSize);
        
        // store the list size
        list_sizes_[i] = static_cast<uint32_t>(listSize);

       raft::neighbors::ivf::resize_list(raft_handle,
                        raft_lists[i],
                       raft_list_spec,
                       (uint32_t)listSize,
                       (uint32_t)0);
   }

  // Update the pointers and the sizes
  raft_knn_index.value().recompute_internal_state(raft_handle);

        for (size_t i = 0; i < nlist; ++i) {
            size_t listSize = ivf->list_size(i);
            addEncodedVectorsToList_(i, ivf->get_codes(i), ivf->get_ids(i), listSize);
        }

    raft::update_device(raft_knn_index.value().list_sizes().data_handle(), list_sizes_.data(), nlist, raft_handle.get_stream());

        // Precompute the centers vector norms for L2Expanded distance
        if (this->metric_ == faiss::METRIC_L2) {
            raft_knn_index.value().allocate_center_norms(raft_handle);
            raft::linalg::rowNorm(raft_knn_index.value().center_norms()->data_handle(),
                            raft_knn_index.value().centers().data_handle(),
                            raft_knn_index.value().dim(),
                            (uint32_t)nlist,
                            raft::linalg::L2Norm,
                            true,
                            raft_handle.get_stream());
        }
        raft_handle.sync_stream();
}

size_t RaftIVFFlat::getGpuVectorsEncodingSize_(idx_t numVecs) const {
        idx_t bits = 32 /* float */;

        // bytes to encode a block of 32 vectors (single dimension)
        idx_t bytesPerDimBlock = bits * 32 / 8; // = 128

        // bytes to fully encode 32 vectors
        idx_t bytesPerBlock = bytesPerDimBlock * dim_;

        // number of blocks of 32 vectors we have
        idx_t numBlocks = utils::divUp(numVecs, raft::neighbors::ivf_flat::kIndexGroupSize);

        // total size to encode numVecs
        return bytesPerBlock * numBlocks;
}


void RaftIVFFlat::addEncodedVectorsToList_(
            idx_t listId,
            const void* codes,
            const idx_t* indices,
            idx_t numVecs) {
   auto stream = resources_->getDefaultStreamCurrentDevice();

   // This list must already exist
   FAISS_ASSERT(raft_knn_index.has_value());

   // This list must currently be empty
   FAISS_ASSERT(getListLength(listId) == 0);

   // If there's nothing to add, then there's nothing we have to do
   if (numVecs == 0) {
       return;
   }

   // The GPU might have a different layout of the memory
   auto gpuListSizeInBytes = getGpuVectorsEncodingSize_(numVecs);
   auto cpuListSizeInBytes = getCpuVectorsEncodingSize_(numVecs);

  // We only have int32 length representations on the GPU per each
  // list; the length is in sizeof(char)
   FAISS_ASSERT(gpuListSizeInBytes <=
   (size_t)std::numeric_limits<int>::max());

        std::vector<uint8_t> interleaved_codes(gpuListSizeInBytes);
   RaftIVFFlatCodePackerInterleaved packer((size_t)numVecs, (uint32_t)dim_, raft_knn_index.value().veclen());
   
   packer.pack_all(reinterpret_cast<const uint8_t*>(codes), interleaved_codes.data());

   float* list_data_ptr;
   const raft::device_resources& raft_handle = resources_->getRaftHandleCurrentDevice();

   /// fetch the list data ptr on host
    raft::update_host(&list_data_ptr, raft_knn_index.value().data_ptrs().data_handle()+listId, 1, stream);
    raft_handle.sync_stream();
   
   raft::update_device(reinterpret_cast<uint8_t*>(list_data_ptr), interleaved_codes.data(), gpuListSizeInBytes, stream);
   raft_handle.sync_stream();

    /// Handle the indices as well
    idx_t* list_indices_ptr;

    // fetch the list indices ptr on host
    raft::update_host(&list_indices_ptr, raft_knn_index.value().inds_ptrs().data_handle()+listId, 1, stream);
        raft_handle.sync_stream();
    raft::update_device(list_indices_ptr, indices, numVecs, stream);
    raft_handle.sync_stream();
}

RaftIVFFlatCodePackerInterleaved::RaftIVFFlatCodePackerInterleaved(size_t list_size, uint32_t dim, uint32_t chunk_size) {
    this->dim = dim;
    this->chunk_size = chunk_size;
    // NB: dim should be divisible by the number of 4 byte records in one chunk
    FAISS_ASSERT(dim % chunk_size == 0);
    nvec = list_size;
    code_size = dim * 4;
    block_size = utils::roundUp(nvec, raft::neighbors::ivf_flat::kIndexGroupSize);
}

void RaftIVFFlatCodePackerInterleaved::pack_1(const uint8_t* flat_code, size_t offset, uint8_t* block) const {
        // printf("packing offset %zu\n", offset);
    raft::neighbors::ivf_flat::codepacker::pack_1(
        reinterpret_cast<const uint32_t*>(flat_code),
        reinterpret_cast<uint32_t*>(block),
        dim,
        chunk_size,
        static_cast<uint32_t>(offset));
}

void RaftIVFFlatCodePackerInterleaved::unpack_1(const uint8_t* block, size_t offset, uint8_t* flat_code) const {
    raft::neighbors::ivf_flat::codepacker::unpack_1(
        reinterpret_cast<const uint32_t*>(block),
        reinterpret_cast<uint32_t*>(flat_code),
        dim,
        chunk_size,
        static_cast<uint32_t>(offset));
}

} // namespace gpu
} // namespace faiss
