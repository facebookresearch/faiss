#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <faiss/IndexFlat.h>
#include <faiss/gpu/GpuDistance.h>
#include <faiss/gpu/StandardGpuResources.h>
#include <faiss/gpu/test/TestUtils.h>
#include <faiss/gpu/utils/DeviceUtils.h>
#include <gtest/gtest.h>
#include <faiss/gpu/utils/CopyUtils.cuh>
#include <faiss/gpu/utils/Transpose.cuh>
#include <sstream>
#include <vector>

void evaluate_bfknn(
        faiss::gpu::GpuDistanceParams& args,
        faiss::gpu::GpuResourcesProvider* res,
        std::vector<float>& cpuDistance,
        std::vector<faiss::idx_t>& cpuIndices,
        std::vector<float>& gpuDistance,
        std::vector<faiss::idx_t>& gpuIndices,
        int numQuery,
        int k,
        bool colMajorVecs,
        bool colMajorQueries,
        faiss::MetricType metric) {
    using namespace faiss::gpu;

    bfKnn(res, args);

    std::stringstream str;
    str << "using raft " << args.use_raft << "metric " << metric
        << " colMajorVecs " << colMajorVecs << " colMajorQueries "
        << colMajorQueries;

    compareLists(
            cpuDistance.data(),
            cpuIndices.data(),
            gpuDistance.data(),
            gpuIndices.data(),
            numQuery,
            k,
            str.str(),
            false,
            false,
            true,
            6e-3f,
            0.1f,
            0.015f);
}

void testTransposition(
        bool colMajorVecs,
        bool colMajorQueries,
        faiss::MetricType metric,
        bool use_raft = false,
        float metricArg = 0) {
    using namespace faiss::gpu;

    int device = randVal(0, getNumDevices() - 1);

    StandardGpuResources res;
    res.noTempMemory();

    int dim = randVal(20, 150);
    int numVecs = randVal(10, 30000);
    int numQuery = randVal(1, 1024);
    int k = std::min(numVecs, randVal(20, 70));

    // Input data for CPU
    std::vector<float> vecs = randVecs(numVecs, dim);
    std::vector<float> queries = randVecs(numQuery, dim);

    if ((metric == faiss::MetricType::METRIC_JensenShannon) ||
        (metric == faiss::MetricType::METRIC_Jaccard)) {
        // make values positive
        for (auto& v : vecs) {
            v = std::abs(v);
            if (v == 0) {
                v = 1e-6;
            }
        }

        for (auto& q : queries) {
            q = std::abs(q);
            if (q == 0) {
                q = 1e-6;
            }
        }
    }

    // The CPU index is our reference for the results
    faiss::IndexFlat cpuIndex(dim, metric);
    cpuIndex.metric_arg = metricArg;
    cpuIndex.add(numVecs, vecs.data());

    std::vector<float> cpuDistance(numQuery * k, 0);
    std::vector<faiss::idx_t> cpuIndices(numQuery * k, -1);

    cpuIndex.search(
            numQuery, queries.data(), k, cpuDistance.data(), cpuIndices.data());

    // The transpose and distance code assumes the desired device is already set
    DeviceScope scope(device);
    auto stream = res.getDefaultStream(device);

    // Copy input data to GPU, and pre-transpose both vectors and queries for
    // passing
    auto gpuVecs = toDeviceNonTemporary<float, 2>(
            res.getResources().get(),
            device,
            vecs.data(),
            stream,
            {numVecs, dim});
    auto gpuQueries = toDeviceNonTemporary<float, 2>(
            res.getResources().get(),
            device,
            queries.data(),
            stream,
            {numQuery, dim});

    DeviceTensor<float, 2, true> vecsT(
            res.getResources().get(),
            makeDevAlloc(AllocType::Other, stream),
            {dim, numVecs});
    runTransposeAny(gpuVecs, 0, 1, vecsT, stream);

    DeviceTensor<float, 2, true> queriesT(
            res.getResources().get(),
            makeDevAlloc(AllocType::Other, stream),
            {dim, numQuery});
    runTransposeAny(gpuQueries, 0, 1, queriesT, stream);

    std::vector<float> gpuDistance(numQuery * k, 0);
    std::vector<faiss::idx_t> gpuIndices(numQuery * k, -1);

    GpuDistanceParams args;
    args.metric = metric;
    args.metricArg = metricArg;
    args.k = k;
    args.dims = dim;
    args.vectors = colMajorVecs ? vecsT.data() : gpuVecs.data();
    args.vectorsRowMajor = !colMajorVecs;
    args.numVectors = numVecs;
    args.queries = colMajorQueries ? queriesT.data() : gpuQueries.data();
    args.queriesRowMajor = !colMajorQueries;
    args.numQueries = numQuery;
    args.outDistances = gpuDistance.data();
    args.outIndices = gpuIndices.data();
    args.device = device;

#if defined USE_NVIDIA_RAFT
    args.use_raft = use_raft;
#else
    FAISS_THROW_IF_NOT_MSG(
            !use_raft,
            "RAFT has not been compiled into the current version so it cannot be used.");
#endif

    evaluate_bfknn(
            args,
            &res,
            cpuDistance,
            cpuIndices,
            gpuDistance,
            gpuIndices,
            numQuery,
            k,
            colMajorVecs,
            colMajorQueries,
            metric);
}

// Test different memory layouts for brute-force k-NN
TEST(TestGpuDistance, Transposition_RR) {
    testTransposition(false, false, faiss::MetricType::METRIC_L2);
    testTransposition(false, false, faiss::MetricType::METRIC_INNER_PRODUCT);
}

#if defined USE_NVIDIA_RAFT
TEST(TestRaftGpuDistance, Transposition_RR) {
    testTransposition(false, false, faiss::MetricType::METRIC_L2, true);
    testTransposition(
            false, false, faiss::MetricType::METRIC_INNER_PRODUCT, true);
}
#endif

TEST(TestGpuDistance, Transposition_RC) {
    testTransposition(false, true, faiss::MetricType::METRIC_L2);
}

#if defined USE_NVIDIA_RAFT
TEST(TestRaftGpuDistance, Transposition_RC) {
    testTransposition(false, true, faiss::MetricType::METRIC_L2, true);
}
#endif

TEST(TestGpuDistance, Transposition_CR) {
    testTransposition(true, false, faiss::MetricType::METRIC_L2);
}

#if defined USE_NVIDIA_RAFT
TEST(TestRaftGpuDistance, Transposition_CR) {
    testTransposition(true, false, faiss::MetricType::METRIC_L2, true);
}
#endif

TEST(TestGpuDistance, Transposition_CC) {
    testTransposition(true, true, faiss::MetricType::METRIC_L2);
}

#if defined USE_NVIDIA_RAFT
TEST(TestRaftGpuDistance, Transposition_CC) {
    testTransposition(true, true, faiss::MetricType::METRIC_L2, true);
}
#endif

TEST(TestGpuDistance, L1) {
    testTransposition(false, false, faiss::MetricType::METRIC_L1);
}

#if defined USE_NVIDIA_RAFT
TEST(TestRaftGpuDistance, L1) {
    testTransposition(false, false, faiss::MetricType::METRIC_L1, true);
}
#endif

// Test other transpositions with the general distance kernel
TEST(TestGpuDistance, L1_RC) {
    testTransposition(false, true, faiss::MetricType::METRIC_L1);
}

#if defined USE_NVIDIA_RAFT
// Test other transpositions with the general distance kernel
TEST(TestRaftGpuDistance, L1_RC) {
    testTransposition(false, true, faiss::MetricType::METRIC_L1, true);
}
#endif

TEST(TestGpuDistance, L1_CR) {
    testTransposition(true, false, faiss::MetricType::METRIC_L1);
}

#if defined USE_NVIDIA_RAFT
TEST(TestRaftGpuDistance, L1_CR) {
    testTransposition(true, false, faiss::MetricType::METRIC_L1, true);
}
#endif

TEST(TestGpuDistance, L1_CC) {
    testTransposition(true, true, faiss::MetricType::METRIC_L1);
}

#if defined USE_NVIDIA_RAFT
TEST(TestRaftGpuDistance, L1_CC) {
    testTransposition(true, true, faiss::MetricType::METRIC_L1, true);
}
#endif

// Test remainder of metric types
TEST(TestGpuDistance, Linf) {
    testTransposition(false, false, faiss::MetricType::METRIC_Linf);
}

#if defined USE_NVIDIA_RAFT
// Test remainder of metric types
TEST(TestRaftGpuDistance, Linf) {
    testTransposition(false, false, faiss::MetricType::METRIC_Linf, true);
}
#endif

TEST(TestGpuDistance, Lp) {
    testTransposition(false, false, faiss::MetricType::METRIC_Lp, false, 3);
}

#if defined USE_NVIDIA_RAFT
TEST(TestRaftGpuDistance, Lp) {
    testTransposition(false, false, faiss::MetricType::METRIC_Lp, true, 3);
}
#endif

TEST(TestGpuDistance, Canberra) {
    testTransposition(false, false, faiss::MetricType::METRIC_Canberra);
}

#if defined USE_NVIDIA_RAFT
TEST(TestRaftGpuDistance, Canberra) {
    testTransposition(false, false, faiss::MetricType::METRIC_Canberra, true);
}
#endif

TEST(TestGpuDistance, BrayCurtis) {
    testTransposition(false, false, faiss::MetricType::METRIC_BrayCurtis);
}

TEST(TestGpuDistance, JensenShannon) {
    testTransposition(false, false, faiss::MetricType::METRIC_JensenShannon);
}

#if defined USE_NVIDIA_RAFT
TEST(TestRaftGpuDistance, JensenShannon) {
    testTransposition(
            false, false, faiss::MetricType::METRIC_JensenShannon, true);
}
#endif

TEST(TestGpuDistance, Jaccard) {
    testTransposition(false, false, faiss::MetricType::METRIC_Jaccard);
}

int main(int argc, char** argv) {
    testing::InitGoogleTest(&argc, argv);

    // just run with a fixed test seed
    faiss::gpu::setTestSeed(100);

    return RUN_ALL_TESTS();
}
