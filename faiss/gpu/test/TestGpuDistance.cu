#include "hip/hip_runtime.h"
// @lint-ignore-every LICENSELINT
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <faiss/IndexFlat.h>
#include <faiss/gpu/GpuDistance.h>
#include <faiss/gpu/StandardGpuResources.h>
#include <faiss/gpu/test/TestUtils.h>
#include <faiss/gpu/utils/DeviceUtils.h>
#include <gtest/gtest.h>
#include <faiss/gpu/utils/CopyUtils.cuh>
#include <faiss/gpu/utils/Transpose.cuh>
#include <sstream>
#include <vector>

enum class TestThresholds {
    Normal,
    BF16,
    // Linf has worse error than the other metrics for bf16
    BF16_Linf,
};

void evaluate_bfknn(
        faiss::gpu::GpuDistanceParams& args,
        faiss::gpu::GpuResourcesProvider* res,
        std::vector<float>& cpuDistance,
        std::vector<faiss::idx_t>& cpuIndices,
        std::vector<float>& gpuDistance,
        std::vector<faiss::idx_t>& gpuIndices,
        int numQuery,
        int k,
        bool colMajorVecs,
        bool colMajorQueries,
        faiss::MetricType metric,
        TestThresholds thresh = TestThresholds::Normal) {
    using namespace faiss::gpu;

    bfKnn(res, args);

    std::stringstream str;
    str << "using cuVS " << args.use_cuvs << " metric " << metric
        << " colMajorVecs " << colMajorVecs << " colMajorQueries "
        << colMajorQueries;

    float maxRelativeError;
    float pctMaxDiff1;
    float pctMaxDiffN;

    switch (thresh) {
        case TestThresholds::Normal:
            maxRelativeError = 6e-3f;
            pctMaxDiff1 = 0.1f;
            pctMaxDiffN = 0.015f;
            break;
        case TestThresholds::BF16:
            maxRelativeError = 1.5e-2f;
            pctMaxDiff1 = 0.3f;
            pctMaxDiffN = 0.1f;
            break;
        case TestThresholds::BF16_Linf:
            maxRelativeError = 1.5e-2f;
            pctMaxDiff1 = 0.53f;
            pctMaxDiffN = 0.2f;
            break;
    }

    compareLists(
            cpuDistance.data(),
            cpuIndices.data(),
            gpuDistance.data(),
            gpuIndices.data(),
            numQuery,
            k,
            str.str(),
            false,
            false,
            true,
            maxRelativeError,
            pctMaxDiff1,
            pctMaxDiffN);
}

void testTransposition(
        bool colMajorVecs,
        bool colMajorQueries,
        faiss::MetricType metric,
        bool use_cuvs = false,
        float metricArg = 0) {
    using namespace faiss::gpu;

    int device = randVal(0, getNumDevices() - 1);

    StandardGpuResources res;
    res.noTempMemory();

    // The transpose and distance code assumes the desired device is already set
    DeviceScope scope(device);
    auto stream = res.getDefaultStream(device);

    int dim = randVal(20, 150);
    int numVecs = randVal(10, 30000);
    int numQuery = randVal(1, 1024);
    int k = std::min(numVecs, randVal(20, 70));

    // Input data for CPU
    std::vector<float> vecs = randVecs(numVecs, dim);
    std::vector<float> queries = randVecs(numQuery, dim);

    if ((metric == faiss::MetricType::METRIC_JensenShannon) ||
        (metric == faiss::MetricType::METRIC_Jaccard)) {
        // make values positive
        for (auto& v : vecs) {
            v = std::abs(v);
            if (v == 0) {
                v = 1e-6;
            }
        }

        for (auto& q : queries) {
            q = std::abs(q);
            if (q == 0) {
                q = 1e-6;
            }
        }
    }

    // The CPU index is our reference for the results
    faiss::IndexFlat cpuIndex(dim, metric);
    cpuIndex.metric_arg = metricArg;
    cpuIndex.add(numVecs, vecs.data());

    std::vector<float> cpuDistance(numQuery * k, 0);
    std::vector<faiss::idx_t> cpuIndices(numQuery * k, -1);

    cpuIndex.search(
            numQuery, queries.data(), k, cpuDistance.data(), cpuIndices.data());

    // Copy input data to GPU, and pre-transpose both vectors and queries for
    // passing
    auto gpuVecs = toDeviceNonTemporary<float, 2>(
            res.getResources().get(),
            device,
            vecs.data(),
            stream,
            {numVecs, dim});
    auto gpuQueries = toDeviceNonTemporary<float, 2>(
            res.getResources().get(),
            device,
            queries.data(),
            stream,
            {numQuery, dim});

    DeviceTensor<float, 2, true> vecsT(
            res.getResources().get(),
            makeDevAlloc(AllocType::Other, stream),
            {dim, numVecs});
    runTransposeAny(gpuVecs, 0, 1, vecsT, stream);

    DeviceTensor<float, 2, true> queriesT(
            res.getResources().get(),
            makeDevAlloc(AllocType::Other, stream),
            {dim, numQuery});
    runTransposeAny(gpuQueries, 0, 1, queriesT, stream);

    std::vector<float> gpuDistance(numQuery * k, 0);
    std::vector<faiss::idx_t> gpuIndices(numQuery * k, -1);

    GpuDistanceParams args;
    args.metric = metric;
    args.metricArg = metricArg;
    args.k = k;
    args.dims = dim;
    args.vectors = colMajorVecs ? vecsT.data() : gpuVecs.data();
    args.vectorsRowMajor = !colMajorVecs;
    args.numVectors = numVecs;
    args.queries = colMajorQueries ? queriesT.data() : gpuQueries.data();
    args.queriesRowMajor = !colMajorQueries;
    args.numQueries = numQuery;
    args.outDistances = gpuDistance.data();
    args.outIndices = gpuIndices.data();
    args.device = device;

#if defined USE_NVIDIA_CUVS
    args.use_cuvs = use_cuvs;
#else
    FAISS_THROW_IF_NOT_MSG(
            !use_cuvs,
            "cuVS has not been compiled into the current version so it cannot be used.");
#endif

    evaluate_bfknn(
            args,
            &res,
            cpuDistance,
            cpuIndices,
            gpuDistance,
            gpuIndices,
            numQuery,
            k,
            colMajorVecs,
            colMajorQueries,
            metric);
}

void testTransposition_bf16(
        bool colMajorVecs,
        bool colMajorQueries,
        faiss::MetricType metric,
        bool use_raft = false,
        float metricArg = 0) {
    using namespace faiss::gpu;

#ifdef USE_AMD_ROCM
    std::cout << "skipping bfloat16 test (no bfloat16 support on AMD)\n";
    EXPECT_TRUE(true);
    return;
#else
    int device = randVal(0, getNumDevices() - 1);

    StandardGpuResources res;
    if (!res.supportsBFloat16(device)) {
        std::cout << "skipping bfloat16 test (no bfloat16 support on device)\n";
        return;
    }

    res.noTempMemory();
    // The transpose and distance code assumes the desired device is already set
    DeviceScope scope(device);
    auto stream = res.getDefaultStream(device);

    int dim = randVal(20, 150);
    int numVecs = randVal(10, 30000);
    int numQuery = randVal(1, 1024);
    int k = std::min(numVecs, randVal(20, 70));

    // Input data for CPU
    std::vector<float> vecs = randVecs(numVecs, dim);
    std::vector<float> queries = randVecs(numQuery, dim);

    if ((metric == faiss::MetricType::METRIC_JensenShannon) ||
        (metric == faiss::MetricType::METRIC_Jaccard)) {
        // make values positive
        for (auto& v : vecs) {
            v = std::abs(v);
            if (v == 0) {
                v = 1e-6;
            }
        }

        for (auto& q : queries) {
            q = std::abs(q);
            if (q == 0) {
                q = 1e-6;
            }
        }
    }

    // The CPU index is our reference for the results
    faiss::IndexFlat cpuIndex(dim, metric);
    cpuIndex.metric_arg = metricArg;
    cpuIndex.add(numVecs, vecs.data());

    std::vector<float> cpuDistance(numQuery * k, 0);
    std::vector<faiss::idx_t> cpuIndices(numQuery * k, -1);

    cpuIndex.search(
            numQuery, queries.data(), k, cpuDistance.data(), cpuIndices.data());

    // Convert float32 data to bfloat16 via truncation not rounding
    // (just copy high 2 bytes)
    std::vector<uint16_t> bf16_vecs(vecs.size());
    std::vector<uint16_t> bf16_queries(queries.size());

    auto fn_f32_bf16 = [](float v) {
        uint32_t vi;
        std::memcpy(&vi, &v, sizeof(uint32_t));
        return uint16_t(vi >> 16);
    };

    std::transform(vecs.begin(), vecs.end(), bf16_vecs.begin(), fn_f32_bf16);
    std::transform(
            queries.begin(), queries.end(), bf16_queries.begin(), fn_f32_bf16);

    // Copy input data to GPU, and pre-transpose both vectors and queries for
    // passing. Just use uint16_t in lieu of __hip_bfloat16
    auto gpuVecs = toDeviceNonTemporary<uint16_t, 2>(
            res.getResources().get(),
            device,
            bf16_vecs.data(),
            stream,
            {numVecs, dim});
    auto gpuQueries = toDeviceNonTemporary<uint16_t, 2>(
            res.getResources().get(),
            device,
            bf16_queries.data(),
            stream,
            {numQuery, dim});

    DeviceTensor<uint16_t, 2, true> vecsT(
            res.getResources().get(),
            makeDevAlloc(AllocType::Other, stream),
            {dim, numVecs});
    runTransposeAny(gpuVecs, 0, 1, vecsT, stream);

    DeviceTensor<uint16_t, 2, true> queriesT(
            res.getResources().get(),
            makeDevAlloc(AllocType::Other, stream),
            {dim, numQuery});
    runTransposeAny(gpuQueries, 0, 1, queriesT, stream);

    std::vector<float> gpuDistance(numQuery * k, 0);
    std::vector<faiss::idx_t> gpuIndices(numQuery * k, -1);

    GpuDistanceParams args;
    args.metric = metric;
    args.metricArg = metricArg;
    args.k = k;
    args.dims = dim;
    args.vectors = colMajorVecs ? vecsT.data() : gpuVecs.data();
    args.vectorType = DistanceDataType::BF16;
    args.vectorsRowMajor = !colMajorVecs;
    args.numVectors = numVecs;
    args.queries = colMajorQueries ? queriesT.data() : gpuQueries.data();
    args.queryType = DistanceDataType::BF16;
    args.queriesRowMajor = !colMajorQueries;
    args.numQueries = numQuery;
    args.outDistances = gpuDistance.data();
    args.outIndices = gpuIndices.data();
    args.device = device;

    evaluate_bfknn(
            args,
            &res,
            cpuDistance,
            cpuIndices,
            gpuDistance,
            gpuIndices,
            numQuery,
            k,
            colMajorVecs,
            colMajorQueries,
            metric,
            metric == faiss::MetricType::METRIC_Linf ? TestThresholds::BF16_Linf
                                                     : TestThresholds::BF16);
#endif
}

// Test different memory layouts for brute-force k-NN
TEST(TestGpuDistance, Transposition_RR) {
    testTransposition(false, false, faiss::MetricType::METRIC_L2);
    testTransposition(false, false, faiss::MetricType::METRIC_INNER_PRODUCT);
}

TEST(TestGpuDistance, Transposition_RR_BF16) {
    testTransposition_bf16(false, false, faiss::MetricType::METRIC_L2);
    testTransposition_bf16(
            false, false, faiss::MetricType::METRIC_INNER_PRODUCT);
}

#if defined USE_NVIDIA_CUVS
TEST(TestCuvsGpuDistance, Transposition_RR) {
    testTransposition(false, false, faiss::MetricType::METRIC_L2, true);
    testTransposition(
            false, false, faiss::MetricType::METRIC_INNER_PRODUCT, true);
}
#endif

TEST(TestGpuDistance, Transposition_RC) {
    testTransposition(false, true, faiss::MetricType::METRIC_L2);
}

TEST(TestGpuDistance, Transposition_RC_BF16) {
    testTransposition_bf16(false, true, faiss::MetricType::METRIC_L2);
}

#if defined USE_NVIDIA_CUVS
TEST(TestCuvsGpuDistance, Transposition_RC) {
    testTransposition(false, true, faiss::MetricType::METRIC_L2, true);
}
#endif

TEST(TestGpuDistance, Transposition_CR) {
    testTransposition(true, false, faiss::MetricType::METRIC_L2);
}

TEST(TestGpuDistance, Transposition_CR_BF16) {
    testTransposition_bf16(true, false, faiss::MetricType::METRIC_L2);
}

#if defined USE_NVIDIA_CUVS
TEST(TestCuvsGpuDistance, Transposition_CR) {
    testTransposition(true, false, faiss::MetricType::METRIC_L2, true);
}
#endif

TEST(TestGpuDistance, Transposition_CC) {
    testTransposition(true, true, faiss::MetricType::METRIC_L2);
}

TEST(TestGpuDistance, Transposition_CC_BF16) {
    testTransposition_bf16(true, true, faiss::MetricType::METRIC_L2);
}

#if defined USE_NVIDIA_CUVS
TEST(TestCuvsGpuDistance, Transposition_CC) {
    testTransposition(true, true, faiss::MetricType::METRIC_L2, true);
}
#endif

TEST(TestGpuDistance, L1) {
    testTransposition(false, false, faiss::MetricType::METRIC_L1);
}

TEST(TestGpuDistance, L1_BF16) {
    testTransposition_bf16(false, false, faiss::MetricType::METRIC_L1);
}

#if defined USE_NVIDIA_CUVS
TEST(TestCuvsGpuDistance, L1) {
    testTransposition(false, false, faiss::MetricType::METRIC_L1, true);
}
#endif

// Test other transpositions with the general distance kernel
TEST(TestGpuDistance, L1_RC) {
    testTransposition(false, true, faiss::MetricType::METRIC_L1);
}

#if defined USE_NVIDIA_CUVS
// Test other transpositions with the general distance kernel
TEST(TestCuvsGpuDistance, L1_RC) {
    testTransposition(false, true, faiss::MetricType::METRIC_L1, true);
}
#endif

TEST(TestGpuDistance, L1_RC_BF16) {
    testTransposition_bf16(false, true, faiss::MetricType::METRIC_L1);
}

TEST(TestGpuDistance, L1_CR) {
    testTransposition(true, false, faiss::MetricType::METRIC_L1);
}

TEST(TestGpuDistance, L1_CR_BF16) {
    testTransposition_bf16(true, false, faiss::MetricType::METRIC_L1);
}

#if defined USE_NVIDIA_CUVS
TEST(TestCuvsGpuDistance, L1_CR) {
    testTransposition(true, false, faiss::MetricType::METRIC_L1, true);
}
#endif

TEST(TestGpuDistance, L1_CC) {
    testTransposition(true, true, faiss::MetricType::METRIC_L1);
}

TEST(TestGpuDistance, L1_CC_BF16) {
    testTransposition_bf16(true, true, faiss::MetricType::METRIC_L1);
}

#if defined USE_NVIDIA_CUVS
TEST(TestCuvsGpuDistance, L1_CC) {
    testTransposition(true, true, faiss::MetricType::METRIC_L1, true);
}
#endif

// Test remainder of metric types
TEST(TestGpuDistance, Linf) {
    testTransposition(false, false, faiss::MetricType::METRIC_Linf);
}

#if defined USE_NVIDIA_CUVS
// Test remainder of metric types
TEST(TestCuvsGpuDistance, Linf) {
    testTransposition(false, false, faiss::MetricType::METRIC_Linf, true);
}
#endif

TEST(TestGpuDistance, Linf_BF16) {
    testTransposition_bf16(false, false, faiss::MetricType::METRIC_Linf);
}

TEST(TestGpuDistance, Lp) {
    testTransposition(false, false, faiss::MetricType::METRIC_Lp, false, 3);
}

TEST(TestGpuDistance, Lp_BF16) {
    testTransposition_bf16(
            false, false, faiss::MetricType::METRIC_Lp, false, 3);
}

#if defined USE_NVIDIA_CUVS
TEST(TestCuvsGpuDistance, Lp) {
    testTransposition(false, false, faiss::MetricType::METRIC_Lp, true, 3);
}
#endif

TEST(TestGpuDistance, Canberra) {
    testTransposition(false, false, faiss::MetricType::METRIC_Canberra);
}

TEST(TestGpuDistance, Canberra_BF16) {
    testTransposition_bf16(false, false, faiss::MetricType::METRIC_Canberra);
}

#if defined USE_NVIDIA_CUVS
TEST(TestCuvsGpuDistance, Canberra) {
    testTransposition(false, false, faiss::MetricType::METRIC_Canberra, true);
}
#endif

TEST(TestGpuDistance, BrayCurtis) {
    testTransposition(false, false, faiss::MetricType::METRIC_BrayCurtis);
}

TEST(TestGpuDistance, BrayCurtis_BF16) {
    testTransposition_bf16(false, false, faiss::MetricType::METRIC_BrayCurtis);
}

TEST(TestGpuDistance, JensenShannon) {
    testTransposition(false, false, faiss::MetricType::METRIC_JensenShannon);
}

TEST(TestGpuDistance, JensenShannon_BF16) {
    testTransposition_bf16(
            false, false, faiss::MetricType::METRIC_JensenShannon);
}

#if defined USE_NVIDIA_CUVS
TEST(TestCuvsGpuDistance, JensenShannon) {
    testTransposition(
            false, false, faiss::MetricType::METRIC_JensenShannon, true);
}
#endif

TEST(TestGpuDistance, Jaccard) {
    testTransposition(false, false, faiss::MetricType::METRIC_Jaccard);
}

TEST(TestGpuDistance, Jaccard_BF16) {
    testTransposition_bf16(false, false, faiss::MetricType::METRIC_Jaccard);
}

int main(int argc, char** argv) {
    testing::InitGoogleTest(&argc, argv);

    // just run with a fixed test seed
    faiss::gpu::setTestSeed(100);

    return RUN_ALL_TESTS();
}
