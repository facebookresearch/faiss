#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <faiss/gpu/StandardGpuResources.h>
#include <faiss/gpu/test/TestUtils.h>
#include <faiss/gpu/utils/DeviceUtils.h>
#include <gflags/gflags.h>
#include <gtest/gtest.h>
#include <algorithm>
#include <faiss/gpu/utils/BlockSelectKernel.cuh>
#include <faiss/gpu/utils/DeviceDefs.cuh>
#include <faiss/gpu/utils/DeviceTensor.cuh>
#include <faiss/gpu/utils/HostTensor.cuh>
#include <faiss/gpu/utils/WarpSelectKernel.cuh>
#include <sstream>
#include <unordered_map>
#include <vector>

DEFINE_int32(rows, 10000, "rows in matrix");
DEFINE_int32(cols, 40000, "cols in matrix");
DEFINE_int32(k, 100, "k");
DEFINE_bool(dir, false, "direction of sort");
DEFINE_bool(warp, false, "warp select");
DEFINE_int32(iter, 5, "iterations to run");
DEFINE_bool(k_powers, false, "test k powers of 2 from 1 -> max k");

int main(int argc, char** argv) {
    using namespace faiss::gpu;

    gflags::ParseCommandLineFlags(&argc, &argv, true);

    std::vector<float> v = randVecs(FLAGS_rows, FLAGS_cols);
    HostTensor<float, 2, true> hostVal({FLAGS_rows, FLAGS_cols});

    for (int r = 0; r < FLAGS_rows; ++r) {
        for (int c = 0; c < FLAGS_cols; ++c) {
            hostVal[r][c] = v[r * FLAGS_cols + c];
        }
    }

    StandardGpuResources res;
    res.noTempMemory();

    auto resUse = res.getResources();

    // Select top-k on GPU
    DeviceTensor<float, 2, true> gpuVal(
            resUse.get(), makeDevAlloc(AllocType::Other, 0), hostVal);

    int startK = FLAGS_k;
    int limitK = FLAGS_k;

    if (FLAGS_k_powers) {
        startK = 1;
        limitK = GPU_MAX_SELECTION_K;
    }

    for (int k = startK; k <= limitK; k *= 2) {
        DeviceTensor<float, 2, true> gpuOutVal(
                resUse.get(),
                makeDevAlloc(AllocType::Other, 0),
                {FLAGS_rows, k});
        DeviceTensor<faiss::idx_t, 2, true> gpuOutInd(
                resUse.get(),
                makeDevAlloc(AllocType::Other, 0),
                {FLAGS_rows, k});

        for (int i = 0; i < FLAGS_iter; ++i) {
            if (FLAGS_warp) {
                runWarpSelect(gpuVal, gpuOutVal, gpuOutInd, FLAGS_dir, k, 0);
            } else {
                runBlockSelect(gpuVal, gpuOutVal, gpuOutInd, FLAGS_dir, k, 0);
            }
        }
    }

    hipDeviceSynchronize();
}
