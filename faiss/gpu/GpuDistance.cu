/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <faiss/gpu/GpuDistance.h>
#include <faiss/gpu/GpuResources.h>
#include <faiss/gpu/utils/DeviceUtils.h>
#include <faiss/impl/FaissAssert.h>
#include <faiss/gpu/impl/Distance.cuh>
#include <faiss/gpu/utils/ConversionOperators.cuh>
#include <faiss/gpu/utils/CopyUtils.cuh>
#include <faiss/gpu/utils/DeviceTensor.cuh>

namespace faiss {
namespace gpu {

template <typename T>
void bfKnnConvert(GpuResourcesProvider* prov, const GpuDistanceParams& args) {
    // Validate the input data
    FAISS_THROW_IF_NOT_MSG(
            args.k > 0 || args.k == -1,
            "bfKnn: k must be > 0 for top-k reduction, "
            "or -1 for all pairwise distances");
    FAISS_THROW_IF_NOT_MSG(args.dims > 0, "bfKnn: dims must be > 0");
    FAISS_THROW_IF_NOT_MSG(
            args.numVectors > 0, "bfKnn: numVectors must be > 0");
    FAISS_THROW_IF_NOT_MSG(
            args.vectors, "bfKnn: vectors must be provided (passed null)");
    FAISS_THROW_IF_NOT_MSG(
            args.numQueries > 0, "bfKnn: numQueries must be > 0");
    FAISS_THROW_IF_NOT_MSG(
            args.queries, "bfKnn: queries must be provided (passed null)");
    FAISS_THROW_IF_NOT_MSG(
            args.outDistances,
            "bfKnn: outDistances must be provided (passed null)");
    FAISS_THROW_IF_NOT_MSG(
            args.outIndices || args.k == -1,
            "bfKnn: outIndices must be provided (passed null)");

    // If the user specified a device, then ensure that it is currently set
    int device = -1;
    if (args.device == -1) {
        // Original behavior if no device is specified, use the current CUDA
        // thread local device
        device = getCurrentDevice();
    } else {
        // Otherwise, use the device specified in `args`
        device = args.device;

        FAISS_THROW_IF_NOT_FMT(
                device >= 0 && device < getNumDevices(),
                "bfKnn: device specified must be -1 (current CUDA thread local device) "
                "or within the range [0, %d)",
                getNumDevices());
    }

    DeviceScope scope(device);

    // Don't let the resources go out of scope
    auto resImpl = prov->getResources();
    auto res = resImpl.get();
    auto stream = res->getDefaultStreamCurrentDevice();

    auto tVectors = toDeviceTemporary<T, 2>(
            res,
            device,
            const_cast<T*>(reinterpret_cast<const T*>(args.vectors)),
            stream,
            {args.vectorsRowMajor ? args.numVectors : args.dims,
             args.vectorsRowMajor ? args.dims : args.numVectors});
    auto tQueries = toDeviceTemporary<T, 2>(
            res,
            device,
            const_cast<T*>(reinterpret_cast<const T*>(args.queries)),
            stream,
            {args.queriesRowMajor ? args.numQueries : args.dims,
             args.queriesRowMajor ? args.dims : args.numQueries});

    DeviceTensor<float, 1, true> tVectorNorms;
    if (args.vectorNorms) {
        tVectorNorms = toDeviceTemporary<float, 1>(
                res,
                device,
                const_cast<float*>(args.vectorNorms),
                stream,
                {args.numVectors});
    }

    auto tOutDistances = toDeviceTemporary<float, 2>(
            res,
            device,
            args.outDistances,
            stream,
            {args.numQueries, args.k == -1 ? args.numVectors : args.k});

    if (args.k == -1) {
        // Reporting all pairwise distances
        allPairwiseDistanceOnDevice<T>(
                res,
                device,
                stream,
                tVectors,
                args.vectorsRowMajor,
                args.vectorNorms ? &tVectorNorms : nullptr,
                tQueries,
                args.queriesRowMajor,
                args.metric,
                args.metricArg,
                tOutDistances);
    } else if (args.outIndicesType == IndicesDataType::I64) {
        auto tOutIndices = toDeviceTemporary<idx_t, 2>(
                res,
                device,
                (idx_t*)args.outIndices,
                stream,
                {args.numQueries, args.k});

        // Since we've guaranteed that all arguments are on device, call the
        // implementation
        bfKnnOnDevice<T>(
                res,
                device,
                stream,
                tVectors,
                args.vectorsRowMajor,
                args.vectorNorms ? &tVectorNorms : nullptr,
                tQueries,
                args.queriesRowMajor,
                args.k,
                args.metric,
                args.metricArg,
                tOutDistances,
                tOutIndices,
                args.ignoreOutDistances);

        fromDevice<idx_t, 2>(tOutIndices, (idx_t*)args.outIndices, stream);

    } else if (args.outIndicesType == IndicesDataType::I32) {
        // The brute-force API supports i64 indices, but our output buffer is
        // i32 so we need to temporarily allocate and then convert back to i32
        // FIXME: convert to int32_t everywhere?
        static_assert(sizeof(int) == 4, "");
        DeviceTensor<idx_t, 2, true> tIntIndices(
                res,
                makeTempAlloc(AllocType::Other, stream),
                {args.numQueries, args.k});

        // Since we've guaranteed that all arguments are on device, call the
        // implementation
        bfKnnOnDevice<T>(
                res,
                device,
                stream,
                tVectors,
                args.vectorsRowMajor,
                args.vectorNorms ? &tVectorNorms : nullptr,
                tQueries,
                args.queriesRowMajor,
                args.k,
                args.metric,
                args.metricArg,
                tOutDistances,
                tIntIndices,
                args.ignoreOutDistances);

        // Convert and copy int indices out
        auto tOutIntIndices = toDeviceTemporary<int, 2>(
                res,
                device,
                (int*)args.outIndices,
                stream,
                {args.numQueries, args.k});

        convertTensor<idx_t, int, 2>(stream, tIntIndices, tOutIntIndices);

        // Copy back if necessary
        fromDevice<int, 2>(tOutIntIndices, (int*)args.outIndices, stream);
    } else {
        FAISS_THROW_MSG("unknown outIndicesType");
    }

    // Copy distances back if necessary
    fromDevice<float, 2>(tOutDistances, args.outDistances, stream);
}

void bfKnn(GpuResourcesProvider* res, const GpuDistanceParams& args) {
    // For now, both vectors and queries must be of the same data type
    FAISS_THROW_IF_NOT_MSG(
            args.vectorType == args.queryType,
            "limitation: both vectorType and queryType must currently "
            "be the same (F32 or F16");

    if (args.vectorType == DistanceDataType::F32) {
        bfKnnConvert<float>(res, args);
    } else if (args.vectorType == DistanceDataType::F16) {
        bfKnnConvert<half>(res, args);
    } else {
        FAISS_THROW_MSG("unknown vectorType");
    }
}

// legacy version
void bruteForceKnn(
        GpuResourcesProvider* res,
        faiss::MetricType metric,
        // A region of memory size numVectors x dims, with dims
        // innermost
        const float* vectors,
        bool vectorsRowMajor,
        idx_t numVectors,
        // A region of memory size numQueries x dims, with dims
        // innermost
        const float* queries,
        bool queriesRowMajor,
        idx_t numQueries,
        int dims,
        int k,
        // A region of memory size numQueries x k, with k
        // innermost
        float* outDistances,
        // A region of memory size numQueries x k, with k
        // innermost
        idx_t* outIndices) {
    std::cerr << "bruteForceKnn is deprecated; call bfKnn instead" << std::endl;

    GpuDistanceParams args;
    args.metric = metric;
    args.k = k;
    args.dims = dims;
    args.vectors = vectors;
    args.vectorsRowMajor = vectorsRowMajor;
    args.numVectors = numVectors;
    args.queries = queries;
    args.queriesRowMajor = queriesRowMajor;
    args.numQueries = numQueries;
    args.outDistances = outDistances;
    args.outIndices = outIndices;

    bfKnn(res, args);
}

} // namespace gpu
} // namespace faiss
