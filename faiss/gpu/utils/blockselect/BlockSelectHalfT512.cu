/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <faiss/gpu/utils/blockselect/BlockSelectImpl.cuh>

namespace faiss { namespace gpu {

BLOCK_SELECT_IMPL(half, true, 512, 8);

} } // namespace
