#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <faiss/gpu/GpuIndex.h>
#include <faiss/gpu/utils/RaftUtils.h>
#include <raft/core/device_mdarray.hpp>
#include <raft/core/device_mdspan.hpp>
#include <raft/linalg/coalesced_reduction.cuh>
#include <raft/linalg/map.cuh>
#include <raft/matrix/gather.cuh>

#include <thrust/copy.h>
#include <thrust/gather.h>
#include <thrust/reduce.h>

namespace faiss {
namespace gpu {

void validRowIndices(
        GpuResources* res,
        Tensor<float, 2, true>& vecs,
        bool* validRows) {
    idx_t n_rows = vecs.getSize(0);
    idx_t dim = vecs.getSize(1);

    raft::linalg::coalescedReduction(
            validRows,
            vecs.data(),
            dim,
            n_rows,
            true,
            res->getDefaultStreamCurrentDevice(),
            false,
            [] __device__(float v, idx_t i) { return isfinite(v); },
            raft::mul_op());
}

idx_t inplaceGatherFilteredRows(
        GpuResources* res,
        Tensor<float, 2, true>& vecs,
        Tensor<idx_t, 1, true>& indices) {
    raft::device_resources& raft_handle = res->getRaftHandleCurrentDevice();
    idx_t n_rows = vecs.getSize(0);
    idx_t dim = vecs.getSize(1);

    auto valid_rows =
            raft::make_device_vector<bool, idx_t>(raft_handle, n_rows);

    validRowIndices(res, vecs, valid_rows.data_handle());

    idx_t n_rows_valid = thrust::reduce(
            raft_handle.get_thrust_policy(),
            valid_rows.data_handle(),
            valid_rows.data_handle() + n_rows,
            0);

    if (n_rows_valid < n_rows) {
        auto gather_indices = raft::make_device_vector<idx_t, idx_t>(
                raft_handle, n_rows_valid);

        auto count = thrust::make_counting_iterator(0);

        thrust::copy_if(
                raft_handle.get_thrust_policy(),
                count,
                count + n_rows,
                gather_indices.data_handle(),
                [valid_rows = valid_rows.data_handle()] __device__(auto i) {
                    return valid_rows[i];
                });

        raft::matrix::gather(
                raft_handle,
                raft::make_device_matrix_view<float, idx_t>(
                        vecs.data(), n_rows, dim),
                raft::make_const_mdspan(gather_indices.view()),
                (idx_t)16);

        auto validIndices = raft::make_device_vector<idx_t, idx_t>(
                raft_handle, n_rows_valid);

        thrust::gather(
                raft_handle.get_thrust_policy(),
                gather_indices.data_handle(),
                gather_indices.data_handle() + gather_indices.size(),
                indices.data(),
                validIndices.data_handle());
        thrust::copy(
                raft_handle.get_thrust_policy(),
                validIndices.data_handle(),
                validIndices.data_handle() + n_rows_valid,
                indices.data());
    }
    return n_rows_valid;
}

} // namespace gpu
} // namespace faiss
