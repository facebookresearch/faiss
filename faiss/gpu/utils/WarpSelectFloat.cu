/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <faiss/gpu/utils/warpselect/WarpSelectImpl.cuh>
#include <faiss/gpu/utils/DeviceDefs.cuh>

namespace faiss { namespace gpu {

// warp Q to thread Q:
// 1, 1
// 32, 2
// 64, 3
// 128, 3
// 256, 4
// 512, 8
// 1024, 8
// 2048, 8

WARP_SELECT_DECL(float, true, 1);
WARP_SELECT_DECL(float, true, 32);
WARP_SELECT_DECL(float, true, 64);
WARP_SELECT_DECL(float, true, 128);
WARP_SELECT_DECL(float, true, 256);
WARP_SELECT_DECL(float, true, 512);
WARP_SELECT_DECL(float, true, 1024);
#if GPU_MAX_SELECTION_K >= 2048
WARP_SELECT_DECL(float, true, 2048);
#endif

WARP_SELECT_DECL(float, false, 1);
WARP_SELECT_DECL(float, false, 32);
WARP_SELECT_DECL(float, false, 64);
WARP_SELECT_DECL(float, false, 128);
WARP_SELECT_DECL(float, false, 256);
WARP_SELECT_DECL(float, false, 512);
WARP_SELECT_DECL(float, false, 1024);
#if GPU_MAX_SELECTION_K >= 2048
WARP_SELECT_DECL(float, false, 2048);
#endif

void runWarpSelect(Tensor<float, 2, true>& in,
                      Tensor<float, 2, true>& outK,
                      Tensor<int, 2, true>& outV,
                      bool dir, int k, hipStream_t stream) {
  FAISS_ASSERT(k <= 2048);

  if (dir) {
    if (k == 1) {
      WARP_SELECT_CALL(float, true, 1);
    } else if (k <= 32) {
      WARP_SELECT_CALL(float, true, 32);
    } else if (k <= 64) {
      WARP_SELECT_CALL(float, true, 64);
    } else if (k <= 128) {
      WARP_SELECT_CALL(float, true, 128);
    } else if (k <= 256) {
      WARP_SELECT_CALL(float, true, 256);
    } else if (k <= 512) {
      WARP_SELECT_CALL(float, true, 512);
    } else if (k <= 1024) {
      WARP_SELECT_CALL(float, true, 1024);
#if GPU_MAX_SELECTION_K >= 2048
    } else if (k <= 2048) {
      WARP_SELECT_CALL(float, true, 2048);
#endif
    }
  } else {
    if (k == 1) {
      WARP_SELECT_CALL(float, false, 1);
    } else if (k <= 32) {
      WARP_SELECT_CALL(float, false, 32);
    } else if (k <= 64) {
      WARP_SELECT_CALL(float, false, 64);
    } else if (k <= 128) {
      WARP_SELECT_CALL(float, false, 128);
    } else if (k <= 256) {
      WARP_SELECT_CALL(float, false, 256);
    } else if (k <= 512) {
      WARP_SELECT_CALL(float, false, 512);
    } else if (k <= 1024) {
      WARP_SELECT_CALL(float, false, 1024);
#if GPU_MAX_SELECTION_K >= 2048
    } else if (k <= 2048) {
      WARP_SELECT_CALL(float, false, 2048);
#endif
    }
  }
}

} } // namespace
