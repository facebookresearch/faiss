/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */


#include <faiss/gpu/utils/DeviceUtils.h>
#include <faiss/gpu/utils/DeviceDefs.cuh>
#include <faiss/impl/FaissAssert.h>
#include <mutex>
#include <unordered_map>
#include <hip/hip_runtime_api.h>

namespace faiss { namespace gpu {

int getCurrentDevice() {
  int dev = -1;
  CUDA_VERIFY(hipGetDevice(&dev));
  FAISS_ASSERT(dev != -1);

  return dev;
}

void setCurrentDevice(int device) {
  CUDA_VERIFY(hipSetDevice(device));
}

int getNumDevices() {
  int numDev = -1;
  hipError_t err = hipGetDeviceCount(&numDev);
  if (hipErrorNoDevice == err) {
    numDev = 0;
  } else {
    CUDA_VERIFY(err);
  }
  FAISS_ASSERT(numDev != -1);

  return numDev;
}

void profilerStart() {
  CUDA_VERIFY(hipProfilerStart());
}

void profilerStop() {
  CUDA_VERIFY(hipProfilerStop());
}

void synchronizeAllDevices() {
  for (int i = 0; i < getNumDevices(); ++i) {
    DeviceScope scope(i);

    CUDA_VERIFY(hipDeviceSynchronize());
  }
}

const hipDeviceProp_t& getDeviceProperties(int device) {
  static std::mutex mutex;
  static std::unordered_map<int, hipDeviceProp_t> properties;

  std::lock_guard<std::mutex> guard(mutex);

  auto it = properties.find(device);
  if (it == properties.end()) {
    hipDeviceProp_t prop;
    CUDA_VERIFY(hipGetDeviceProperties(&prop, device));

    properties[device] = prop;
    it = properties.find(device);
  }

  return it->second;
}

const hipDeviceProp_t& getCurrentDeviceProperties() {
  return getDeviceProperties(getCurrentDevice());
}

int getMaxThreads(int device) {
  return getDeviceProperties(device).maxThreadsPerBlock;
}

int getMaxThreadsCurrentDevice() {
  return getMaxThreads(getCurrentDevice());
}

size_t getMaxSharedMemPerBlock(int device) {
  return getDeviceProperties(device).sharedMemPerBlock;
}

size_t getMaxSharedMemPerBlockCurrentDevice() {
  return getMaxSharedMemPerBlock(getCurrentDevice());
}

int getDeviceForAddress(const void* p) {
  if (!p) {
    return -1;
  }

  hipPointerAttribute_t att;
  hipError_t err = hipPointerGetAttributes(&att, p);
  FAISS_ASSERT_FMT(err == hipSuccess ||
                   err == hipErrorInvalidValue,
                   "unknown error %d", (int) err);

  if (err == hipErrorInvalidValue) {
    // Make sure the current thread error status has been reset
    err = hipGetLastError();
    FAISS_ASSERT_FMT(err == hipErrorInvalidValue,
                     "unknown error %d", (int) err);
    return -1;
  }

  // memoryType is deprecated for CUDA 10.0+
#if CUDA_VERSION < 10000
  if (att.memoryType == hipMemoryTypeHost) {
    return -1;
  } else {
    return att.device;
  }
#else
  // FIXME: what to use for managed memory?
  if (att.type == hipMemoryTypeDevice) {
    return att.device;
  } else {
    return -1;
  }
#endif
}

bool getFullUnifiedMemSupport(int device) {
  const auto& prop = getDeviceProperties(device);
  return (prop.major >= 6);
}

bool getFullUnifiedMemSupportCurrentDevice() {
  return getFullUnifiedMemSupport(getCurrentDevice());
}

bool getTensorCoreSupport(int device) {
  const auto& prop = getDeviceProperties(device);
  return (prop.major >= 7);
}

bool getTensorCoreSupportCurrentDevice() {
  return getTensorCoreSupport(getCurrentDevice());
}

int getMaxKSelection() {
  // Don't use the device at the moment, just base this based on the CUDA SDK
  // that we were compiled with
  return GPU_MAX_SELECTION_K;
}

DeviceScope::DeviceScope(int device) {
  prevDevice_ = getCurrentDevice();

  if (prevDevice_ != device) {
    setCurrentDevice(device);
  } else {
    prevDevice_ = -1;
  }
}

DeviceScope::~DeviceScope() {
  if (prevDevice_ != -1) {
    setCurrentDevice(prevDevice_);
  }
}

CublasHandleScope::CublasHandleScope() {
  auto blasStatus = hipblasCreate(&blasHandle_);
  FAISS_ASSERT(blasStatus == HIPBLAS_STATUS_SUCCESS);
}

CublasHandleScope::~CublasHandleScope() {
  auto blasStatus = hipblasDestroy(blasHandle_);
  FAISS_ASSERT(blasStatus == HIPBLAS_STATUS_SUCCESS);
}

CudaEvent::CudaEvent(hipStream_t stream)
    : event_(0) {
  CUDA_VERIFY(hipEventCreateWithFlags(&event_, hipEventDisableTiming));
  CUDA_VERIFY(hipEventRecord(event_, stream));
}

CudaEvent::CudaEvent(CudaEvent&& event) noexcept
    : event_(std::move(event.event_)) {
  event.event_ = 0;
}

CudaEvent::~CudaEvent() {
  if (event_) {
    CUDA_VERIFY(hipEventDestroy(event_));
  }
}

CudaEvent&
CudaEvent::operator=(CudaEvent&& event) noexcept {
  event_ = std::move(event.event_);
  event.event_ = 0;

  return *this;
}

void
CudaEvent::streamWaitOnEvent(hipStream_t stream) {
  CUDA_VERIFY(hipStreamWaitEvent(stream, event_, 0));
}

void
CudaEvent::cpuWaitOnEvent() {
  CUDA_VERIFY(hipEventSynchronize(event_));
}

} } // namespace
