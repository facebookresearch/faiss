/**
 * Copyright (c) 2015-present, Facebook, Inc.
 * All rights reserved.
 *
 * This source code is licensed under the BSD+Patents license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include "WarpSelectImpl.cuh"
#include "../DeviceDefs.cuh"

namespace faiss { namespace gpu {

#if GPU_MAX_SELECTION_K >= 2048
#ifdef FAISS_USE_FLOAT16
WARP_SELECT_IMPL(half, false, 2048, 8);
#endif
#endif

} } // namespace
