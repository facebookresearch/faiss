/**
 * Copyright (c) 2015-present, Facebook, Inc.
 * All rights reserved.
 *
 * This source code is licensed under the BSD+Patents license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include "WarpSelectImpl.cuh"

namespace faiss { namespace gpu {

#ifdef FAISS_USE_FLOAT16
WARP_SELECT_IMPL(half, true, 1, 1);
WARP_SELECT_IMPL(half, false, 1, 1);
#endif

} } // namespace
