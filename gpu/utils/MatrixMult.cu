/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */


#include "MatrixMult.cuh"
#include "DeviceMemory.h"
#include "DeviceUtils.h" // CUDA_VERIFY
#include "DeviceTensor.cuh"
#include "HostTensor.cuh"

namespace faiss { namespace gpu {

template <typename T>
struct CublasGemm {
};

template <>
struct CublasGemm<float> {
  static hipblasStatus_t gemm(hipblasHandle_t handle,
                             hipblasOperation_t transa,
                             hipblasOperation_t transb,
                             int m,
                             int n,
                             int k,
                             float fAlpha,
                             const float *A,
                             int lda,
                             const float *B,
                             int ldb,
                             float fBeta,
                             float *C,
                             int ldc,
                             bool useHgemm) {
    return hipblasSgemm(handle, transa, transb, m, n, k,
                       &fAlpha, A, lda, B, ldb, &fBeta, C, ldc);
  }
};

#ifdef FAISS_USE_FLOAT16
template <>
struct CublasGemm<half> {
  static hipblasStatus_t gemm(hipblasHandle_t handle,
                             hipblasOperation_t transa,
                             hipblasOperation_t transb,
                             int m,
                             int n,
                             int k,
                             const float fAlpha,
                             const half *A,
                             int lda,
                             const half *B,
                             int ldb,
                             const float fBeta,
                             half *C,
                             int ldc,
                             bool useHgemm) {
    if (getDeviceSupportsFloat16Math(getCurrentDevice()) && useHgemm) {
      half hAlpha = hostFloat2Half(fAlpha);
      half hBeta = hostFloat2Half(fBeta);

      return hipblasHgemm(handle, transa, transb, m, n, k,
                         &hAlpha, A, lda, B, ldb, &hBeta, C, ldc);
    }

    // CUDA 8.0 changes the half datatype specifier
#if CUDA_VERSION == 7050
    auto halfType = CUBLAS_DATA_HALF;
#else
    auto halfType = HIP_R_16F;
#endif // CUDA_VERSION

    return cublasSgemmEx(handle, transa, transb, m, n, k,
                         &fAlpha, A, halfType, lda,
                         B, halfType, ldb,
                         &fBeta,
                         C, halfType, ldc);
  }
};
#endif // FAISS_USE_FLOAT16


template <typename T>
void
runMatrixMult(Tensor<T, 2, true>& c, bool transC,
              Tensor<T, 2, true>& a, bool transA,
              Tensor<T, 2, true>& b, bool transB,
              float alpha,
              float beta,
              bool useHgemm,
              hipblasHandle_t handle,
              hipStream_t stream) {
  hipblasSetStream(handle, stream);

  // Check that we have (m x k) * (k x n) = (m x n)
  // using the input row-major layout
  int aM = transA ? a.getSize(1) : a.getSize(0);
  int aK = transA ? a.getSize(0) : a.getSize(1);

  int bK = transB ? b.getSize(1) : b.getSize(0);
  int bN = transB ? b.getSize(0) : b.getSize(1);

  int cM = transC ? c.getSize(1) : c.getSize(0);
  int cN = transC ? c.getSize(0) : c.getSize(1);

  FAISS_ASSERT(aM == cM);
  FAISS_ASSERT(aK == bK);
  FAISS_ASSERT(bN == cN);

  FAISS_ASSERT(a.getStride(1) == 1);
  FAISS_ASSERT(b.getStride(1) == 1);
  FAISS_ASSERT(c.getStride(1) == 1);

  // Now, we have to represent the matrix multiplication in
  // column-major layout
  T* pA = transC ? a.data() : b.data();
  T* pB = transC ? b.data() : a.data();
  T* pC = c.data();

  int m = c.getSize(1); // stride 1 size
  int n = c.getSize(0); // other size
  int k = transA ? a.getSize(0) : a.getSize(1);

  int lda = transC ? a.getStride(0) : b.getStride(0);
  int ldb = transC ? b.getStride(0) : a.getStride(0);
  int ldc = c.getStride(0);

  auto gemmTrA = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  auto gemmTrB = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  if (transC) {
    gemmTrA = transA ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    gemmTrB = transB ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  }

  auto err = CublasGemm<T>::gemm(handle,
                                 gemmTrA, gemmTrB,
                                 m, n, k, alpha,
                                 pA, lda, pB, ldb, beta,
                                 pC, ldc, useHgemm);

  FAISS_ASSERT_FMT(err == HIPBLAS_STATUS_SUCCESS,
                   "cublas failed (%d): %s "
                   "(%d, %d)%s x (%d, %d)%s = (%d, %d)%s",
                   (int) err,
                   useHgemm ? "Hgemm" : "Sgemm",
                   a.getSize(0), a.getSize(1), transA ? "'" : "",
                   b.getSize(0), b.getSize(1), transB ? "'" : "",
                   c.getSize(0), c.getSize(1), transC ? "'" : "");
  CUDA_TEST_ERROR();
}

void runMatrixMult(Tensor<float, 2, true>& c, bool transC,
                   Tensor<float, 2, true>& a, bool transA,
                   Tensor<float, 2, true>& b, bool transB,
                   float alpha,
                   float beta,
                   bool useHgemm,
                   hipblasHandle_t handle,
                   hipStream_t stream) {
  return runMatrixMult<float>(c, transC, a, transA, b, transB,
                              alpha, beta, useHgemm, handle, stream);
}

#ifdef FAISS_USE_FLOAT16
void runMatrixMult(Tensor<half, 2, true>& c, bool transC,
                   Tensor<half, 2, true>& a, bool transA,
                   Tensor<half, 2, true>& b, bool transB,
                   float alpha,
                   float beta,
                   bool useHgemm,
                   hipblasHandle_t handle,
                   hipStream_t stream) {
  return runMatrixMult<half>(c, transC, a, transA, b, transB,
                             alpha, beta, useHgemm, handle, stream);
}
#endif

void
runIteratedMatrixMult(Tensor<float, 3, true>& c, bool transC,
                      Tensor<float, 3, true>& a, bool transA,
                      Tensor<float, 3, true>& b, bool transB,
                      float alpha,
                      float beta,
                      hipblasHandle_t handle,
                      hipStream_t stream) {
  FAISS_ASSERT(c.getSize(0) == a.getSize(0));
  FAISS_ASSERT(a.getSize(0) == b.getSize(0));

  for (int i = 0; i < a.getSize(0); ++i) {
    auto cView = c[i].view();
    auto aView = a[i].view();
    auto bView = b[i].view();

    runMatrixMult(cView, transC,
                  aView, transA,
                  bView, transB,
                  alpha, beta, false, handle, stream);
  }
}

void
runBatchMatrixMult(Tensor<float, 3, true>& c, bool transC,
                   Tensor<float, 3, true>& a, bool transA,
                   Tensor<float, 3, true>& b, bool transB,
                   float alpha,
                   float beta,
                   DeviceMemory& mem,
                   hipblasHandle_t handle,
                   hipStream_t stream) {
  FAISS_ASSERT(c.getSize(0) == a.getSize(0));
  FAISS_ASSERT(a.getSize(0) == b.getSize(0));
  hipblasSetStream(handle, stream);

  // Check that we have (m x k) * (k x n) = (m x n)
  // using the input row-major layout
  int aM = transA ? a.getSize(2) : a.getSize(1);
  int aK = transA ? a.getSize(1) : a.getSize(2);

  int bK = transB ? b.getSize(2) : b.getSize(1);
  int bN = transB ? b.getSize(1) : b.getSize(2);

  int cM = transC ? c.getSize(2) : c.getSize(1);
  int cN = transC ? c.getSize(1) : c.getSize(2);

  FAISS_ASSERT(aM == cM);
  FAISS_ASSERT(aK == bK);
  FAISS_ASSERT(bN == cN);

  // Now, we have to represent the matrix multiplication in
  // column-major layout
  float* pA = transC ? a.data() : b.data();
  float* pB = transC ? b.data() : a.data();
  float* pC = c.data();

  int m = c.getSize(2); // stride 1 size
  int n = c.getSize(1); // other size
  int k = transA ? a.getSize(1) : a.getSize(2);

  int lda = transC ? a.getStride(1) : b.getStride(1);
  int ldb = transC ? b.getStride(1) : a.getStride(1);
  int ldc = c.getStride(1);

  auto gemmTrA = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  auto gemmTrB = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  if (transC) {
    gemmTrA = transA ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    gemmTrB = transB ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  }

  HostTensor<float*, 1, true> hostA({a.getSize(0)});
  HostTensor<float*, 1, true> hostB({b.getSize(0)});
  HostTensor<float*, 1, true> hostC({c.getSize(0)});

  size_t aOffset = a.getStride(0);
  size_t bOffset = b.getStride(0);
  size_t cOffset = c.getStride(0);

  for (int i = 0; i < a.getSize(0); ++i) {
    hostA[i] = transC ? a.data() + i * aOffset : b.data() + i * bOffset;
    hostB[i] = transC ? b.data() + i * bOffset : a.data() + i * aOffset;
    hostC[i] = c.data() + i * cOffset;
  }

  DeviceTensor<float*, 1, true> deviceA(mem, hostA, stream);
  DeviceTensor<float*, 1, true> deviceB(mem, hostB, stream);
  DeviceTensor<float*, 1, true> deviceC(mem, hostC, stream);

  auto err =
    hipblasSgemmBatched(handle,
                       gemmTrA, gemmTrB,
                       m, n, k, &alpha,
                       (const float**) deviceA.data(), lda,
                       (const float**) deviceB.data(), ldb, &beta,
                       deviceC.data(), ldc, a.getSize(0));
  FAISS_ASSERT_FMT(err == HIPBLAS_STATUS_SUCCESS,
                   "hipblasSgemmBatched failed (%d)", (int) err);
  CUDA_TEST_ERROR();
}

} } // namespace
