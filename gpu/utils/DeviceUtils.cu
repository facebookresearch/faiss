/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */


#include "DeviceUtils.h"
#include "DeviceDefs.cuh"
#include "../../FaissAssert.h"
#include <mutex>
#include <unordered_map>

namespace faiss { namespace gpu {

int getCurrentDevice() {
  int dev = -1;
  CUDA_VERIFY(hipGetDevice(&dev));
  FAISS_ASSERT(dev != -1);

  return dev;
}

void setCurrentDevice(int device) {
  CUDA_VERIFY(hipSetDevice(device));
}

int getNumDevices() {
  int numDev = -1;
  hipError_t err = hipGetDeviceCount(&numDev);
  if (hipErrorNoDevice == err) {
    numDev = 0;
  } else {
    CUDA_VERIFY(err);
  }
  FAISS_ASSERT(numDev != -1);

  return numDev;
}

void synchronizeAllDevices() {
  for (int i = 0; i < getNumDevices(); ++i) {
    DeviceScope scope(i);

    CUDA_VERIFY(hipDeviceSynchronize());
  }
}

const hipDeviceProp_t& getDeviceProperties(int device) {
  static std::mutex mutex;
  static std::unordered_map<int, hipDeviceProp_t> properties;

  std::lock_guard<std::mutex> guard(mutex);

  auto it = properties.find(device);
  if (it == properties.end()) {
    hipDeviceProp_t prop;
    CUDA_VERIFY(hipGetDeviceProperties(&prop, device));

    properties[device] = prop;
    it = properties.find(device);
  }

  return it->second;
}

const hipDeviceProp_t& getCurrentDeviceProperties() {
  return getDeviceProperties(getCurrentDevice());
}

int getMaxThreads(int device) {
  return getDeviceProperties(device).maxThreadsPerBlock;
}

int getMaxThreadsCurrentDevice() {
  return getMaxThreads(getCurrentDevice());
}

size_t getMaxSharedMemPerBlock(int device) {
  return getDeviceProperties(device).sharedMemPerBlock;
}

size_t getMaxSharedMemPerBlockCurrentDevice() {
  return getMaxSharedMemPerBlock(getCurrentDevice());
}

int getDeviceForAddress(const void* p) {
  if (!p) {
    return -1;
  }

  hipPointerAttribute_t att;
  hipError_t err = hipPointerGetAttributes(&att, p);
  FAISS_ASSERT(err == hipSuccess ||
         err == hipErrorInvalidValue);

  if (err == hipErrorInvalidValue) {
    // Make sure the current thread error status has been reset
    err = hipGetLastError();
    FAISS_ASSERT(err == hipErrorInvalidValue);
    return -1;
  } else if (att.memoryType == hipMemoryTypeHost) {
    return -1;
  } else {
    return att.device;
  }
}

bool getFullUnifiedMemSupport(int device) {
  const auto& prop = getDeviceProperties(device);
  return (prop.major >= 6);
}

bool getFullUnifiedMemSupportCurrentDevice() {
  return getFullUnifiedMemSupport(getCurrentDevice());
}

int getMaxKSelection() {
  // Don't use the device at the moment, just base this based on the CUDA SDK
  // that we were compiled with
  return GPU_MAX_SELECTION_K;
}

DeviceScope::DeviceScope(int device) {
  prevDevice_ = getCurrentDevice();

  if (prevDevice_ != device) {
    setCurrentDevice(device);
  } else {
    prevDevice_ = -1;
  }
}

DeviceScope::~DeviceScope() {
  if (prevDevice_ != -1) {
    setCurrentDevice(prevDevice_);
  }
}

CublasHandleScope::CublasHandleScope() {
  auto blasStatus = hipblasCreate(&blasHandle_);
  FAISS_ASSERT(blasStatus == HIPBLAS_STATUS_SUCCESS);
}

CublasHandleScope::~CublasHandleScope() {
  auto blasStatus = hipblasDestroy(blasHandle_);
  FAISS_ASSERT(blasStatus == HIPBLAS_STATUS_SUCCESS);
}

CudaEvent::CudaEvent(hipStream_t stream)
    : event_(0) {
  CUDA_VERIFY(hipEventCreateWithFlags(&event_, hipEventDisableTiming));
  CUDA_VERIFY(hipEventRecord(event_, stream));
}

CudaEvent::CudaEvent(CudaEvent&& event) noexcept
    : event_(std::move(event.event_)) {
  event.event_ = 0;
}

CudaEvent::~CudaEvent() {
  if (event_) {
    CUDA_VERIFY(hipEventDestroy(event_));
  }
}

CudaEvent&
CudaEvent::operator=(CudaEvent&& event) noexcept {
  event_ = std::move(event.event_);
  event.event_ = 0;

  return *this;
}

void
CudaEvent::streamWaitOnEvent(hipStream_t stream) {
  CUDA_VERIFY(hipStreamWaitEvent(stream, event_, 0));
}

void
CudaEvent::cpuWaitOnEvent() {
  CUDA_VERIFY(hipEventSynchronize(event_));
}

} } // namespace
