/**
 * Copyright (c) 2015-present, Facebook, Inc.
 * All rights reserved.
 *
 * This source code is licensed under the BSD+Patents license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include "BlockSelectImpl.cuh"

namespace faiss { namespace gpu {

#ifdef FAISS_USE_FLOAT16
BLOCK_SELECT_IMPL(half, true, 1024, 8);
#endif

} } // namespace
