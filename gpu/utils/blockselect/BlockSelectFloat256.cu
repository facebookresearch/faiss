/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include "BlockSelectImpl.cuh"

namespace faiss { namespace gpu {

BLOCK_SELECT_IMPL(float, true, 256, 4);
BLOCK_SELECT_IMPL(float, false, 256, 4);

} } // namespace
