/**
 * Copyright (c) 2015-present, Facebook, Inc.
 * All rights reserved.
 *
 * This source code is licensed under the BSD+Patents license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include "warpselect/WarpSelectImpl.cuh"

namespace faiss { namespace gpu {

// warp Q to thread Q:
// 1, 1
// 32, 2
// 64, 3
// 128, 3
// 256, 4
// 512, 8
// 1024, 8

WARP_SELECT_DECL(float, true, 1);
WARP_SELECT_DECL(float, true, 32);
WARP_SELECT_DECL(float, true, 64);
WARP_SELECT_DECL(float, true, 128);
WARP_SELECT_DECL(float, true, 256);
WARP_SELECT_DECL(float, true, 512);
WARP_SELECT_DECL(float, true, 1024);

WARP_SELECT_DECL(float, false, 1);
WARP_SELECT_DECL(float, false, 32);
WARP_SELECT_DECL(float, false, 64);
WARP_SELECT_DECL(float, false, 128);
WARP_SELECT_DECL(float, false, 256);
WARP_SELECT_DECL(float, false, 512);
WARP_SELECT_DECL(float, false, 1024);

void runWarpSelect(Tensor<float, 2, true>& in,
                      Tensor<float, 2, true>& outK,
                      Tensor<int, 2, true>& outV,
                      bool dir, int k, hipStream_t stream) {
  FAISS_ASSERT(k <= 1024);

  if (dir) {
    if (k == 1) {
      WARP_SELECT_CALL(float, true, 1);
    } else if (k <= 32) {
      WARP_SELECT_CALL(float, true, 32);
    } else if (k <= 64) {
      WARP_SELECT_CALL(float, true, 64);
    } else if (k <= 128) {
      WARP_SELECT_CALL(float, true, 128);
    } else if (k <= 256) {
      WARP_SELECT_CALL(float, true, 256);
    } else if (k <= 512) {
      WARP_SELECT_CALL(float, true, 512);
    } else if (k <= 1024) {
      WARP_SELECT_CALL(float, true, 1024);
    }
  } else {
    if (k == 1) {
      WARP_SELECT_CALL(float, false, 1);
    } else if (k <= 32) {
      WARP_SELECT_CALL(float, false, 32);
    } else if (k <= 64) {
      WARP_SELECT_CALL(float, false, 64);
    } else if (k <= 128) {
      WARP_SELECT_CALL(float, false, 128);
    } else if (k <= 256) {
      WARP_SELECT_CALL(float, false, 256);
    } else if (k <= 512) {
      WARP_SELECT_CALL(float, false, 512);
    } else if (k <= 1024) {
      WARP_SELECT_CALL(float, false, 1024);
    }
  }
}

} } // namespace
