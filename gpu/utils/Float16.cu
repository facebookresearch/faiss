#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */


#include "Float16.cuh"
#include "nvidia/fp16_emu.cuh"
#include <thrust/execution_policy.h>
#include <thrust/transform.h>

#ifdef FAISS_USE_FLOAT16

namespace faiss { namespace gpu {

bool getDeviceSupportsFloat16Math(int device) {
  const auto& prop = getDeviceProperties(device);

  return (prop.major >= 6 ||
          (prop.major == 5 && prop.minor >= 3));
}

struct FloatToHalf {
  __device__ half operator()(float v) const { return __float2half(v); }
};

struct HalfToFloat {
  __device__ float operator()(half v) const { return __half2float(v); }
};

void runConvertToFloat16(half* out,
                         const float* in,
                         size_t num,
                         hipStream_t stream) {
  thrust::transform(thrust::cuda::par.on(stream),
                    in, in + num, out, FloatToHalf());
}

void runConvertToFloat32(float* out,
                         const half* in,
                         size_t num,
                         hipStream_t stream) {
  thrust::transform(thrust::cuda::par.on(stream),
                    in, in + num, out, HalfToFloat());
}

__half hostFloat2Half(float a) {
#if CUDA_VERSION >= 9000
  __half_raw raw;
  raw.x = cpu_float2half_rn(a).x;
  return __half(raw);
#else
  __half h;
  h.x = cpu_float2half_rn(a).x;
  return h;
#endif
}

} } // namespace

#endif // FAISS_USE_FLOAT16
