#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */


#include "IVFPQ.cuh"
#include "../GpuResources.h"
#include "BroadcastSum.cuh"
#include "Distance.cuh"
#include "FlatIndex.cuh"
#include "InvertedListAppend.cuh"
#include "L2Norm.cuh"
#include "PQCodeDistances.cuh"
#include "PQScanMultiPassNoPrecomputed.cuh"
#include "PQScanMultiPassPrecomputed.cuh"
#include "RemapIndices.h"
#include "VectorResidual.cuh"
#include "../utils/DeviceDefs.cuh"
#include "../utils/DeviceUtils.h"
#include "../utils/HostTensor.cuh"
#include "../utils/MatrixMult.cuh"
#include "../utils/NoTypeTensor.cuh"
#include "../utils/Transpose.cuh"
#include <limits>
#include <thrust/host_vector.h>
#include <unordered_map>

namespace faiss { namespace gpu {

IVFPQ::IVFPQ(GpuResources* resources,
             FlatIndex* quantizer,
             int numSubQuantizers,
             int bitsPerSubQuantizer,
             float* pqCentroidData,
             IndicesOptions indicesOptions,
             bool useFloat16LookupTables,
             MemorySpace space) :
    IVFBase(resources,
            quantizer,
            numSubQuantizers,
            indicesOptions,
            space),
    numSubQuantizers_(numSubQuantizers),
    bitsPerSubQuantizer_(bitsPerSubQuantizer),
    numSubQuantizerCodes_(utils::pow2(bitsPerSubQuantizer_)),
    dimPerSubQuantizer_(dim_ / numSubQuantizers),
    precomputedCodes_(false),
    useFloat16LookupTables_(useFloat16LookupTables) {
  FAISS_ASSERT(pqCentroidData);

  FAISS_ASSERT(bitsPerSubQuantizer_ <= 8);
  FAISS_ASSERT(dim_ % numSubQuantizers_ == 0);
  FAISS_ASSERT(isSupportedPQCodeLength(bytesPerVector_));

#ifndef FAISS_USE_FLOAT16
  FAISS_ASSERT(!useFloat16LookupTables_);
#endif

  setPQCentroids_(pqCentroidData);
}

IVFPQ::~IVFPQ() {
}


bool
IVFPQ::isSupportedPQCodeLength(int size) {
  switch (size) {
    case 1:
    case 2:
    case 3:
    case 4:
    case 8:
    case 12:
    case 16:
    case 20:
    case 24:
    case 28:
    case 32:
    case 40:
    case 48:
    case 56: // only supported with float16
    case 64: // only supported with float16
    case 96: // only supported with float16
      return true;
    default:
      return false;
  }
}

bool
IVFPQ::isSupportedNoPrecomputedSubDimSize(int dims) {
  return faiss::gpu::isSupportedNoPrecomputedSubDimSize(dims);
}

void
IVFPQ::setPrecomputedCodes(bool enable) {
  if (precomputedCodes_ != enable) {
    precomputedCodes_ = enable;

    if (precomputedCodes_) {
      precomputeCodes_();
    } else {
      // Clear out old precomputed code data
      precomputedCode_ = std::move(DeviceTensor<float, 3, true>());

#ifdef FAISS_USE_FLOAT16
      precomputedCodeHalf_ = std::move(DeviceTensor<half, 3, true>());
#endif
    }
  }
}

int
IVFPQ::classifyAndAddVectors(Tensor<float, 2, true>& vecs,
                             Tensor<long, 1, true>& indices) {
  FAISS_ASSERT(vecs.getSize(0) == indices.getSize(0));
  FAISS_ASSERT(vecs.getSize(1) == dim_);

  FAISS_ASSERT(!quantizer_->getUseFloat16());
  auto& coarseCentroids = quantizer_->getVectorsFloat32Ref();
  auto& mem = resources_->getMemoryManagerCurrentDevice();
  auto stream = resources_->getDefaultStreamCurrentDevice();

  // Number of valid vectors that we actually add; we return this
  int numAdded = 0;

  // We don't actually need this
  DeviceTensor<float, 2, true> listDistance(mem, {vecs.getSize(0), 1}, stream);
  // We use this
  DeviceTensor<int, 2, true> listIds2d(mem, {vecs.getSize(0), 1}, stream);
  auto listIds = listIds2d.view<1>({vecs.getSize(0)});

  quantizer_->query(vecs, 1, listDistance, listIds2d, false);

  // Copy the lists that we wish to append to back to the CPU
  // FIXME: really this can be into pinned memory and a true async
  // copy on a different stream; we can start the copy early, but it's
  // tiny
  HostTensor<int, 1, true> listIdsHost(listIds, stream);

  // Calculate the residual for each closest centroid
  DeviceTensor<float, 2, true> residuals(
    mem, {vecs.getSize(0), vecs.getSize(1)}, stream);

  runCalcResidual(vecs, coarseCentroids, listIds, residuals, stream);

  // Residuals are in the form
  // (vec x numSubQuantizer x dimPerSubQuantizer)
  // transpose to
  // (numSubQuantizer x vec x dimPerSubQuantizer)
  auto residualsView = residuals.view<3>(
    {residuals.getSize(0), numSubQuantizers_, dimPerSubQuantizer_});

  DeviceTensor<float, 3, true> residualsTranspose(
    mem,
    {numSubQuantizers_, residuals.getSize(0), dimPerSubQuantizer_},
    stream);

  runTransposeAny(residualsView, 0, 1, residualsTranspose, stream);

  // Get the product quantizer centroids in the form
  // (numSubQuantizer x numSubQuantizerCodes x dimPerSubQuantizer)
  // which is pqCentroidsMiddleCode_

  // We now have a batch operation to find the top-1 distances:
  // batch size: numSubQuantizer
  // centroids: (numSubQuantizerCodes x dimPerSubQuantizer)
  // residuals: (vec x dimPerSubQuantizer)
  // => (numSubQuantizer x vec x 1)

  DeviceTensor<float, 3, true> closestSubQDistance(
    mem, {numSubQuantizers_, residuals.getSize(0), 1}, stream);
  DeviceTensor<int, 3, true> closestSubQIndex(
    mem, {numSubQuantizers_, residuals.getSize(0), 1}, stream);

  for (int subQ = 0; subQ < numSubQuantizers_; ++subQ) {
    auto closestSubQDistanceView = closestSubQDistance[subQ].view();
    auto closestSubQIndexView = closestSubQIndex[subQ].view();

    auto pqCentroidsMiddleCodeView = pqCentroidsMiddleCode_[subQ].view();
    auto residualsTransposeView = residualsTranspose[subQ].view();

    runL2Distance(resources_,
                  pqCentroidsMiddleCodeView,
                  true, // pqCentroidsMiddleCodeView is row major
                  nullptr, // no precomputed norms
                  residualsTransposeView,
                  true, // residualsTransposeView is row major
                  1,
                  closestSubQDistanceView,
                  closestSubQIndexView,
                  // We don't care about distances
                  true);
  }

  // Now, we have the nearest sub-q centroid for each slice of the
  // residual vector.
  auto closestSubQIndexView = closestSubQIndex.view<2>(
    {numSubQuantizers_, residuals.getSize(0)});

  // Transpose this for easy use
  DeviceTensor<int, 2, true> encodings(
    mem, {residuals.getSize(0), numSubQuantizers_}, stream);

  runTransposeAny(closestSubQIndexView, 0, 1, encodings, stream);

  // Now we add the encoded vectors to the individual lists
  // First, make sure that there is space available for adding the new
  // encoded vectors and indices

  // list id -> # being added
  std::unordered_map<int, int> assignCounts;

  // vector id -> offset in list
  // (we already have vector id -> list id in listIds)
  HostTensor<int, 1, true> listOffsetHost({listIdsHost.getSize(0)});

  for (int i = 0; i < listIdsHost.getSize(0); ++i) {
    int listId = listIdsHost[i];

    // Add vector could be invalid (contains NaNs etc)
    if (listId < 0) {
      listOffsetHost[i] = -1;
      continue;
    }

    FAISS_ASSERT(listId < numLists_);
    ++numAdded;

    int offset = deviceListData_[listId]->size() / bytesPerVector_;

    auto it = assignCounts.find(listId);
    if (it != assignCounts.end()) {
      offset += it->second;
      it->second++;
    } else {
      assignCounts[listId] = 1;
    }

    listOffsetHost[i] = offset;
  }

  // If we didn't add anything (all invalid vectors), no need to
  // continue
  if (numAdded == 0) {
    return 0;
  }

  // We need to resize the data structures for the inverted lists on
  // the GPUs, which means that they might need reallocation, which
  // means that their base address may change. Figure out the new base
  // addresses, and update those in a batch on the device
  {
    // Resize all of the lists that we are appending to
    for (auto& counts : assignCounts) {
      auto& codes = deviceListData_[counts.first];
      codes->resize(codes->size() + counts.second * bytesPerVector_,
                    stream);
      int newNumVecs = (int) (codes->size() / bytesPerVector_);

      auto& indices = deviceListIndices_[counts.first];
      if ((indicesOptions_ == INDICES_32_BIT) ||
          (indicesOptions_ == INDICES_64_BIT)) {
        size_t indexSize =
          (indicesOptions_ == INDICES_32_BIT) ? sizeof(int) : sizeof(long);

        indices->resize(indices->size() + counts.second * indexSize, stream);
      } else if (indicesOptions_ == INDICES_CPU) {
        // indices are stored on the CPU side
        FAISS_ASSERT(counts.first < listOffsetToUserIndex_.size());

        auto& userIndices = listOffsetToUserIndex_[counts.first];
        userIndices.resize(newNumVecs);
      } else {
        // indices are not stored on the GPU or CPU side
        FAISS_ASSERT(indicesOptions_ == INDICES_IVF);
      }

      // This is used by the multi-pass query to decide how much scratch
      // space to allocate for intermediate results
      maxListLength_ = std::max(maxListLength_, newNumVecs);
    }

    // Update all pointers and sizes on the device for lists that we
    // appended to
    {
      std::vector<int> listIds(assignCounts.size());
      int i = 0;
      for (auto& counts : assignCounts) {
        listIds[i++] = counts.first;
      }

      updateDeviceListInfo_(listIds, stream);
    }
  }

  // If we're maintaining the indices on the CPU side, update our
  // map. We already resized our map above.
  if (indicesOptions_ == INDICES_CPU) {
    // We need to maintain the indices on the CPU side
    HostTensor<long, 1, true> hostIndices(indices, stream);

    for (int i = 0; i < hostIndices.getSize(0); ++i) {
      int listId = listIdsHost[i];

      // Add vector could be invalid (contains NaNs etc)
      if (listId < 0) {
        continue;
      }

      int offset = listOffsetHost[i];

      FAISS_ASSERT(listId < listOffsetToUserIndex_.size());
      auto& userIndices = listOffsetToUserIndex_[listId];

      FAISS_ASSERT(offset < userIndices.size());
      userIndices[offset] = hostIndices[i];
    }
  }

  // We similarly need to actually append the new encoded vectors
  {
    DeviceTensor<int, 1, true> listOffset(mem, listOffsetHost, stream);

    // This kernel will handle appending each encoded vector + index to
    // the appropriate list
    runIVFPQInvertedListAppend(listIds,
                               listOffset,
                               encodings,
                               indices,
                               deviceListDataPointers_,
                               deviceListIndexPointers_,
                               indicesOptions_,
                               stream);
  }

  return numAdded;
}

void
IVFPQ::addCodeVectorsFromCpu(int listId,
                             const void* codes,
                             const long* indices,
                             size_t numVecs) {
  // This list must already exist
  FAISS_ASSERT(listId < deviceListData_.size());
  auto stream = resources_->getDefaultStreamCurrentDevice();

  // If there's nothing to add, then there's nothing we have to do
  if (numVecs == 0) {
    return;
  }

  size_t lengthInBytes = numVecs * bytesPerVector_;

  auto& listCodes = deviceListData_[listId];
  auto prevCodeData = listCodes->data();

  // We only have int32 length representations on the GPU per each
  // list; the length is in sizeof(char)
  FAISS_ASSERT(listCodes->size() % bytesPerVector_ == 0);
  FAISS_ASSERT(listCodes->size() + lengthInBytes <=
               (size_t) std::numeric_limits<int>::max());

  listCodes->append((unsigned char*) codes,
                    lengthInBytes,
                    stream,
                    true /* exact reserved size */);

  // Handle the indices as well
  addIndicesFromCpu_(listId, indices, numVecs);

  // This list address may have changed due to vector resizing, but
  // only bother updating it on the device if it has changed
  if (prevCodeData != listCodes->data()) {
    deviceListDataPointers_[listId] = listCodes->data();
  }

  // And our size has changed too
  int listLength = listCodes->size() / bytesPerVector_;
  deviceListLengths_[listId] = listLength;

  // We update this as well, since the multi-pass algorithm uses it
  maxListLength_ = std::max(maxListLength_, listLength);

  // device_vector add is potentially happening on a different stream
  // than our default stream
  if (resources_->getDefaultStreamCurrentDevice() != 0) {
    streamWait({stream}, {0});
  }
}

void
IVFPQ::setPQCentroids_(float* data) {
  size_t pqSize =
    numSubQuantizers_ * numSubQuantizerCodes_ * dimPerSubQuantizer_;

  // Make sure the data is on the host
  // FIXME: why are we doing this?
  thrust::host_vector<float> hostMemory;
  hostMemory.insert(hostMemory.end(), data, data + pqSize);

  HostTensor<float, 3, true> pqHost(
    hostMemory.data(),
    {numSubQuantizers_, numSubQuantizerCodes_, dimPerSubQuantizer_});
  DeviceTensor<float, 3, true> pqDevice(
    pqHost,
    resources_->getDefaultStreamCurrentDevice());

  DeviceTensor<float, 3, true> pqDeviceTranspose(
    {numSubQuantizers_, dimPerSubQuantizer_, numSubQuantizerCodes_});
  runTransposeAny(pqDevice, 1, 2, pqDeviceTranspose,
                  resources_->getDefaultStreamCurrentDevice());

  pqCentroidsInnermostCode_ = std::move(pqDeviceTranspose);

  // Also maintain the PQ centroids in the form
  // (sub q)(code id)(sub dim)
  DeviceTensor<float, 3, true> pqCentroidsMiddleCode(
    {numSubQuantizers_, numSubQuantizerCodes_, dimPerSubQuantizer_});
  runTransposeAny(pqCentroidsInnermostCode_, 1, 2, pqCentroidsMiddleCode,
                  resources_->getDefaultStreamCurrentDevice());

  pqCentroidsMiddleCode_ = std::move(pqCentroidsMiddleCode);
}

void
IVFPQ::precomputeCodes_() {
  //
  //    d = || x - y_C ||^2 + || y_R ||^2 + 2 * (y_C|y_R) - 2 * (x|y_R)
  //        ---------------   ---------------------------       -------
  //            term 1                 term 2                   term 3
  //

  // Terms 1 and 3 are available only at query time. We compute term 2
  // here.
  FAISS_ASSERT(!quantizer_->getUseFloat16());
  auto& coarseCentroids = quantizer_->getVectorsFloat32Ref();

  // Compute ||y_R||^2 by treating
  // (sub q)(code id)(sub dim) as (sub q * code id)(sub dim)
  auto pqCentroidsMiddleCodeView =
    pqCentroidsMiddleCode_.view<2>(
      {numSubQuantizers_ * numSubQuantizerCodes_, dimPerSubQuantizer_});
  DeviceTensor<float, 1, true> subQuantizerNorms(
    {numSubQuantizers_ * numSubQuantizerCodes_});

  runL2Norm(pqCentroidsMiddleCodeView, true,
            subQuantizerNorms, true,
            resources_->getDefaultStreamCurrentDevice());

  // Compute 2 * (y_C|y_R) via batch matrix multiplication
  // batch size (sub q) x {(centroid id)(sub dim) x (code id)(sub dim)'}
  //         => (sub q) x {(centroid id)(code id)}
  //         => (sub q)(centroid id)(code id)

  // View (centroid id)(dim) as
  //      (centroid id)(sub q)(dim)
  // Transpose (centroid id)(sub q)(sub dim) to
  //           (sub q)(centroid id)(sub dim)
  auto centroidView = coarseCentroids.view<3>(
    {coarseCentroids.getSize(0), numSubQuantizers_, dimPerSubQuantizer_});
  DeviceTensor<float, 3, true> centroidsTransposed(
    {numSubQuantizers_, coarseCentroids.getSize(0), dimPerSubQuantizer_});

  runTransposeAny(centroidView, 0, 1, centroidsTransposed,
                  resources_->getDefaultStreamCurrentDevice());

  DeviceTensor<float, 3, true> coarsePQProduct(
    {numSubQuantizers_, coarseCentroids.getSize(0), numSubQuantizerCodes_});

  runIteratedMatrixMult(coarsePQProduct, false,
                        centroidsTransposed, false,
                        pqCentroidsMiddleCode_, true,
                        2.0f, 0.0f,
                        resources_->getBlasHandleCurrentDevice(),
                        resources_->getDefaultStreamCurrentDevice());

  // Transpose (sub q)(centroid id)(code id) to
  //           (centroid id)(sub q)(code id)
  DeviceTensor<float, 3, true> coarsePQProductTransposed(
    {coarseCentroids.getSize(0), numSubQuantizers_, numSubQuantizerCodes_});
  runTransposeAny(coarsePQProduct, 0, 1, coarsePQProductTransposed,
                  resources_->getDefaultStreamCurrentDevice());

  // View (centroid id)(sub q)(code id) as
  //      (centroid id)(sub q * code id)
  auto coarsePQProductTransposedView = coarsePQProductTransposed.view<2>(
    {coarseCentroids.getSize(0), numSubQuantizers_ * numSubQuantizerCodes_});

  // Sum || y_R ||^2 + 2 * (y_C|y_R)
  // i.e., add norms                              (sub q * code id)
  // along columns of inner product  (centroid id)(sub q * code id)
  runSumAlongColumns(subQuantizerNorms, coarsePQProductTransposedView,
                     resources_->getDefaultStreamCurrentDevice());

#ifdef FAISS_USE_FLOAT16
  if (useFloat16LookupTables_) {
    precomputedCodeHalf_ = toHalf(resources_,
                                  resources_->getDefaultStreamCurrentDevice(),
                                  coarsePQProductTransposed);
    return;
  }
#endif

  // We added into the view, so `coarsePQProductTransposed` is now our
  // precomputed term 2.
  precomputedCode_ = std::move(coarsePQProductTransposed);
}

void
IVFPQ::query(Tensor<float, 2, true>& queries,
             int nprobe,
             int k,
             Tensor<float, 2, true>& outDistances,
             Tensor<long, 2, true>& outIndices) {
  // These are caught at a higher level
  FAISS_ASSERT(nprobe <= GPU_MAX_SELECTION_K);
  FAISS_ASSERT(k <= GPU_MAX_SELECTION_K);

  auto& mem = resources_->getMemoryManagerCurrentDevice();
  auto stream = resources_->getDefaultStreamCurrentDevice();
  nprobe = std::min(nprobe, quantizer_->getSize());

  FAISS_ASSERT(queries.getSize(1) == dim_);
  FAISS_ASSERT(outDistances.getSize(0) == queries.getSize(0));
  FAISS_ASSERT(outIndices.getSize(0) == queries.getSize(0));

  // Reserve space for the closest coarse centroids
  DeviceTensor<float, 2, true>
    coarseDistances(mem, {queries.getSize(0), nprobe}, stream);
  DeviceTensor<int, 2, true>
    coarseIndices(mem, {queries.getSize(0), nprobe}, stream);

  // Find the `nprobe` closest coarse centroids; we can use int
  // indices both internally and externally
  quantizer_->query(queries,
                    nprobe,
                    coarseDistances,
                    coarseIndices,
                    true);

  if (precomputedCodes_) {
    runPQPrecomputedCodes_(queries,
                           coarseDistances,
                           coarseIndices,
                           k,
                           outDistances,
                           outIndices);
  } else {
    runPQNoPrecomputedCodes_(queries,
                             coarseDistances,
                             coarseIndices,
                             k,
                             outDistances,
                             outIndices);
  }

  // If the GPU isn't storing indices (they are on the CPU side), we
  // need to perform the re-mapping here
  // FIXME: we might ultimately be calling this function with inputs
  // from the CPU, these are unnecessary copies
  if (indicesOptions_ == INDICES_CPU) {
    HostTensor<long, 2, true> hostOutIndices(outIndices, stream);

    ivfOffsetToUserIndex(hostOutIndices.data(),
                         numLists_,
                         hostOutIndices.getSize(0),
                         hostOutIndices.getSize(1),
                         listOffsetToUserIndex_);

    // Copy back to GPU, since the input to this function is on the
    // GPU
    outIndices.copyFrom(hostOutIndices, stream);
  }
}

std::vector<unsigned char>
IVFPQ::getListCodes(int listId) const {
  FAISS_ASSERT(listId < deviceListData_.size());

  return deviceListData_[listId]->copyToHost<unsigned char>(
    resources_->getDefaultStreamCurrentDevice());
}

Tensor<float, 3, true>
IVFPQ::getPQCentroids() {
  return pqCentroidsMiddleCode_;
}

void
IVFPQ::runPQPrecomputedCodes_(
  Tensor<float, 2, true>& queries,
  DeviceTensor<float, 2, true>& coarseDistances,
  DeviceTensor<int, 2, true>& coarseIndices,
  int k,
  Tensor<float, 2, true>& outDistances,
  Tensor<long, 2, true>& outIndices) {
  auto& mem = resources_->getMemoryManagerCurrentDevice();
  auto stream = resources_->getDefaultStreamCurrentDevice();

  // Compute precomputed code term 3, - 2 * (x|y_R)
  // This is done via batch MM
  // {sub q} x {(query id)(sub dim) * (code id)(sub dim)'} =>
  // {sub q} x {(query id)(code id)}
  DeviceTensor<float, 3, true> term3Transposed(
    mem,
    {queries.getSize(0), numSubQuantizers_, numSubQuantizerCodes_},
    stream);

  // These allocations within are only temporary, so release them when
  // we're done to maximize free space
  {
    auto querySubQuantizerView = queries.view<3>(
      {queries.getSize(0), numSubQuantizers_, dimPerSubQuantizer_});
    DeviceTensor<float, 3, true> queriesTransposed(
      mem,
      {numSubQuantizers_, queries.getSize(0), dimPerSubQuantizer_},
      stream);
    runTransposeAny(querySubQuantizerView, 0, 1, queriesTransposed, stream);

    DeviceTensor<float, 3, true> term3(
      mem,
      {numSubQuantizers_, queries.getSize(0), numSubQuantizerCodes_},
      stream);

    runIteratedMatrixMult(term3, false,
                          queriesTransposed, false,
                          pqCentroidsMiddleCode_, true,
                          -2.0f, 0.0f,
                          resources_->getBlasHandleCurrentDevice(),
                          stream);

    runTransposeAny(term3, 0, 1, term3Transposed, stream);
  }

  NoTypeTensor<3, true> term2;
  NoTypeTensor<3, true> term3;
#ifdef FAISS_USE_FLOAT16
  DeviceTensor<half, 3, true> term3Half;

  if (useFloat16LookupTables_) {
    term3Half = toHalf(resources_, stream, term3Transposed);
    term2 = NoTypeTensor<3, true>(precomputedCodeHalf_);
    term3 = NoTypeTensor<3, true>(term3Half);
  }
#endif

  if (!useFloat16LookupTables_) {
    term2 = NoTypeTensor<3, true>(precomputedCode_);
    term3 = NoTypeTensor<3, true>(term3Transposed);
  }

  runPQScanMultiPassPrecomputed(queries,
                                coarseDistances, // term 1
                                term2, // term 2
                                term3, // term 3
                                coarseIndices,
                                useFloat16LookupTables_,
                                bytesPerVector_,
                                numSubQuantizers_,
                                numSubQuantizerCodes_,
                                deviceListDataPointers_,
                                deviceListIndexPointers_,
                                indicesOptions_,
                                deviceListLengths_,
                                maxListLength_,
                                k,
                                outDistances,
                                outIndices,
                                resources_);
}

void
IVFPQ::runPQNoPrecomputedCodes_(
  Tensor<float, 2, true>& queries,
  DeviceTensor<float, 2, true>& coarseDistances,
  DeviceTensor<int, 2, true>& coarseIndices,
  int k,
  Tensor<float, 2, true>& outDistances,
  Tensor<long, 2, true>& outIndices) {
  FAISS_ASSERT(!quantizer_->getUseFloat16());
  auto& coarseCentroids = quantizer_->getVectorsFloat32Ref();

  runPQScanMultiPassNoPrecomputed(queries,
                                  coarseCentroids,
                                  pqCentroidsInnermostCode_,
                                  coarseIndices,
                                  useFloat16LookupTables_,
                                  bytesPerVector_,
                                  numSubQuantizers_,
                                  numSubQuantizerCodes_,
                                  deviceListDataPointers_,
                                  deviceListIndexPointers_,
                                  indicesOptions_,
                                  deviceListLengths_,
                                  maxListLength_,
                                  k,
                                  outDistances,
                                  outIndices,
                                  resources_);
}

} } // namespace
