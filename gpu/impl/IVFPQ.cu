#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */


#include <faiss/gpu/impl/IVFPQ.cuh>
#include <faiss/gpu/GpuResources.h>
#include <faiss/gpu/impl/BroadcastSum.cuh>
#include <faiss/gpu/impl/Distance.cuh>
#include <faiss/gpu/impl/FlatIndex.cuh>
#include <faiss/gpu/impl/IVFAppend.cuh>
#include <faiss/gpu/impl/L2Norm.cuh>
#include <faiss/gpu/impl/PQCodeDistances.cuh>
#include <faiss/gpu/impl/PQScanMultiPassNoPrecomputed.cuh>
#include <faiss/gpu/impl/PQScanMultiPassPrecomputed.cuh>
#include <faiss/gpu/impl/RemapIndices.h>
#include <faiss/gpu/impl/VectorResidual.cuh>
#include <faiss/gpu/utils/ConversionOperators.cuh>
#include <faiss/gpu/utils/DeviceDefs.cuh>
#include <faiss/gpu/utils/DeviceUtils.h>
#include <faiss/gpu/utils/HostTensor.cuh>
#include <faiss/gpu/utils/MatrixMult.cuh>
#include <faiss/gpu/utils/NoTypeTensor.cuh>
#include <faiss/gpu/utils/Transpose.cuh>
#include <limits>
#include <thrust/host_vector.h>
#include <unordered_map>

namespace faiss { namespace gpu {

IVFPQ::IVFPQ(GpuResources* resources,
             faiss::MetricType metric,
             float metricArg,
             FlatIndex* quantizer,
             int numSubQuantizers,
             int bitsPerSubQuantizer,
             float* pqCentroidData,
             IndicesOptions indicesOptions,
             bool useFloat16LookupTables,
             MemorySpace space) :
    IVFBase(resources,
            metric,
            metricArg,
            quantizer,
            numSubQuantizers,
            indicesOptions,
            space),
    numSubQuantizers_(numSubQuantizers),
    bitsPerSubQuantizer_(bitsPerSubQuantizer),
    numSubQuantizerCodes_(utils::pow2(bitsPerSubQuantizer_)),
    dimPerSubQuantizer_(dim_ / numSubQuantizers),
    useFloat16LookupTables_(useFloat16LookupTables),
    precomputedCodes_(false) {
  FAISS_ASSERT(pqCentroidData);

  FAISS_ASSERT(bitsPerSubQuantizer_ <= 8);
  FAISS_ASSERT(dim_ % numSubQuantizers_ == 0);
  FAISS_ASSERT(isSupportedPQCodeLength(bytesPerVector_));

  setPQCentroids_(pqCentroidData);
}

IVFPQ::~IVFPQ() {
}


bool
IVFPQ::isSupportedPQCodeLength(int size) {
  switch (size) {
    case 1:
    case 2:
    case 3:
    case 4:
    case 8:
    case 12:
    case 16:
    case 20:
    case 24:
    case 28:
    case 32:
    case 40:
    case 48:
    case 56: // only supported with float16
    case 64: // only supported with float16
    case 96: // only supported with float16
      return true;
    default:
      return false;
  }
}

bool
IVFPQ::isSupportedNoPrecomputedSubDimSize(int dims) {
  return faiss::gpu::isSupportedNoPrecomputedSubDimSize(dims);
}

void
IVFPQ::setPrecomputedCodes(bool enable) {
  if (enable && metric_ == MetricType::METRIC_INNER_PRODUCT) {
    FAISS_THROW_MSG("Precomputed codes are not needed for GpuIndexIVFPQ "
                    "with METRIC_INNER_PRODUCT");
  }

  if (precomputedCodes_ != enable) {
    precomputedCodes_ = enable;

    if (precomputedCodes_) {
      precomputeCodes_();
    } else {
      // Clear out old precomputed code data
      precomputedCode_ = DeviceTensor<float, 3, true>();
      precomputedCodeHalf_ = DeviceTensor<half, 3, true>();
    }
  }
}

int
IVFPQ::classifyAndAddVectors(Tensor<float, 2, true>& vecs,
                             Tensor<long, 1, true>& indices) {
  FAISS_ASSERT(vecs.getSize(0) == indices.getSize(0));
  FAISS_ASSERT(vecs.getSize(1) == dim_);

  auto stream = resources_->getDefaultStreamCurrentDevice();

  // Number of valid vectors that we actually add; we return this
  int numAdded = 0;

  // We don't actually need this
  DeviceTensor<float, 2, true> listDistance(
    resources_, makeTempAlloc(AllocType::Other, stream),
    {vecs.getSize(0), 1});
  // We use this
  DeviceTensor<int, 2, true> listIds2d(
    resources_, makeTempAlloc(AllocType::Other, stream),
    {vecs.getSize(0), 1});
  auto listIds = listIds2d.view<1>({vecs.getSize(0)});

  quantizer_->query(vecs,
                    1,
                    metric_,
                    metricArg_,
                    listDistance,
                    listIds2d,
                    false);

  // Copy the lists that we wish to append to back to the CPU
  // FIXME: really this can be into pinned memory and a true async
  // copy on a different stream; we can start the copy early, but it's
  // tiny
  HostTensor<int, 1, true> listIdsHost(listIds, stream);

  // Calculate the residual for each closest centroid
  DeviceTensor<float, 2, true> residuals(
    resources_, makeTempAlloc(AllocType::Other, stream),
    {vecs.getSize(0), vecs.getSize(1)});

  if (quantizer_->getUseFloat16()) {
    auto& coarseCentroids = quantizer_->getVectorsFloat16Ref();
    runCalcResidual(vecs, coarseCentroids, listIds, residuals, stream);
  } else {
    auto& coarseCentroids = quantizer_->getVectorsFloat32Ref();
    runCalcResidual(vecs, coarseCentroids, listIds, residuals, stream);
  }

  // Residuals are in the form
  // (vec x numSubQuantizer x dimPerSubQuantizer)
  // transpose to
  // (numSubQuantizer x vec x dimPerSubQuantizer)
  auto residualsView = residuals.view<3>(
    {residuals.getSize(0), numSubQuantizers_, dimPerSubQuantizer_});

  DeviceTensor<float, 3, true> residualsTranspose(
    resources_, makeTempAlloc(AllocType::Other, stream),
    {numSubQuantizers_, residuals.getSize(0), dimPerSubQuantizer_});

  runTransposeAny(residualsView, 0, 1, residualsTranspose, stream);

  // Get the product quantizer centroids in the form
  // (numSubQuantizer x numSubQuantizerCodes x dimPerSubQuantizer)
  // which is pqCentroidsMiddleCode_

  // We now have a batch operation to find the top-1 distances:
  // batch size: numSubQuantizer
  // centroids: (numSubQuantizerCodes x dimPerSubQuantizer)
  // residuals: (vec x dimPerSubQuantizer)
  // => (numSubQuantizer x vec x 1)

  DeviceTensor<float, 3, true> closestSubQDistance(
    resources_, makeTempAlloc(AllocType::Other, stream),
    {numSubQuantizers_, residuals.getSize(0), 1});
  DeviceTensor<int, 3, true> closestSubQIndex(
    resources_, makeTempAlloc(AllocType::Other, stream),
    {numSubQuantizers_, residuals.getSize(0), 1});

  for (int subQ = 0; subQ < numSubQuantizers_; ++subQ) {
    auto closestSubQDistanceView = closestSubQDistance[subQ].view();
    auto closestSubQIndexView = closestSubQIndex[subQ].view();

    auto pqCentroidsMiddleCodeView = pqCentroidsMiddleCode_[subQ].view();
    auto residualsTransposeView = residualsTranspose[subQ].view();

    runL2Distance(resources_,
                  pqCentroidsMiddleCodeView,
                  true, // pqCentroidsMiddleCodeView is row major
                  nullptr, // no precomputed norms
                  residualsTransposeView,
                  true, // residualsTransposeView is row major
                  1,
                  closestSubQDistanceView,
                  closestSubQIndexView,
                  // We don't care about distances
                  true);
  }

  // Now, we have the nearest sub-q centroid for each slice of the
  // residual vector.
  auto closestSubQIndexView = closestSubQIndex.view<2>(
    {numSubQuantizers_, residuals.getSize(0)});

  // Transpose this for easy use
  DeviceTensor<int, 2, true> encodings(
    resources_, makeTempAlloc(AllocType::Other, stream),
    {residuals.getSize(0), numSubQuantizers_});

  runTransposeAny(closestSubQIndexView, 0, 1, encodings, stream);

  // Now we add the encoded vectors to the individual lists
  // First, make sure that there is space available for adding the new
  // encoded vectors and indices

  // list id -> # being added
  std::unordered_map<int, int> assignCounts;

  // vector id -> offset in list
  // (we already have vector id -> list id in listIds)
  HostTensor<int, 1, true> listOffsetHost({listIdsHost.getSize(0)});

  for (int i = 0; i < listIdsHost.getSize(0); ++i) {
    int listId = listIdsHost[i];

    // Add vector could be invalid (contains NaNs etc)
    if (listId < 0) {
      listOffsetHost[i] = -1;
      continue;
    }

    FAISS_ASSERT(listId < numLists_);
    ++numAdded;

    int offset = deviceListData_[listId]->size() / bytesPerVector_;

    auto it = assignCounts.find(listId);
    if (it != assignCounts.end()) {
      offset += it->second;
      it->second++;
    } else {
      assignCounts[listId] = 1;
    }

    listOffsetHost[i] = offset;
  }

  // If we didn't add anything (all invalid vectors), no need to
  // continue
  if (numAdded == 0) {
    return 0;
  }

  // We need to resize the data structures for the inverted lists on
  // the GPUs, which means that they might need reallocation, which
  // means that their base address may change. Figure out the new base
  // addresses, and update those in a batch on the device
  {
    // Resize all of the lists that we are appending to
    for (auto& counts : assignCounts) {
      auto& codes = deviceListData_[counts.first];
      codes->resize(codes->size() + counts.second * bytesPerVector_,
                    stream);
      int newNumVecs = (int) (codes->size() / bytesPerVector_);

      auto& indices = deviceListIndices_[counts.first];
      if ((indicesOptions_ == INDICES_32_BIT) ||
          (indicesOptions_ == INDICES_64_BIT)) {
        size_t indexSize =
          (indicesOptions_ == INDICES_32_BIT) ? sizeof(int) : sizeof(long);

        indices->resize(indices->size() + counts.second * indexSize, stream);
      } else if (indicesOptions_ == INDICES_CPU) {
        // indices are stored on the CPU side
        FAISS_ASSERT(counts.first < listOffsetToUserIndex_.size());

        auto& userIndices = listOffsetToUserIndex_[counts.first];
        userIndices.resize(newNumVecs);
      } else {
        // indices are not stored on the GPU or CPU side
        FAISS_ASSERT(indicesOptions_ == INDICES_IVF);
      }

      // This is used by the multi-pass query to decide how much scratch
      // space to allocate for intermediate results
      maxListLength_ = std::max(maxListLength_, newNumVecs);
    }

    // Update all pointers and sizes on the device for lists that we
    // appended to
    {
      std::vector<int> listIds(assignCounts.size());
      int i = 0;
      for (auto& counts : assignCounts) {
        listIds[i++] = counts.first;
      }

      updateDeviceListInfo_(listIds, stream);
    }
  }

  // If we're maintaining the indices on the CPU side, update our
  // map. We already resized our map above.
  if (indicesOptions_ == INDICES_CPU) {
    // We need to maintain the indices on the CPU side
    HostTensor<long, 1, true> hostIndices(indices, stream);

    for (int i = 0; i < hostIndices.getSize(0); ++i) {
      int listId = listIdsHost[i];

      // Add vector could be invalid (contains NaNs etc)
      if (listId < 0) {
        continue;
      }

      int offset = listOffsetHost[i];

      FAISS_ASSERT(listId < listOffsetToUserIndex_.size());
      auto& userIndices = listOffsetToUserIndex_[listId];

      FAISS_ASSERT(offset < userIndices.size());
      userIndices[offset] = hostIndices[i];
    }
  }

  // We similarly need to actually append the new encoded vectors
  {
    DeviceTensor<int, 1, true> listOffset(
      resources_, makeTempAlloc(AllocType::Other, stream),
      listOffsetHost);

    // This kernel will handle appending each encoded vector + index to
    // the appropriate list
    runIVFPQInvertedListAppend(listIds,
                               listOffset,
                               encodings,
                               indices,
                               deviceListDataPointers_,
                               deviceListIndexPointers_,
                               indicesOptions_,
                               stream);
  }

  return numAdded;
}

void
IVFPQ::addCodeVectorsFromCpu(int listId,
                             const void* codes,
                             const long* indices,
                             size_t numVecs) {
  // This list must already exist
  FAISS_ASSERT(listId < deviceListData_.size());
  auto stream = resources_->getDefaultStreamCurrentDevice();

  // If there's nothing to add, then there's nothing we have to do
  if (numVecs == 0) {
    return;
  }

  size_t lengthInBytes = numVecs * bytesPerVector_;

  auto& listCodes = deviceListData_[listId];
  auto prevCodeData = listCodes->data();

  // We only have int32 length representations on the GPU per each
  // list; the length is in sizeof(char)
  FAISS_ASSERT(listCodes->size() % bytesPerVector_ == 0);
  FAISS_ASSERT(listCodes->size() + lengthInBytes <=
               (size_t) std::numeric_limits<int>::max());

  listCodes->append((unsigned char*) codes,
                    lengthInBytes,
                    stream,
                    true /* exact reserved size */);

  // Handle the indices as well
  addIndicesFromCpu_(listId, indices, numVecs);

  // This list address may have changed due to vector resizing, but
  // only bother updating it on the device if it has changed
  if (prevCodeData != listCodes->data()) {
    deviceListDataPointers_[listId] = listCodes->data();
  }

  // And our size has changed too
  int listLength = listCodes->size() / bytesPerVector_;
  deviceListLengths_[listId] = listLength;

  // We update this as well, since the multi-pass algorithm uses it
  maxListLength_ = std::max(maxListLength_, listLength);

  // device_vector add is potentially happening on a different stream
  // than our default stream
  if (resources_->getDefaultStreamCurrentDevice() != 0) {
    streamWait({stream}, {0});
  }
}

void
IVFPQ::setPQCentroids_(float* data) {
  auto stream = resources_->getDefaultStreamCurrentDevice();

  size_t pqSize =
    numSubQuantizers_ * numSubQuantizerCodes_ * dimPerSubQuantizer_;

  // Make sure the data is on the host
  // FIXME: why are we doing this?
  thrust::host_vector<float> hostMemory;
  hostMemory.insert(hostMemory.end(), data, data + pqSize);

  HostTensor<float, 3, true> pqHost(
    hostMemory.data(),
    {numSubQuantizers_, numSubQuantizerCodes_, dimPerSubQuantizer_});

  DeviceTensor<float, 3, true> pqDeviceTranspose(
    resources_,
    makeDevAlloc(AllocType::Quantizer, stream),
    {numSubQuantizers_, dimPerSubQuantizer_, numSubQuantizerCodes_});

  {
    // Only needed for the duration of the transposition
    DeviceTensor<float, 3, true> pqDevice(
      resources_,
      makeTempAlloc(AllocType::Quantizer, stream),
      pqHost);

    runTransposeAny(pqDevice, 1, 2, pqDeviceTranspose,
                    stream);
  }

  pqCentroidsInnermostCode_ = std::move(pqDeviceTranspose);

  // Also maintain the PQ centroids in the form
  // (sub q)(code id)(sub dim)
  DeviceTensor<float, 3, true> pqCentroidsMiddleCode(
    resources_,
    makeDevAlloc(AllocType::Quantizer, stream),
    {numSubQuantizers_, numSubQuantizerCodes_, dimPerSubQuantizer_});

  runTransposeAny(pqCentroidsInnermostCode_, 1, 2, pqCentroidsMiddleCode,
                  stream);

  pqCentroidsMiddleCode_ = std::move(pqCentroidsMiddleCode);
}

template <typename CentroidT>
void
IVFPQ::precomputeCodesT_() {
  FAISS_ASSERT(metric_ == MetricType::METRIC_L2);

  auto stream = resources_->getDefaultStreamCurrentDevice();

  //
  //    d = || x - y_C ||^2 + || y_R ||^2 + 2 * (y_C|y_R) - 2 * (x|y_R)
  //        ---------------   ---------------------------       -------
  //            term 1                 term 2                   term 3
  //

  // Terms 1 and 3 are available only at query time. We compute term 2
  // here.

  // Compute 2 * (y_C|y_R) via batch matrix multiplication
  // batch size (sub q) x {(centroid id)(sub dim) x (code id)(sub dim)'}
  //         => (sub q) x {(centroid id)(code id)}
  //         => (sub q)(centroid id)(code id)

  // View (centroid id)(dim) as
  //      (centroid id)(sub q)(dim)
  // Transpose (centroid id)(sub q)(sub dim) to
  //           (sub q)(centroid id)(sub dim)
  auto& coarseCentroids = quantizer_->template getVectorsRef<CentroidT>();

  // Create the coarse PQ product
  DeviceTensor<float, 3, true> coarsePQProduct(
    resources_,
    makeTempAlloc(AllocType::QuantizerPrecomputedCodes, stream),
    {numSubQuantizers_, coarseCentroids.getSize(0), numSubQuantizerCodes_});

  {
    auto centroidView = coarseCentroids.template view<3>(
      {coarseCentroids.getSize(0), numSubQuantizers_, dimPerSubQuantizer_});

    // This is only needed temporarily
    DeviceTensor<CentroidT, 3, true> centroidsTransposed(
      resources_,
      makeTempAlloc(AllocType::QuantizerPrecomputedCodes, stream),
      {numSubQuantizers_, coarseCentroids.getSize(0), dimPerSubQuantizer_});

    runTransposeAny(centroidView, 0, 1, centroidsTransposed,
                    stream);

    runIteratedMatrixMult(coarsePQProduct, false,
                          centroidsTransposed, false,
                          pqCentroidsMiddleCode_, true,
                          2.0f, 0.0f,
                          resources_->getBlasHandleCurrentDevice(),
                          stream);
  }

  // Transpose (sub q)(centroid id)(code id) to
  //           (centroid id)(sub q)(code id)
  // This will become our precomputed code output
  DeviceTensor<float, 3, true> coarsePQProductTransposed(
    resources_,
    makeDevAlloc(AllocType::QuantizerPrecomputedCodes, stream),
    {coarseCentroids.getSize(0), numSubQuantizers_, numSubQuantizerCodes_});
  runTransposeAny(coarsePQProduct, 0, 1, coarsePQProductTransposed,
                  stream);

  // View (centroid id)(sub q)(code id) as
  //      (centroid id)(sub q * code id)
  auto coarsePQProductTransposedView = coarsePQProductTransposed.view<2>(
    {coarseCentroids.getSize(0), numSubQuantizers_ * numSubQuantizerCodes_});

  // Sum || y_R ||^2 + 2 * (y_C|y_R)
  // i.e., add norms                              (sub q * code id)
  // along columns of inner product  (centroid id)(sub q * code id)
  {
    // Compute ||y_R||^2 by treating
    // (sub q)(code id)(sub dim) as (sub q * code id)(sub dim)
    auto pqCentroidsMiddleCodeView =
      pqCentroidsMiddleCode_.view<2>(
        {numSubQuantizers_ * numSubQuantizerCodes_, dimPerSubQuantizer_});
    DeviceTensor<float, 1, true> subQuantizerNorms(
      resources_,
      makeTempAlloc(AllocType::QuantizerPrecomputedCodes, stream),
      {numSubQuantizers_ * numSubQuantizerCodes_});

    runL2Norm(pqCentroidsMiddleCodeView, true,
              subQuantizerNorms, true,
              stream);

    runSumAlongColumns(subQuantizerNorms, coarsePQProductTransposedView,
                       stream);
  }

  // We added into the view, so `coarsePQProductTransposed` is now our
  // precomputed term 2.
  if (useFloat16LookupTables_) {
    precomputedCodeHalf_ =
      DeviceTensor<half, 3, true>(
        resources_,
        makeDevAlloc(AllocType::QuantizerPrecomputedCodes, stream),
        {coarseCentroids.getSize(0), numSubQuantizers_, numSubQuantizerCodes_});

    convertTensor(stream, coarsePQProductTransposed, precomputedCodeHalf_);
  } else {
    precomputedCode_ = std::move(coarsePQProductTransposed);
  }
}

void
IVFPQ::precomputeCodes_() {
  if (quantizer_->getUseFloat16()) {
    precomputeCodesT_<half>();
  } else {
    precomputeCodesT_<float>();
  }
}

void
IVFPQ::query(Tensor<float, 2, true>& queries,
             int nprobe,
             int k,
             Tensor<float, 2, true>& outDistances,
             Tensor<long, 2, true>& outIndices) {
  // These are caught at a higher level
  FAISS_ASSERT(nprobe <= GPU_MAX_SELECTION_K);
  FAISS_ASSERT(k <= GPU_MAX_SELECTION_K);

  auto stream = resources_->getDefaultStreamCurrentDevice();
  nprobe = std::min(nprobe, quantizer_->getSize());

  FAISS_ASSERT(queries.getSize(1) == dim_);
  FAISS_ASSERT(outDistances.getSize(0) == queries.getSize(0));
  FAISS_ASSERT(outIndices.getSize(0) == queries.getSize(0));

  // Reserve space for the closest coarse centroids
  DeviceTensor<float, 2, true>
    coarseDistances(
      resources_, makeTempAlloc(AllocType::Other, stream),
      {queries.getSize(0), nprobe});
  DeviceTensor<int, 2, true>
    coarseIndices(
      resources_, makeTempAlloc(AllocType::Other, stream),
      {queries.getSize(0), nprobe});

  // Find the `nprobe` closest coarse centroids; we can use int
  // indices both internally and externally
  quantizer_->query(queries,
                    nprobe,
                    metric_,
                    metricArg_,
                    coarseDistances,
                    coarseIndices,
                    true);

  if (precomputedCodes_) {
    FAISS_ASSERT(metric_ == MetricType::METRIC_L2);

    runPQPrecomputedCodes_(queries,
                           coarseDistances,
                           coarseIndices,
                           k,
                           outDistances,
                           outIndices);
  } else {
    runPQNoPrecomputedCodes_(queries,
                             coarseDistances,
                             coarseIndices,
                             k,
                             outDistances,
                             outIndices);
  }

  // If the GPU isn't storing indices (they are on the CPU side), we
  // need to perform the re-mapping here
  // FIXME: we might ultimately be calling this function with inputs
  // from the CPU, these are unnecessary copies
  if (indicesOptions_ == INDICES_CPU) {
    HostTensor<long, 2, true> hostOutIndices(outIndices, stream);

    ivfOffsetToUserIndex(hostOutIndices.data(),
                         numLists_,
                         hostOutIndices.getSize(0),
                         hostOutIndices.getSize(1),
                         listOffsetToUserIndex_);

    // Copy back to GPU, since the input to this function is on the
    // GPU
    outIndices.copyFrom(hostOutIndices, stream);
  }
}

std::vector<unsigned char>
IVFPQ::getListCodes(int listId) const {
  FAISS_ASSERT(listId < deviceListData_.size());

  return deviceListData_[listId]->copyToHost<unsigned char>(
    resources_->getDefaultStreamCurrentDevice());
}

Tensor<float, 3, true>
IVFPQ::getPQCentroids() {
  return pqCentroidsMiddleCode_;
}

void
IVFPQ::runPQPrecomputedCodes_(
  Tensor<float, 2, true>& queries,
  DeviceTensor<float, 2, true>& coarseDistances,
  DeviceTensor<int, 2, true>& coarseIndices,
  int k,
  Tensor<float, 2, true>& outDistances,
  Tensor<long, 2, true>& outIndices) {
  FAISS_ASSERT(metric_ == MetricType::METRIC_L2);

  auto stream = resources_->getDefaultStreamCurrentDevice();

  // Compute precomputed code term 3, - 2 * (x|y_R)
  // This is done via batch MM
  // {sub q} x {(query id)(sub dim) * (code id)(sub dim)'} =>
  // {sub q} x {(query id)(code id)}
  DeviceTensor<float, 3, true> term3Transposed(
    resources_, makeTempAlloc(AllocType::Other, stream),
    {queries.getSize(0), numSubQuantizers_, numSubQuantizerCodes_});

  // These allocations within are only temporary, so release them when
  // we're done to maximize free space
  {
    auto querySubQuantizerView = queries.view<3>(
      {queries.getSize(0), numSubQuantizers_, dimPerSubQuantizer_});
    DeviceTensor<float, 3, true> queriesTransposed(
      resources_, makeTempAlloc(AllocType::Other, stream),
      {numSubQuantizers_, queries.getSize(0), dimPerSubQuantizer_});
    runTransposeAny(querySubQuantizerView, 0, 1, queriesTransposed, stream);

    DeviceTensor<float, 3, true> term3(
      resources_, makeTempAlloc(AllocType::Other, stream),
      {numSubQuantizers_, queries.getSize(0), numSubQuantizerCodes_});

    runIteratedMatrixMult(term3, false,
                          queriesTransposed, false,
                          pqCentroidsMiddleCode_, true,
                          -2.0f, 0.0f,
                          resources_->getBlasHandleCurrentDevice(),
                          stream);

    runTransposeAny(term3, 0, 1, term3Transposed, stream);
  }

  NoTypeTensor<3, true> term2;
  NoTypeTensor<3, true> term3;
  DeviceTensor<half, 3, true> term3Half;

  if (useFloat16LookupTables_) {
    term3Half =
      convertTensorTemporary<float, half, 3>(
        resources_, stream, term3Transposed);

    term2 = NoTypeTensor<3, true>(precomputedCodeHalf_);
    term3 = NoTypeTensor<3, true>(term3Half);
  } else {
    term2 = NoTypeTensor<3, true>(precomputedCode_);
    term3 = NoTypeTensor<3, true>(term3Transposed);
  }

  runPQScanMultiPassPrecomputed(queries,
                                coarseDistances, // term 1
                                term2, // term 2
                                term3, // term 3
                                coarseIndices,
                                useFloat16LookupTables_,
                                bytesPerVector_,
                                numSubQuantizers_,
                                numSubQuantizerCodes_,
                                deviceListDataPointers_,
                                deviceListIndexPointers_,
                                indicesOptions_,
                                deviceListLengths_,
                                maxListLength_,
                                k,
                                outDistances,
                                outIndices,
                                resources_);
}

template <typename CentroidT>
void
IVFPQ::runPQNoPrecomputedCodesT_(
  Tensor<float, 2, true>& queries,
  DeviceTensor<float, 2, true>& coarseDistances,
  DeviceTensor<int, 2, true>& coarseIndices,
  int k,
  Tensor<float, 2, true>& outDistances,
  Tensor<long, 2, true>& outIndices) {
  auto& coarseCentroids = quantizer_->template getVectorsRef<CentroidT>();

  runPQScanMultiPassNoPrecomputed(queries,
                                  coarseCentroids,
                                  pqCentroidsInnermostCode_,
                                  coarseIndices,
                                  useFloat16LookupTables_,
                                  bytesPerVector_,
                                  numSubQuantizers_,
                                  numSubQuantizerCodes_,
                                  deviceListDataPointers_,
                                  deviceListIndexPointers_,
                                  indicesOptions_,
                                  deviceListLengths_,
                                  maxListLength_,
                                  k,
                                  metric_,
                                  outDistances,
                                  outIndices,
                                  resources_);
}

void
IVFPQ::runPQNoPrecomputedCodes_(
  Tensor<float, 2, true>& queries,
  DeviceTensor<float, 2, true>& coarseDistances,
  DeviceTensor<int, 2, true>& coarseIndices,
  int k,
  Tensor<float, 2, true>& outDistances,
  Tensor<long, 2, true>& outIndices) {
  if (quantizer_->getUseFloat16()) {
    runPQNoPrecomputedCodesT_<half>(queries,
                                    coarseDistances,
                                    coarseIndices,
                                    k,
                                    outDistances,
                                    outIndices);
  } else {
    runPQNoPrecomputedCodesT_<float>(queries,
                                     coarseDistances,
                                     coarseIndices,
                                     k,
                                     outDistances,
                                     outIndices);
  }
}

} } // namespace
