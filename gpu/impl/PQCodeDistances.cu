#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */


#include "PQCodeDistances.cuh"

#include "BroadcastSum.cuh"
#include "Distance.cuh"
#include "L2Norm.cuh"
#include "../utils/DeviceDefs.cuh"
#include "../utils/DeviceUtils.h"
#include "../utils/Float16.cuh"
#include "../utils/MatrixMult.cuh"
#include "../utils/PtxUtils.cuh"
#include "../utils/StaticUtils.h"
#include "../utils/Transpose.cuh"

namespace faiss { namespace gpu {

template <typename T>
struct Converter {
};

#ifdef FAISS_USE_FLOAT16
template <>
struct Converter<half> {
  inline static __device__ half to(float v) { return __float2half(v); }
};
#endif

template <>
struct Converter<float> {
  inline static __device__ float to(float v) { return v; }
};

// Kernel responsible for calculating distance from residual vector to
// each product quantizer code centroid
template <typename OutCodeT, int DimsPerSubQuantizer>
__global__ void
__launch_bounds__(288, 4)
pqCodeDistances(Tensor<float, 2, true> queries,
                int queriesPerBlock,
                Tensor<float, 2, true> coarseCentroids,
                Tensor<float, 3, true> pqCentroids,
                Tensor<int, 2, true> topQueryToCentroid,
                // (query id)(coarse)(subquantizer)(code) -> dist
                Tensor<OutCodeT, 4, true> outCodeDistances) {
  const auto numSubQuantizers = pqCentroids.getSize(0);
  const auto dimsPerSubQuantizer = pqCentroids.getSize(1);
  assert(DimsPerSubQuantizer == dimsPerSubQuantizer);
  const auto codesPerSubQuantizer = pqCentroids.getSize(2);

  bool isLoadingThread = threadIdx.x >= codesPerSubQuantizer;
  int loadingThreadId = threadIdx.x - codesPerSubQuantizer;

  extern __shared__ float smem[];

  // Each thread calculates a single code
  float subQuantizerData[DimsPerSubQuantizer];

  auto code = threadIdx.x;
  auto subQuantizer = blockIdx.y;

  // Each thread will load the pq centroid data for the code that it
  // is processing
#pragma unroll
  for (int i = 0; i < DimsPerSubQuantizer; ++i) {
    subQuantizerData[i] = pqCentroids[subQuantizer][i][code].ldg();
  }

  // Where we store our query vector
  float* smemQuery = smem;

  // Where we store our residual vector; this is double buffered so we
  // can be loading the next one while processing the current one
  float* smemResidual1 = &smemQuery[DimsPerSubQuantizer];
  float* smemResidual2 = &smemResidual1[DimsPerSubQuantizer];

  // Where we pre-load the coarse centroid IDs
  int* coarseIds = (int*) &smemResidual2[DimsPerSubQuantizer];

  // Each thread is calculating the distance for a single code,
  // performing the reductions locally

  // Handle multiple queries per block
  auto startQueryId = blockIdx.x * queriesPerBlock;
  auto numQueries = queries.getSize(0) - startQueryId;
  if (numQueries > queriesPerBlock) {
    numQueries = queriesPerBlock;
  }

  for (int query = 0; query < numQueries; ++query) {
    auto queryId = startQueryId + query;

    auto querySubQuantizer =
      queries[queryId][subQuantizer * DimsPerSubQuantizer].data();

    // Load current query vector
    for (int i = threadIdx.x; i < DimsPerSubQuantizer; i += blockDim.x) {
      smemQuery[i] = querySubQuantizer[i];
    }

    // Load list of coarse centroids found
    for (int i = threadIdx.x;
         i < topQueryToCentroid.getSize(1); i += blockDim.x) {
      coarseIds[i] = topQueryToCentroid[queryId][i];
    }

    // We need coarseIds below
    // FIXME: investigate loading separately, so we don't need this
    __syncthreads();

    // Preload first buffer of residual data
    if (isLoadingThread) {
      for (int i = loadingThreadId;
           i < DimsPerSubQuantizer;
           i += blockDim.x - codesPerSubQuantizer) {
        auto coarseId = coarseIds[0];
        // In case NaNs were in the original query data
        coarseId = coarseId == -1 ? 0 : coarseId;
        auto coarseCentroidSubQuantizer =
          coarseCentroids[coarseId][subQuantizer * dimsPerSubQuantizer].data();

        smemResidual1[i] = smemQuery[i] - coarseCentroidSubQuantizer[i];
      }
    }

    // The block walks the list for a single query
    for (int coarse = 0; coarse < topQueryToCentroid.getSize(1); ++coarse) {
      // Wait for smemResidual1 to be loaded
      __syncthreads();

      if (isLoadingThread) {
        // Preload second buffer of residual data
        for (int i = loadingThreadId;
             i < DimsPerSubQuantizer;
             i += blockDim.x - codesPerSubQuantizer) {
          // FIXME: try always making this centroid id 0 so we can
          // terminate
          if (coarse != (topQueryToCentroid.getSize(1) - 1)) {
            auto coarseId = coarseIds[coarse + 1];
            // In case NaNs were in the original query data
            coarseId = coarseId == -1 ? 0 : coarseId;

            auto coarseCentroidSubQuantizer =
              coarseCentroids[coarseId][subQuantizer * dimsPerSubQuantizer].data();

            smemResidual2[i] = smemQuery[i] - coarseCentroidSubQuantizer[i];
          }
        }
      } else {
        // These are the processing threads
        float dist = 0.0f;

        constexpr int kUnroll = 4;
        constexpr int kRemainder = DimsPerSubQuantizer % kUnroll;
        constexpr int kRemainderBase = DimsPerSubQuantizer - kRemainder;
        float vals[kUnroll];

        // Calculate residual - pqCentroid for each dim that we're
        // processing

        // Unrolled loop
#pragma unroll
        for (int i = 0; i < DimsPerSubQuantizer / kUnroll; ++i) {

#pragma unroll
          for (int j = 0; j < kUnroll; ++j) {
            vals[j] = smemResidual1[i * kUnroll + j];
          }

#pragma unroll
          for (int j = 0; j < kUnroll; ++j) {
            vals[j] -= subQuantizerData[i * kUnroll + j];
          }

#pragma unroll
          for (int j = 0; j < kUnroll; ++j) {
            vals[j] *= vals[j];
          }

#pragma unroll
          for (int j = 0; j < kUnroll; ++j) {
            dist += vals[j];
          }
        }

        // Remainder loop
#pragma unroll
        for (int j = 0; j < kRemainder; ++j) {
          vals[j] = smemResidual1[kRemainderBase + j];
        }

#pragma unroll
        for (int j = 0; j < kRemainder; ++j) {
          vals[j] -= subQuantizerData[kRemainderBase + j];
        }

#pragma unroll
        for (int j = 0; j < kRemainder; ++j) {
          vals[j] *= vals[j];
        }

#pragma unroll
        for (int j = 0; j < kRemainder; ++j) {
          dist += vals[j];
        }

        // We have the distance for our code; write it out
        outCodeDistances[queryId][coarse][subQuantizer][code] =
          Converter<OutCodeT>::to(dist);
      } // !isLoadingThread

      // Swap residual buffers
      float* tmp = smemResidual1;
      smemResidual1 = smemResidual2;
      smemResidual2 = tmp;
    }
  }
}

__global__ void
residualVector(Tensor<float, 2, true> queries,
               Tensor<float, 2, true> coarseCentroids,
               Tensor<int, 2, true> topQueryToCentroid,
               int numSubDim,
               // output is transposed:
               // (sub q)(query id)(centroid id)(sub dim)
               Tensor<float, 4, true> residual) {
  // block x is query id
  // block y is centroid id
  // thread x is dim
  auto queryId = blockIdx.x;
  auto centroidId = blockIdx.y;

  int realCentroidId = topQueryToCentroid[queryId][centroidId];

  for (int dim = threadIdx.x; dim < queries.getSize(1); dim += blockDim.x) {
    float q = queries[queryId][dim];
    float c = coarseCentroids[realCentroidId][dim];

    residual[dim / numSubDim][queryId][centroidId][dim % numSubDim] =
      q - c;
  }
}

void
runResidualVector(Tensor<float, 3, true>& pqCentroids,
                  Tensor<float, 2, true>& queries,
                  Tensor<float, 2, true>& coarseCentroids,
                  Tensor<int, 2, true>& topQueryToCentroid,
                  Tensor<float, 4, true>& residual,
                  hipStream_t stream) {
  auto grid =
    dim3(topQueryToCentroid.getSize(0), topQueryToCentroid.getSize(1));
  auto block = dim3(std::min(queries.getSize(1), getMaxThreadsCurrentDevice()));

  residualVector<<<grid, block, 0, stream>>>(
    queries, coarseCentroids, topQueryToCentroid, pqCentroids.getSize(1),
    residual);

  CUDA_TEST_ERROR();
}

void
runPQCodeDistancesMM(Tensor<float, 3, true>& pqCentroids,
                     Tensor<float, 2, true>& queries,
                     Tensor<float, 2, true>& coarseCentroids,
                     Tensor<int, 2, true>& topQueryToCentroid,
                     NoTypeTensor<4, true>& outCodeDistances,
                     bool useFloat16Lookup,
                     DeviceMemory& mem,
                     hipblasHandle_t handle,
                     hipStream_t stream) {
  // Calculate (q - c) residual vector
  // (sub q)(query id)(centroid id)(sub dim)
  DeviceTensor<float, 4, true> residual(
    mem,
    {pqCentroids.getSize(0),
        topQueryToCentroid.getSize(0),
        topQueryToCentroid.getSize(1),
        pqCentroids.getSize(1)},
    stream);

  runResidualVector(pqCentroids, queries,
                    coarseCentroids, topQueryToCentroid,
                    residual, stream);

  // Calculate ||q - c||^2
  DeviceTensor<float, 1, true> residualNorms(
    mem,
    {pqCentroids.getSize(0) *
        topQueryToCentroid.getSize(0) *
        topQueryToCentroid.getSize(1)},
    stream);

  auto residualView2 = residual.view<2>(
    {pqCentroids.getSize(0) *
        topQueryToCentroid.getSize(0) *
        topQueryToCentroid.getSize(1),
        pqCentroids.getSize(1)});

  runL2Norm(residualView2, true, residualNorms, true, stream);

  // Perform a batch MM:
  // (sub q) x {(q * c)(sub dim) x (sub dim)(code)} =>
  // (sub q) x {(q * c)(code)}
  auto residualView3 = residual.view<3>(
    {pqCentroids.getSize(0),
        topQueryToCentroid.getSize(0) * topQueryToCentroid.getSize(1),
        pqCentroids.getSize(1)});

  DeviceTensor<float, 3, true> residualDistance(
    mem,
    {pqCentroids.getSize(0),
        topQueryToCentroid.getSize(0) * topQueryToCentroid.getSize(1),
        pqCentroids.getSize(2)},
    stream);

  runIteratedMatrixMult(residualDistance, false,
                        residualView3, false,
                        pqCentroids, false,
                        -2.0f, 0.0f,
                        handle,
                        stream);

  // Sum ||q - c||^2 along rows
  auto residualDistanceView2 = residualDistance.view<2>(
    {pqCentroids.getSize(0) *
        topQueryToCentroid.getSize(0) *
        topQueryToCentroid.getSize(1),
        pqCentroids.getSize(2)});

  runSumAlongRows(residualNorms, residualDistanceView2, false, stream);

  Tensor<float, 4, true> outCodeDistancesF;
  DeviceTensor<float, 4, true> outCodeDistancesFloatMem;

#ifdef FAISS_USE_FLOAT16
  if (useFloat16Lookup) {
    outCodeDistancesFloatMem = DeviceTensor<float, 4, true>(
      mem, {outCodeDistances.getSize(0),
          outCodeDistances.getSize(1),
          outCodeDistances.getSize(2),
          outCodeDistances.getSize(3)},
      stream);

    outCodeDistancesF = outCodeDistancesFloatMem;
  }
#endif

  if (!useFloat16Lookup) {
    outCodeDistancesF = outCodeDistances.toTensor<float>();
  }

  // Transpose -2(sub q)(q * c)(code) to -2(q * c)(sub q)(code) (which
  // is where we build our output distances)
  auto outCodeDistancesView = outCodeDistancesF.view<3>(
    {topQueryToCentroid.getSize(0) * topQueryToCentroid.getSize(1),
        outCodeDistances.getSize(2),
        outCodeDistances.getSize(3)});

  runTransposeAny(residualDistance, 0, 1, outCodeDistancesView, stream);

  // Calculate code norms per each sub-dim
  // (sub q)(sub dim)(code) is pqCentroids
  // transpose to (sub q)(code)(sub dim)
  DeviceTensor<float, 3, true> pqCentroidsTranspose(
    mem,
    {pqCentroids.getSize(0), pqCentroids.getSize(2), pqCentroids.getSize(1)},
    stream);

  runTransposeAny(pqCentroids, 1, 2, pqCentroidsTranspose, stream);

  auto pqCentroidsTransposeView = pqCentroidsTranspose.view<2>(
    {pqCentroids.getSize(0) * pqCentroids.getSize(2),
        pqCentroids.getSize(1)});

  DeviceTensor<float, 1, true> pqCentroidsNorm(
    mem,
    {pqCentroids.getSize(0) * pqCentroids.getSize(2)},
    stream);

  runL2Norm(pqCentroidsTransposeView, true, pqCentroidsNorm, true, stream);

  // View output as (q * c)(sub q * code), and add centroid norm to
  // each row
  auto outDistancesCodeViewCols = outCodeDistancesView.view<2>(
    {topQueryToCentroid.getSize(0) * topQueryToCentroid.getSize(1),
        outCodeDistances.getSize(2) * outCodeDistances.getSize(3)});

  runSumAlongColumns(pqCentroidsNorm, outDistancesCodeViewCols, stream);

#ifdef FAISS_USE_FLOAT16
  if (useFloat16Lookup) {
    // Need to convert back
    auto outCodeDistancesH = outCodeDistances.toTensor<half>();
    toHalf(stream, outCodeDistancesF, outCodeDistancesH);
  }
#endif
}

void
runPQCodeDistances(Tensor<float, 3, true>& pqCentroids,
                   Tensor<float, 2, true>& queries,
                   Tensor<float, 2, true>& coarseCentroids,
                   Tensor<int, 2, true>& topQueryToCentroid,
                   NoTypeTensor<4, true>& outCodeDistances,
                   bool useFloat16Lookup,
                   hipStream_t stream) {
  const auto numSubQuantizers = pqCentroids.getSize(0);
  const auto dimsPerSubQuantizer = pqCentroids.getSize(1);
  const auto codesPerSubQuantizer = pqCentroids.getSize(2);

  // FIXME: tune
  // Reuse of pq centroid data is based on both # of queries * nprobe,
  // and we should really be tiling in both dimensions
  constexpr int kQueriesPerBlock = 8;

  auto grid = dim3(utils::divUp(queries.getSize(0), kQueriesPerBlock),
                   numSubQuantizers);

  // Reserve one block of threads for double buffering
  // FIXME: probably impractical for large # of dims?
  auto loadingThreads = utils::roundUp(dimsPerSubQuantizer, kWarpSize);
  auto block = dim3(codesPerSubQuantizer + loadingThreads);

  auto smem = (3 * dimsPerSubQuantizer) * sizeof(float)
    + topQueryToCentroid.getSize(1) * sizeof(int);

#ifdef FAISS_USE_FLOAT16
#define CODE_DISTANCE(DIMS)                                             \
  do {                                                                  \
    if (useFloat16Lookup) {                                             \
      auto outCodeDistancesT = outCodeDistances.toTensor<half>();       \
                                                                        \
      pqCodeDistances<half, DIMS><<<grid, block, smem, stream>>>(       \
        queries, kQueriesPerBlock,                                      \
        coarseCentroids, pqCentroids,                                   \
        topQueryToCentroid, outCodeDistancesT);                         \
    } else {                                                            \
      auto outCodeDistancesT = outCodeDistances.toTensor<float>();      \
                                                                        \
      pqCodeDistances<float, DIMS><<<grid, block, smem, stream>>>(      \
        queries, kQueriesPerBlock,                                      \
        coarseCentroids, pqCentroids,                                   \
        topQueryToCentroid, outCodeDistancesT);                         \
    }                                                                   \
  } while (0)
#else
#define CODE_DISTANCE(DIMS)                                             \
  do {                                                                  \
    if (!useFloat16Lookup) {                                            \
      auto outCodeDistancesT = outCodeDistances.toTensor<float>();      \
                                                                        \
      pqCodeDistances<float, DIMS><<<grid, block, smem, stream>>>(      \
        queries, kQueriesPerBlock,                                      \
        coarseCentroids, pqCentroids,                                   \
        topQueryToCentroid, outCodeDistancesT);                         \
    }                                                                   \
  } while (0)
#endif

  switch (dimsPerSubQuantizer) {
    case 1:
      CODE_DISTANCE(1);
      break;
    case 2:
      CODE_DISTANCE(2);
      break;
    case 3:
      CODE_DISTANCE(3);
      break;
    case 4:
      CODE_DISTANCE(4);
      break;
    case 6:
      CODE_DISTANCE(6);
      break;
    case 8:
      CODE_DISTANCE(8);
      break;
    case 10:
      CODE_DISTANCE(10);
      break;
    case 12:
      CODE_DISTANCE(12);
      break;
    case 16:
      CODE_DISTANCE(16);
      break;
    case 20:
      CODE_DISTANCE(20);
      break;
    case 24:
      CODE_DISTANCE(24);
      break;
    case 28:
      CODE_DISTANCE(28);
      break;
    case 32:
      CODE_DISTANCE(32);
      break;
      // FIXME: larger sizes require too many registers - we need the
      // MM implementation working
    default:
      FAISS_ASSERT(false);
      break;
  }

#undef CODE_DISTANCE

  CUDA_TEST_ERROR();
}

} } // namespace
