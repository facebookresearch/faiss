#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */


#include "Distance.cuh"
#include "BroadcastSum.cuh"
#include "L2Norm.cuh"
#include "L2Select.cuh"
#include "../../FaissAssert.h"
#include "../../AuxIndexStructures.h"
#include "../GpuResources.h"
#include "../utils/DeviceDefs.cuh"
#include "../utils/DeviceUtils.h"
#include "../utils/Limits.cuh"
#include "../utils/MatrixMult.cuh"
#include "../utils/BlockSelectKernel.cuh"

#include <memory>
#include <algorithm>
#include <thrust/fill.h>
#include <thrust/for_each.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

namespace faiss { namespace gpu {

namespace {

template <typename T>
Tensor<T, 2, true> sliceCentroids(Tensor<T, 2, true>& centroids,
                                  bool centroidsRowMajor,
                                  int startCentroid,
                                  int num) {
  // Row major is (num, dim)
  // Col major is (dim, num)
  if (startCentroid == 0 &&
      num == centroids.getSize(centroidsRowMajor ? 0 : 1)) {
    return centroids;
  }

  return centroids.narrow(centroidsRowMajor ? 0 : 1, startCentroid, num);
}

// For each chunk of k indices, increment the index by chunk * increment
template <typename T>
__global__ void incrementIndex(Tensor<T, 2, true> indices,
                               int k,
                               int increment) {
  for (int i = threadIdx.x; i < k; i += blockDim.x) {
    indices[blockIdx.y][blockIdx.x * k + i] += blockIdx.x * increment;
  }
}

// Used to update result indices in distance computation where the number of
// centroids is high, and is tiled
template <typename T>
void runIncrementIndex(Tensor<T, 2, true>& indices,
                       int k,
                       int increment,
                       hipStream_t stream) {
  dim3 grid(indices.getSize(1) / k, indices.getSize(0));
  int block = std::min(k, 512);

  // should be exact
  FAISS_ASSERT(grid.x * k == indices.getSize(1));

  incrementIndex<<<grid, block, 0, stream>>>(indices, k, increment);

  hipDeviceSynchronize();
}

// If the inner size (dim) of the vectors is small, we want a larger query tile
// size, like 1024

void chooseTileSize(int numQueries,
                    int numCentroids,
                    int dim,
                    int elementSize,
                    size_t tempMemAvailable,
                    int& tileRows,
                    int& tileCols) {
  // The matrix multiplication should be large enough to be efficient, but if it
  // is too large, we seem to lose efficiency as opposed to double-streaming.
  // Each tile size here defines 1/2 of the memory use due to double streaming.
  // We ignore available temporary memory, as that is adjusted independently by
  // the user and can thus meet these requirements (or not).
  // For <= 4 GB GPUs, prefer 512 MB of usage.
  // For <= 8 GB GPUs, prefer 768 MB of usage.
  // Otherwise, prefer 1 GB of usage.
  auto totalMem = getCurrentDeviceProperties().totalGlobalMem;

  int targetUsage = 0;

  if (totalMem <= ((size_t) 4) * 1024 * 1024 * 1024) {
    targetUsage = 512 * 1024 * 1024;
  } else if (totalMem <= ((size_t) 8) * 1024 * 1024 * 1024) {
    targetUsage = 768 * 1024 * 1024;
  } else {
    targetUsage = 1024 * 1024 * 1024;
  }

  targetUsage /= 2 * elementSize;

  // 512 seems to be a batch size sweetspot for float32.
  // If we are on float16, increase to 512.
  // If the k size (vec dim) of the matrix multiplication is small (<= 32),
  // increase to 1024.
  int preferredTileRows = 512;
  if (dim <= 32) {
    preferredTileRows = 1024;
  }

  tileRows = std::min(preferredTileRows, numQueries);

  // tileCols is the remainder size
  tileCols = std::min(targetUsage / preferredTileRows, numCentroids);
}

}

template <typename T>
void runDistance(bool computeL2,
                 GpuResources* resources,
                 Tensor<T, 2, true>& centroids,
                 bool centroidsRowMajor,
                 Tensor<T, 1, true>* centroidNorms,
                 Tensor<T, 2, true>& queries,
                 bool queriesRowMajor,
                 int k,
                 Tensor<T, 2, true>& outDistances,
                 Tensor<int, 2, true>& outIndices,
                 bool useHgemm,
                 bool ignoreOutDistances) {
  // The # of centroids in `centroids` based on memory layout
  auto numCentroids = centroids.getSize(centroidsRowMajor ? 0 : 1);

  // The # of queries in `queries` based on memory layout
  auto numQueries = queries.getSize(queriesRowMajor ? 0 : 1);

  // The dimensions of the vectors to consider
  auto dim = queries.getSize(queriesRowMajor ? 1 : 0);
  FAISS_ASSERT((numQueries == 0 || numCentroids == 0) ||
               dim == centroids.getSize(centroidsRowMajor ? 1 : 0));

  FAISS_ASSERT(outDistances.getSize(0) == numQueries);
  FAISS_ASSERT(outIndices.getSize(0) == numQueries);
  FAISS_ASSERT(outDistances.getSize(1) == k);
  FAISS_ASSERT(outIndices.getSize(1) == k);

  auto& mem = resources->getMemoryManagerCurrentDevice();
  auto defaultStream = resources->getDefaultStreamCurrentDevice();

  // If we're quering against a 0 sized set, just return empty results
  if (centroids.numElements() == 0) {
    thrust::fill(thrust::cuda::par.on(defaultStream),
                 outDistances.data(), outDistances.end(),
                 Limits<T>::getMax());

    thrust::fill(thrust::cuda::par.on(defaultStream),
                 outIndices.data(), outIndices.end(),
                 -1);

    return;
  }

  // L2: If ||c||^2 is not pre-computed, calculate it
  DeviceTensor<T, 1, true> cNorms;
  if (computeL2 && !centroidNorms) {
    cNorms =
      std::move(DeviceTensor<T, 1, true>(mem,
                                         {numCentroids}, defaultStream));
    runL2Norm(centroids, centroidsRowMajor, cNorms, true, defaultStream);
    centroidNorms = &cNorms;
  }

  //
  // Prepare norm vector ||q||^2; ||c||^2 is already pre-computed
  //
  int qNormSize[1] = {numQueries};
  DeviceTensor<T, 1, true> queryNorms(mem, qNormSize, defaultStream);

  // ||q||^2
  if (computeL2) {
    runL2Norm(queries, queriesRowMajor, queryNorms, true, defaultStream);
  }

  // By default, aim to use up to 512 MB of memory for the processing, with both
  // number of queries and number of centroids being at least 512.
  int tileRows = 0;
  int tileCols = 0;
  chooseTileSize(numQueries,
                 numCentroids,
                 dim,
                 sizeof(T),
                 mem.getSizeAvailable(),
                 tileRows,
                 tileCols);

  int numColTiles = utils::divUp(numCentroids, tileCols);

  // We can have any number of vectors to query against, even less than k, in
  // which case we'll return -1 for the index
  FAISS_ASSERT(k <= GPU_MAX_SELECTION_K); // select limitation

  // Temporary output memory space we'll use
  DeviceTensor<T, 2, true> distanceBuf1(
    mem, {tileRows, tileCols}, defaultStream);
  DeviceTensor<T, 2, true> distanceBuf2(
    mem, {tileRows, tileCols}, defaultStream);
  DeviceTensor<T, 2, true>* distanceBufs[2] =
    {&distanceBuf1, &distanceBuf2};

  DeviceTensor<T, 2, true> outDistanceBuf1(
    mem, {tileRows, numColTiles * k}, defaultStream);
  DeviceTensor<T, 2, true> outDistanceBuf2(
    mem, {tileRows, numColTiles * k}, defaultStream);
  DeviceTensor<T, 2, true>* outDistanceBufs[2] =
    {&outDistanceBuf1, &outDistanceBuf2};

  DeviceTensor<int, 2, true> outIndexBuf1(
    mem, {tileRows, numColTiles * k}, defaultStream);
  DeviceTensor<int, 2, true> outIndexBuf2(
    mem, {tileRows, numColTiles * k}, defaultStream);
  DeviceTensor<int, 2, true>* outIndexBufs[2] =
    {&outIndexBuf1, &outIndexBuf2};

  auto streams = resources->getAlternateStreamsCurrentDevice();
  streamWait(streams, {defaultStream});

  int curStream = 0;
  bool interrupt = false;

  // Tile over the input queries
  for (int i = 0; i < numQueries; i += tileRows) {
    if (interrupt || InterruptCallback::is_interrupted()) {
      interrupt = true;
      break;
    }

    int curQuerySize = std::min(tileRows, numQueries - i);

    auto outDistanceView =
      outDistances.narrow(0, i, curQuerySize);
    auto outIndexView =
      outIndices.narrow(0, i, curQuerySize);

    auto queryView =
      queries.narrow(queriesRowMajor ? 0 : 1, i, curQuerySize);
    auto queryNormNiew =
      queryNorms.narrow(0, i, curQuerySize);

    auto outDistanceBufRowView =
      outDistanceBufs[curStream]->narrow(0, 0, curQuerySize);
    auto outIndexBufRowView =
      outIndexBufs[curStream]->narrow(0, 0, curQuerySize);

    // Tile over the centroids
    for (int j = 0; j < numCentroids; j += tileCols) {
      if (InterruptCallback::is_interrupted()) {
        interrupt = true;
        break;
      }

      int curCentroidSize = std::min(tileCols, numCentroids - j);
      int curColTile = j / tileCols;

      auto centroidsView =
        sliceCentroids(centroids, centroidsRowMajor, j, curCentroidSize);

      auto distanceBufView = distanceBufs[curStream]->
        narrow(0, 0, curQuerySize).narrow(1, 0, curCentroidSize);

      auto outDistanceBufColView =
        outDistanceBufRowView.narrow(1, k * curColTile, k);
      auto outIndexBufColView =
        outIndexBufRowView.narrow(1, k * curColTile, k);

      // L2: distance is ||c||^2 - 2qc + ||q||^2, we compute -2qc
      // IP: just compute qc
      // (query id x dim) x (centroid id, dim)' = (query id, centroid id)
      runMatrixMult(distanceBufView,
                    false, // not transposed
                    queryView,
                    !queriesRowMajor, // transposed MM if col major
                    centroidsView,
                    centroidsRowMajor, // transposed MM if row major
                    computeL2 ? -2.0f : 1.0f,
                    0.0f,
                    useHgemm,
                    resources->getBlasHandleCurrentDevice(),
                    streams[curStream]);

      if (computeL2) {
        // For L2 distance, we use this fused kernel that performs both
        // adding ||c||^2 to -2qc and k-selection, so we only need two
        // passes (one write by the gemm, one read here) over the huge
        // region of output memory
        //
        // If we aren't tiling along the number of centroids, we can perform the
        // output work directly
        if (tileCols == numCentroids) {
          // Write into the final output
          runL2SelectMin(distanceBufView,
                         *centroidNorms,
                         outDistanceView,
                         outIndexView,
                         k,
                         streams[curStream]);

          if (!ignoreOutDistances) {
            // expand (query id) to (query id, k) by duplicating along rows
            // top-k ||c||^2 - 2qc + ||q||^2 in the form (query id, k)
            runSumAlongRows(queryNormNiew,
                            outDistanceView,
                            true, // L2 distances should not go below zero due
                                  // to roundoff error
                            streams[curStream]);
          }
        } else {
          auto centroidNormsView = centroidNorms->narrow(0, j, curCentroidSize);

          // Write into our intermediate output
          runL2SelectMin(distanceBufView,
                         centroidNormsView,
                         outDistanceBufColView,
                         outIndexBufColView,
                         k,
                         streams[curStream]);

          if (!ignoreOutDistances) {
            // expand (query id) to (query id, k) by duplicating along rows
            // top-k ||c||^2 - 2qc + ||q||^2 in the form (query id, k)
            runSumAlongRows(queryNormNiew,
                            outDistanceBufColView,
                            true, // L2 distances should not go below zero due
                                  // to roundoff error
                            streams[curStream]);
          }
        }
      } else {
        // For IP, just k-select the output for this tile
        if (tileCols == numCentroids) {
          // Write into the final output
          runBlockSelect(distanceBufView,
                         outDistanceView,
                         outIndexView,
                         true, k, streams[curStream]);
        } else {
          // Write into the intermediate output
          runBlockSelect(distanceBufView,
                         outDistanceBufColView,
                         outIndexBufColView,
                         true, k, streams[curStream]);
        }
      }
    }

    // As we're finished with processing a full set of centroids, perform the
    // final k-selection
    if (tileCols != numCentroids) {
      // The indices are tile-relative; for each tile of k, we need to add
      // tileCols to the index
      runIncrementIndex(outIndexBufRowView, k, tileCols, streams[curStream]);

      runBlockSelectPair(outDistanceBufRowView,
                         outIndexBufRowView,
                         outDistanceView,
                         outIndexView,
                         computeL2 ? false : true, k, streams[curStream]);
    }

    curStream = (curStream + 1) % 2;
  }

  // Have the desired ordering stream wait on the multi-stream
  streamWait({defaultStream}, streams);

  if (interrupt) {
    FAISS_THROW_MSG("interrupted");
  }
}

template <typename T>
void runL2Distance(GpuResources* resources,
                   Tensor<T, 2, true>& centroids,
                   bool centroidsRowMajor,
                   Tensor<T, 1, true>* centroidNorms,
                   Tensor<T, 2, true>& queries,
                   bool queriesRowMajor,
                   int k,
                   Tensor<T, 2, true>& outDistances,
                   Tensor<int, 2, true>& outIndices,
                   bool useHgemm,
                   bool ignoreOutDistances = false) {
  runDistance<T>(true, // L2
                 resources,
                 centroids,
                 centroidsRowMajor,
                 centroidNorms,
                 queries,
                 queriesRowMajor,
                 k,
                 outDistances,
                 outIndices,
                 useHgemm,
                 ignoreOutDistances);
}

template <typename T>
void runIPDistance(GpuResources* resources,
                   Tensor<T, 2, true>& centroids,
                   bool centroidsRowMajor,
                   Tensor<T, 2, true>& queries,
                   bool queriesRowMajor,
                   int k,
                   Tensor<T, 2, true>& outDistances,
                   Tensor<int, 2, true>& outIndices,
                   bool useHgemm) {
  runDistance<T>(false, // IP
                 resources,
                 centroids,
                 centroidsRowMajor,
                 nullptr, // no centroid norms provided
                 queries,
                 queriesRowMajor,
                 k,
                 outDistances,
                 outIndices,
                 useHgemm,
                 false);
}

//
// Instantiations of the distance templates
//

void
runIPDistance(GpuResources* resources,
              Tensor<float, 2, true>& vectors,
              bool vectorsRowMajor,
              Tensor<float, 2, true>& queries,
              bool queriesRowMajor,
              int k,
              Tensor<float, 2, true>& outDistances,
              Tensor<int, 2, true>& outIndices) {
  runIPDistance<float>(resources,
                       vectors,
                       vectorsRowMajor,
                       queries,
                       queriesRowMajor,
                       k,
                       outDistances,
                       outIndices,
                       false);
}

#ifdef FAISS_USE_FLOAT16
void
runIPDistance(GpuResources* resources,
              Tensor<half, 2, true>& vectors,
              bool vectorsRowMajor,
              Tensor<half, 2, true>& queries,
              bool queriesRowMajor,
              int k,
              Tensor<half, 2, true>& outDistances,
              Tensor<int, 2, true>& outIndices,
              bool useHgemm) {
  runIPDistance<half>(resources,
                      vectors,
                      vectorsRowMajor,
                      queries,
                      queriesRowMajor,
                      k,
                      outDistances,
                      outIndices,
                      useHgemm);
}
#endif

void
runL2Distance(GpuResources* resources,
              Tensor<float, 2, true>& vectors,
              bool vectorsRowMajor,
              Tensor<float, 1, true>* vectorNorms,
              Tensor<float, 2, true>& queries,
              bool queriesRowMajor,
              int k,
              Tensor<float, 2, true>& outDistances,
              Tensor<int, 2, true>& outIndices,
              bool ignoreOutDistances) {
  runL2Distance<float>(resources,
                       vectors,
                       vectorsRowMajor,
                       vectorNorms,
                       queries,
                       queriesRowMajor,
                       k,
                       outDistances,
                       outIndices,
                       false,
                       ignoreOutDistances);
}

#ifdef FAISS_USE_FLOAT16
void
runL2Distance(GpuResources* resources,
              Tensor<half, 2, true>& vectors,
              bool vectorsRowMajor,
              Tensor<half, 1, true>* vectorNorms,
              Tensor<half, 2, true>& queries,
              bool queriesRowMajor,
              int k,
              Tensor<half, 2, true>& outDistances,
              Tensor<int, 2, true>& outIndices,
              bool useHgemm,
              bool ignoreOutDistances) {
  runL2Distance<half>(resources,
                      vectors,
                      vectorsRowMajor,
                      vectorNorms,
                      queries,
                      queriesRowMajor,
                      k,
                      outDistances,
                      outIndices,
                      useHgemm,
                      ignoreOutDistances);
}
#endif

} } // namespace
