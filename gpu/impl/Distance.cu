#include "hip/hip_runtime.h"

/**
 * Copyright (c) 2015-present, Facebook, Inc.
 * All rights reserved.
 *
 * This source code is licensed under the CC-by-NC license found in the
 * LICENSE file in the root directory of this source tree.
 */

// Copyright 2004-present Facebook. All Rights Reserved.

#include "Distance.cuh"
#include "BroadcastSum.cuh"
#include "L2Norm.cuh"
#include "L2Select.cuh"
#include "../../FaissAssert.h"
#include "../GpuResources.h"
#include "../utils/DeviceUtils.h"
#include "../utils/Limits.cuh"
#include "../utils/MatrixMult.cuh"
#include "../utils/BlockSelectKernel.cuh"

#include <memory>
#include <thrust/fill.h>
#include <thrust/for_each.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

namespace faiss { namespace gpu {

constexpr int kDefaultTileSize = 256;

template <typename T>
void runL2Distance(GpuResources* resources,
                   Tensor<T, 2, true>& centroids,
                   Tensor<T, 1, true>* centroidNorms,
                   Tensor<T, 2, true>& queries,
                   int k,
                   Tensor<T, 2, true>& outDistances,
                   Tensor<int, 2, true>& outIndices,
                   bool ignoreOutDistances = false,
                   int tileSize = -1) {
  FAISS_ASSERT(outDistances.getSize(0) == queries.getSize(0));
  FAISS_ASSERT(outIndices.getSize(0) == queries.getSize(0));
  FAISS_ASSERT(outDistances.getSize(1) == k);
  FAISS_ASSERT(outIndices.getSize(1) == k);

  auto& mem = resources->getMemoryManagerCurrentDevice();
  auto defaultStream = resources->getDefaultStreamCurrentDevice();

  // If we're quering against a 0 sized set, just return empty results
  if (centroids.numElements() == 0) {
    thrust::fill(thrust::cuda::par.on(defaultStream),
                 outDistances.data(), outDistances.end(),
                 Limits<T>::getMax());

    thrust::fill(thrust::cuda::par.on(defaultStream),
                 outIndices.data(), outIndices.end(),
                 -1);

    return;
  }

  // If ||c||^2 is not pre-computed, calculate it
  DeviceTensor<T, 1, true> cNorms;
  if (!centroidNorms) {
    cNorms = std::move(DeviceTensor<T, 1, true>(
                       mem,
                       {centroids.getSize(0)}, defaultStream));
    runL2Norm(centroids, cNorms, true, defaultStream);
    centroidNorms = &cNorms;
  }

  //
  // Prepare norm vector ||q||^2; ||c||^2 is already pre-computed
  //
  int qNormSize[1] = {queries.getSize(0)};
  DeviceTensor<T, 1, true> queryNorms(mem, qNormSize, defaultStream);

  // ||q||^2
  runL2Norm(queries, queryNorms, true, defaultStream);

  //
  // Handle the problem in row tiles, to avoid excessive temporary
  // memory requests
  //

  FAISS_ASSERT(k <= centroids.getSize(0));
  FAISS_ASSERT(k <= 1024); // select limitation

  // To allocate all of (#queries, #centroids) is potentially too much
  // memory. Limit our total size requested
  size_t distanceRowSize = centroids.getSize(0) * sizeof(T);

  // FIXME: parameterize based on # of centroids and DeviceMemory
  int defaultTileSize = sizeof(T) < 4 ? kDefaultTileSize * 2 : kDefaultTileSize;
  tileSize = tileSize <= 0 ? defaultTileSize : tileSize;

  int maxQueriesPerIteration = std::min(tileSize, queries.getSize(0));

  // Temporary output memory space we'll use
  DeviceTensor<T, 2, true> distanceBuf1(
    mem, {maxQueriesPerIteration, centroids.getSize(0)}, defaultStream);
  DeviceTensor<T, 2, true> distanceBuf2(
    mem, {maxQueriesPerIteration, centroids.getSize(0)}, defaultStream);
  DeviceTensor<T, 2, true>* distanceBufs[2] =
    {&distanceBuf1, &distanceBuf2};

  auto streams = resources->getAlternateStreamsCurrentDevice();
  streamWait(streams, {defaultStream});

  int curStream = 0;

  for (int i = 0; i < queries.getSize(0); i += maxQueriesPerIteration) {
    int numQueriesForIteration = std::min(maxQueriesPerIteration,
                                          queries.getSize(0) - i);

    auto distanceBufView =
      distanceBufs[curStream]->narrowOutermost(0, numQueriesForIteration);
    auto queryView =
      queries.narrowOutermost(i, numQueriesForIteration);
    auto outDistanceView =
      outDistances.narrowOutermost(i, numQueriesForIteration);
    auto outIndexView =
      outIndices.narrowOutermost(i, numQueriesForIteration);
    auto queryNormNiew =
      queryNorms.narrowOutermost(i, numQueriesForIteration);

    // L2 distance is ||c||^2 - 2qc + ||q||^2

    // -2qc
    // (query id x dim) x (centroid id, dim)' = (query id, centroid id)
    runMatrixMult(distanceBufView, false,
                  queryView, false,
                  centroids, true,
                  -2.0f, 0.0f,
                  resources->getBlasHandleCurrentDevice(),
                  streams[curStream]);

    // For L2 distance, we use this fused kernel that performs both
    // adding ||c||^2 to -2qc and k-selection, so we only need two
    // passes (one write by the gemm, one read here) over the huge
    // region of output memory
    runL2SelectMin(distanceBufView,
                   *centroidNorms,
                   outDistanceView,
                   outIndexView,
                   k,
                   streams[curStream]);

    if (!ignoreOutDistances) {
      // expand (query id) to (query id, k) by duplicating along rows
      // top-k ||c||^2 - 2qc + ||q||^2 in the form (query id, k)
      runSumAlongRows(queryNormNiew, outDistanceView, streams[curStream]);
    }

    curStream = (curStream + 1) % 2;
  }

  // Have the desired ordering stream wait on the multi-stream
  streamWait({defaultStream}, streams);
}

template <typename T>
void runIPDistance(GpuResources* resources,
                   Tensor<T, 2, true>& centroids,
                   Tensor<T, 2, true>& queries,
                   int k,
                   Tensor<T, 2, true>& outDistances,
                   Tensor<int, 2, true>& outIndices,
                   int tileSize = -1) {
  FAISS_ASSERT(outDistances.getSize(0) == queries.getSize(0));
  FAISS_ASSERT(outIndices.getSize(0) == queries.getSize(0));
  FAISS_ASSERT(outDistances.getSize(1) == k);
  FAISS_ASSERT(outIndices.getSize(1) == k);

  auto& mem = resources->getMemoryManagerCurrentDevice();
  auto defaultStream = resources->getDefaultStreamCurrentDevice();

  // If we're quering against a 0 sized set, just return empty results
  if (centroids.numElements() == 0) {
    thrust::fill(thrust::cuda::par.on(defaultStream),
                 outDistances.data(), outDistances.end(),
                 Limits<T>::getMax());

    thrust::fill(thrust::cuda::par.on(defaultStream),
                 outIndices.data(), outIndices.end(),
                 -1);

    return;
  }

  //
  // Handle the problem in row tiles, to avoid excessive temporary
  // memory requests
  //

  FAISS_ASSERT(k <= centroids.getSize(0));
  FAISS_ASSERT(k <= 1024); // select limitation

  // To allocate all of (#queries, #centroids) is potentially too much
  // memory. Limit our total size requested
  size_t distanceRowSize = centroids.getSize(0) * sizeof(T);

  // FIXME: parameterize based on # of centroids and DeviceMemory
  int defaultTileSize = sizeof(T) < 4 ? kDefaultTileSize * 2 : kDefaultTileSize;
  tileSize = tileSize <= 0 ? defaultTileSize : tileSize;

  int maxQueriesPerIteration = std::min(tileSize, queries.getSize(0));

  // Temporary output memory space we'll use
  DeviceTensor<T, 2, true> distanceBuf1(
    mem, {maxQueriesPerIteration, centroids.getSize(0)}, defaultStream);
  DeviceTensor<T, 2, true> distanceBuf2(
    mem, {maxQueriesPerIteration, centroids.getSize(0)}, defaultStream);
  DeviceTensor<T, 2, true>* distanceBufs[2] =
    {&distanceBuf1, &distanceBuf2};

  auto streams = resources->getAlternateStreamsCurrentDevice();
  streamWait(streams, {defaultStream});

  int curStream = 0;

  for (int i = 0; i < queries.getSize(0); i += maxQueriesPerIteration) {
    int numQueriesForIteration = std::min(maxQueriesPerIteration,
                                          queries.getSize(0) - i);

    auto distanceBufView =
      distanceBufs[curStream]->narrowOutermost(0, numQueriesForIteration);
    auto queryView =
      queries.narrowOutermost(i, numQueriesForIteration);
    auto outDistanceView =
      outDistances.narrowOutermost(i, numQueriesForIteration);
    auto outIndexView =
      outIndices.narrowOutermost(i, numQueriesForIteration);

    // (query id x dim) x (centroid id, dim)' = (query id, centroid id)
    runMatrixMult(distanceBufView, false,
                  queryView, false, centroids, true,
                  1.0f, 0.0f,
                  resources->getBlasHandleCurrentDevice(),
                  streams[curStream]);

    // top-k of dot products
    // (query id, top k centroids)
    runBlockSelect(distanceBufView,
                 outDistanceView,
                 outIndexView,
                 true, k, streams[curStream]);

    curStream = (curStream + 1) % 2;
  }

  streamWait({defaultStream}, streams);
}

//
// Instantiations of the distance templates
//

void
runIPDistance(GpuResources* resources,
              Tensor<float, 2, true>& vectors,
              Tensor<float, 2, true>& queries,
              int k,
              Tensor<float, 2, true>& outDistances,
              Tensor<int, 2, true>& outIndices,
              int tileSize) {
  runIPDistance<float>(resources,
                       vectors,
                       queries,
                       k,
                       outDistances,
                       outIndices,
                       tileSize);
}

#ifdef FAISS_USE_FLOAT16
void
runIPDistance(GpuResources* resources,
              Tensor<half, 2, true>& vectors,
              Tensor<half, 2, true>& queries,
              int k,
              Tensor<half, 2, true>& outDistances,
              Tensor<int, 2, true>& outIndices,
              int tileSize) {
  runIPDistance<half>(resources,
                      vectors,
                      queries,
                      k,
                      outDistances,
                      outIndices,
                      tileSize);
}
#endif

void
runL2Distance(GpuResources* resources,
              Tensor<float, 2, true>& vectors,
              Tensor<float, 1, true>* vectorNorms,
              Tensor<float, 2, true>& queries,
              int k,
              Tensor<float, 2, true>& outDistances,
              Tensor<int, 2, true>& outIndices,
              bool ignoreOutDistances,
              int tileSize) {
  runL2Distance<float>(resources,
                       vectors,
                       vectorNorms,
                       queries,
                       k,
                       outDistances,
                       outIndices,
                       ignoreOutDistances,
                       tileSize);
}

#ifdef FAISS_USE_FLOAT16
void
runL2Distance(GpuResources* resources,
              Tensor<half, 2, true>& vectors,
              Tensor<half, 1, true>* vectorNorms,
              Tensor<half, 2, true>& queries,
              int k,
              Tensor<half, 2, true>& outDistances,
              Tensor<int, 2, true>& outIndices,
              bool ignoreOutDistances,
              int tileSize) {
  runL2Distance<half>(resources,
                      vectors,
                      vectorNorms,
                      queries,
                      k,
                      outDistances,
                      outIndices,
                      ignoreOutDistances,
                      tileSize);
}
#endif

} } // namespace
