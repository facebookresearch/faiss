#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2015-present, Facebook, Inc.
 * All rights reserved.
 *
 * This source code is licensed under the BSD+Patents license found in the
 * LICENSE file in the root directory of this source tree.
 */


#include "Distance.cuh"
#include "BroadcastSum.cuh"
#include "L2Norm.cuh"
#include "L2Select.cuh"
#include "../../FaissAssert.h"
#include "../../AuxIndexStructures.h"
#include "../GpuResources.h"
#include "../utils/DeviceDefs.cuh"
#include "../utils/DeviceUtils.h"
#include "../utils/Limits.cuh"
#include "../utils/MatrixMult.cuh"
#include "../utils/BlockSelectKernel.cuh"

#include <memory>
#include <algorithm>
#include <thrust/fill.h>
#include <thrust/for_each.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

namespace faiss { namespace gpu {

namespace {

template <typename T>
Tensor<T, 2, true> sliceCentroids(Tensor<T, 2, true>& centroids,
                                  Tensor<T, 2, true>* centroidsTransposed,
                                  int startCentroid,
                                  int num) {
  if (startCentroid == 0 && num == centroids.getSize(0)) {
    if (centroidsTransposed) {
      return *centroidsTransposed;
    } else {
      return centroids;
    }
  }

  if (centroidsTransposed) {
    // (dim, num)
    return centroidsTransposed->narrow(1, startCentroid, num);
  } else {
    return centroids.narrow(0, startCentroid, num);
  }
}

// For each chunk of k indices, increment the index by chunk * increment
template <typename T>
__global__ void incrementIndex(Tensor<T, 2, true> indices,
                               int k,
                               int increment) {
  for (int i = threadIdx.x; i < k; i += blockDim.x) {
    indices[blockIdx.y][blockIdx.x * k + i] += blockIdx.x * increment;
  }
}

// Used to update result indices in distance computation where the number of
// centroids is high, and is tiled
template <typename T>
void runIncrementIndex(Tensor<T, 2, true>& indices,
                       int k,
                       int increment,
                       hipStream_t stream) {
  dim3 grid(indices.getSize(1) / k, indices.getSize(0));
  int block = std::min(k, 512);

  // should be exact
  FAISS_ASSERT(grid.x * k == indices.getSize(1));

  incrementIndex<<<grid, block, 0, stream>>>(indices, k, increment);

  hipDeviceSynchronize();
}

// If the inner size (dim) of the vectors is small, we want a larger query tile
// size, like 1024

void chooseTileSize(int numQueries,
                    int numCentroids,
                    int dim,
                    int elementSize,
                    size_t tempMemAvailable,
                    int& tileRows,
                    int& tileCols) {
  // The matrix multiplication should be large enough to be efficient, but if it
  // is too large, we seem to lose efficiency as opposed to double-streaming.
  // Each tile size here defines 1/2 of the memory use due to double streaming.
  // We ignore available temporary memory, as that is adjusted independently by
  // the user and can thus meet these requirements (or not).
  // For <= 4 GB GPUs, prefer 512 MB of usage.
  // For <= 8 GB GPUs, prefer 768 MB of usage.
  // Otherwise, prefer 1 GB of usage.
  auto totalMem = getCurrentDeviceProperties().totalGlobalMem;

  int targetUsage = 0;

  if (totalMem <= ((size_t) 4) * 1024 * 1024 * 1024) {
    targetUsage = 512 * 1024 * 1024;
  } else if (totalMem <= ((size_t) 8) * 1024 * 1024 * 1024) {
    targetUsage = 768 * 1024 * 1024;
  } else {
    targetUsage = 1024 * 1024 * 1024;
  }

  targetUsage /= 2 * elementSize;

  // 512 seems to be a batch size sweetspot for float32.
  // If we are on float16, increase to 512.
  // If the k size (vec dim) of the matrix multiplication is small (<= 32),
  // increase to 1024.
  int preferredTileRows = 512;
  if (dim <= 32) {
    preferredTileRows = 1024;
  }

  tileRows = std::min(preferredTileRows, numQueries);

  // tileCols is the remainder size
  tileCols = std::min(targetUsage / preferredTileRows, numCentroids);
}

}

template <typename T>
void runDistance(bool computeL2,
                 GpuResources* resources,
                 Tensor<T, 2, true>& centroids,
                 Tensor<T, 2, true>* centroidsTransposed,
                 Tensor<T, 1, true>* centroidNorms,
                 Tensor<T, 2, true>& queries,
                 int k,
                 Tensor<T, 2, true>& outDistances,
                 Tensor<int, 2, true>& outIndices,
                 bool useHgemm,
                 bool ignoreOutDistances) {
  FAISS_ASSERT(outDistances.getSize(0) == queries.getSize(0));
  FAISS_ASSERT(outIndices.getSize(0) == queries.getSize(0));
  FAISS_ASSERT(outDistances.getSize(1) == k);
  FAISS_ASSERT(outIndices.getSize(1) == k);

  auto& mem = resources->getMemoryManagerCurrentDevice();
  auto defaultStream = resources->getDefaultStreamCurrentDevice();

  // If we're quering against a 0 sized set, just return empty results
  if (centroids.numElements() == 0) {
    thrust::fill(thrust::cuda::par.on(defaultStream),
                 outDistances.data(), outDistances.end(),
                 Limits<T>::getMax());

    thrust::fill(thrust::cuda::par.on(defaultStream),
                 outIndices.data(), outIndices.end(),
                 -1);

    return;
  }

  // L2: If ||c||^2 is not pre-computed, calculate it
  DeviceTensor<T, 1, true> cNorms;
  if (computeL2 && !centroidNorms) {
    cNorms = std::move(DeviceTensor<T, 1, true>(
                       mem,
                       {centroids.getSize(0)}, defaultStream));
    runL2Norm(centroids, cNorms, true, defaultStream);
    centroidNorms = &cNorms;
  }

  //
  // Prepare norm vector ||q||^2; ||c||^2 is already pre-computed
  //
  int qNormSize[1] = {queries.getSize(0)};
  DeviceTensor<T, 1, true> queryNorms(mem, qNormSize, defaultStream);

  // ||q||^2
  if (computeL2) {
    runL2Norm(queries, queryNorms, true, defaultStream);
  }

  // By default, aim to use up to 512 MB of memory for the processing, with both
  // number of queries and number of centroids being at least 512.
  int tileRows = 0;
  int tileCols = 0;
  chooseTileSize(queries.getSize(0),
                 centroids.getSize(0),
                 queries.getSize(1),
                 sizeof(T),
                 mem.getSizeAvailable(),
                 tileRows,
                 tileCols);

  int numColTiles = utils::divUp(centroids.getSize(0), tileCols);

  // We can have any number of vectors to query against, even less than k, in
  // which case we'll return -1 for the index
  FAISS_ASSERT(k <= GPU_MAX_SELECTION_K); // select limitation

  // Temporary output memory space we'll use
  DeviceTensor<T, 2, true> distanceBuf1(
    mem, {tileRows, tileCols}, defaultStream);
  DeviceTensor<T, 2, true> distanceBuf2(
    mem, {tileRows, tileCols}, defaultStream);
  DeviceTensor<T, 2, true>* distanceBufs[2] =
    {&distanceBuf1, &distanceBuf2};

  DeviceTensor<T, 2, true> outDistanceBuf1(
    mem, {tileRows, numColTiles * k}, defaultStream);
  DeviceTensor<T, 2, true> outDistanceBuf2(
    mem, {tileRows, numColTiles * k}, defaultStream);
  DeviceTensor<T, 2, true>* outDistanceBufs[2] =
    {&outDistanceBuf1, &outDistanceBuf2};

  DeviceTensor<int, 2, true> outIndexBuf1(
    mem, {tileRows, numColTiles * k}, defaultStream);
  DeviceTensor<int, 2, true> outIndexBuf2(
    mem, {tileRows, numColTiles * k}, defaultStream);
  DeviceTensor<int, 2, true>* outIndexBufs[2] =
    {&outIndexBuf1, &outIndexBuf2};

  auto streams = resources->getAlternateStreamsCurrentDevice();
  streamWait(streams, {defaultStream});

  int curStream = 0;
  bool interrupt = false;

  // Tile over the input queries
  for (int i = 0; i < queries.getSize(0); i += tileRows) {

    if (interrupt || InterruptCallback::is_interrupted()) {
      interrupt = true;
      break;
    }

    int curQuerySize = std::min(tileRows, queries.getSize(0) - i);

    auto outDistanceView =
      outDistances.narrow(0, i, curQuerySize);
    auto outIndexView =
      outIndices.narrow(0, i, curQuerySize);

    auto queryView =
      queries.narrow(0, i, curQuerySize);
    auto queryNormNiew =
      queryNorms.narrow(0, i, curQuerySize);

    auto outDistanceBufRowView =
      outDistanceBufs[curStream]->narrow(0, 0, curQuerySize);
    auto outIndexBufRowView =
      outIndexBufs[curStream]->narrow(0, 0, curQuerySize);

    // Tile over the centroids
    for (int j = 0; j < centroids.getSize(0); j += tileCols) {

      if (InterruptCallback::is_interrupted()) {
        interrupt = true;
        break;
      }

      int curCentroidSize = std::min(tileCols, centroids.getSize(0) - j);

      int curColTile = j / tileCols;

      auto centroidsView =
        sliceCentroids(centroids, centroidsTransposed, j, curCentroidSize);

      auto distanceBufView = distanceBufs[curStream]->
        narrow(0, 0, curQuerySize).narrow(1, 0, curCentroidSize);

      auto outDistanceBufColView =
        outDistanceBufRowView.narrow(1, k * curColTile, k);
      auto outIndexBufColView =
        outIndexBufRowView.narrow(1, k * curColTile, k);

      // L2: distance is ||c||^2 - 2qc + ||q||^2, we compute -2qc
      // IP: just compute qc
      // (query id x dim) x (centroid id, dim)' = (query id, centroid id)
      runMatrixMult(distanceBufView, false,
                    queryView, false,
                    centroidsView,
                    centroidsTransposed ? false : true,
                    computeL2 ? -2.0f : 1.0f, 0.0f, useHgemm,
                    resources->getBlasHandleCurrentDevice(),
                    streams[curStream]);

      if (computeL2) {
        // For L2 distance, we use this fused kernel that performs both
        // adding ||c||^2 to -2qc and k-selection, so we only need two
        // passes (one write by the gemm, one read here) over the huge
        // region of output memory
        //
        // If we aren't tiling along the number of centroids, we can perform the
        // output work directly
        if (tileCols == centroids.getSize(0)) {
          // Write into the final output
          runL2SelectMin(distanceBufView,
                         *centroidNorms,
                         outDistanceView,
                         outIndexView,
                         k,
                         streams[curStream]);

          if (!ignoreOutDistances) {
            // expand (query id) to (query id, k) by duplicating along rows
            // top-k ||c||^2 - 2qc + ||q||^2 in the form (query id, k)
            runSumAlongRows(queryNormNiew,
                            outDistanceView,
                            true, // L2 distances should not go below zero due
                                  // to roundoff error
                            streams[curStream]);
          }
        } else {
          auto centroidNormsView =
            centroidNorms->narrow(0, j, curCentroidSize);

          // Write into our intermediate output
          runL2SelectMin(distanceBufView,
                         centroidNormsView,
                         outDistanceBufColView,
                         outIndexBufColView,
                         k,
                         streams[curStream]);

          if (!ignoreOutDistances) {
            // expand (query id) to (query id, k) by duplicating along rows
            // top-k ||c||^2 - 2qc + ||q||^2 in the form (query id, k)
            runSumAlongRows(queryNormNiew,
                            outDistanceBufColView,
                            true, // L2 distances should not go below zero due
                                  // to roundoff error
                            streams[curStream]);
          }
        }
      } else {
        // For IP, just k-select the output for this tile
        if (tileCols == centroids.getSize(0)) {
          // Write into the final output
          runBlockSelect(distanceBufView,
                         outDistanceView,
                         outIndexView,
                         true, k, streams[curStream]);
        } else {
          // Write into the intermediate output
          runBlockSelect(distanceBufView,
                         outDistanceBufColView,
                         outIndexBufColView,
                         true, k, streams[curStream]);
        }
      }


    }

    // As we're finished with processing a full set of centroids, perform the
    // final k-selection
    if (tileCols != centroids.getSize(0)) {
      // The indices are tile-relative; for each tile of k, we need to add
      // tileCols to the index
      runIncrementIndex(outIndexBufRowView, k, tileCols, streams[curStream]);

      runBlockSelectPair(outDistanceBufRowView,
                         outIndexBufRowView,
                         outDistanceView,
                         outIndexView,
                         computeL2 ? false : true, k, streams[curStream]);
    }

    curStream = (curStream + 1) % 2;
  }

  // Have the desired ordering stream wait on the multi-stream
  streamWait({defaultStream}, streams);

  if (interrupt) {
    FAISS_THROW_MSG("interrupted");
  }
}

template <typename T>
void runL2Distance(GpuResources* resources,
                   Tensor<T, 2, true>& centroids,
                   Tensor<T, 2, true>* centroidsTransposed,
                   Tensor<T, 1, true>* centroidNorms,
                   Tensor<T, 2, true>& queries,
                   int k,
                   Tensor<T, 2, true>& outDistances,
                   Tensor<int, 2, true>& outIndices,
                   bool useHgemm,
                   bool ignoreOutDistances = false) {
  runDistance<T>(true, // L2
                 resources,
                 centroids,
                 centroidsTransposed,
                 centroidNorms,
                 queries,
                 k,
                 outDistances,
                 outIndices,
                 useHgemm,
                 ignoreOutDistances);
}

template <typename T>
void runIPDistance(GpuResources* resources,
                   Tensor<T, 2, true>& centroids,
                   Tensor<T, 2, true>* centroidsTransposed,
                   Tensor<T, 2, true>& queries,
                   int k,
                   Tensor<T, 2, true>& outDistances,
                   Tensor<int, 2, true>& outIndices,
                   bool useHgemm) {
  runDistance<T>(false, // IP
                 resources,
                 centroids,
                 centroidsTransposed,
                 nullptr,
                 queries,
                 k,
                 outDistances,
                 outIndices,
                 useHgemm,
                 false);
}

//
// Instantiations of the distance templates
//

void
runIPDistance(GpuResources* resources,
              Tensor<float, 2, true>& vectors,
              Tensor<float, 2, true>* vectorsTransposed,
              Tensor<float, 2, true>& queries,
              int k,
              Tensor<float, 2, true>& outDistances,
              Tensor<int, 2, true>& outIndices) {
  runIPDistance<float>(resources,
                       vectors,
                       vectorsTransposed,
                       queries,
                       k,
                       outDistances,
                       outIndices,
                       false);
}

#ifdef FAISS_USE_FLOAT16
void
runIPDistance(GpuResources* resources,
              Tensor<half, 2, true>& vectors,
              Tensor<half, 2, true>* vectorsTransposed,
              Tensor<half, 2, true>& queries,
              int k,
              Tensor<half, 2, true>& outDistances,
              Tensor<int, 2, true>& outIndices,
              bool useHgemm) {
  runIPDistance<half>(resources,
                      vectors,
                      vectorsTransposed,
                      queries,
                      k,
                      outDistances,
                      outIndices,
                      useHgemm);
}
#endif

void
runL2Distance(GpuResources* resources,
              Tensor<float, 2, true>& vectors,
              Tensor<float, 2, true>* vectorsTransposed,
              Tensor<float, 1, true>* vectorNorms,
              Tensor<float, 2, true>& queries,
              int k,
              Tensor<float, 2, true>& outDistances,
              Tensor<int, 2, true>& outIndices,
              bool ignoreOutDistances) {
  runL2Distance<float>(resources,
                       vectors,
                       vectorsTransposed,
                       vectorNorms,
                       queries,
                       k,
                       outDistances,
                       outIndices,
                       false,
                       ignoreOutDistances);
}

#ifdef FAISS_USE_FLOAT16
void
runL2Distance(GpuResources* resources,
              Tensor<half, 2, true>& vectors,
              Tensor<half, 2, true>* vectorsTransposed,
              Tensor<half, 1, true>* vectorNorms,
              Tensor<half, 2, true>& queries,
              int k,
              Tensor<half, 2, true>& outDistances,
              Tensor<int, 2, true>& outIndices,
              bool useHgemm,
              bool ignoreOutDistances) {
  runL2Distance<half>(resources,
                      vectors,
                      vectorsTransposed,
                      vectorNorms,
                      queries,
                      k,
                      outDistances,
                      outIndices,
                      useHgemm,
                      ignoreOutDistances);
}
#endif

} } // namespace
