#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */


#include "L2Norm.cuh"
#include "../../FaissAssert.h"
#include "../utils/ConversionOperators.cuh"
#include "../utils/DeviceDefs.cuh"
#include "../utils/DeviceUtils.h"
#include "../utils/Float16.cuh"
#include "../utils/MathOperators.cuh"
#include "../utils/PtxUtils.cuh"
#include "../utils/StaticUtils.h"
#include "../utils/Reductions.cuh"

namespace faiss { namespace gpu {

// Input: (batch x dim)
// Output: (batch norm)
// Done under the presumption that the dimension size is not too large
// (<10k or so), since there wouldn't be enough parallelism applying a
// single block to the problem. Also that each vector is large enough
// (>64), since a single block works on multiple rows' norms at the
// same time.
// T: the type we are doing the math in (e.g., float, half)
// TVec: the potentially vectorized type we are loading in (e.g.,
// float4, half2)
template <typename T, typename TVec, typename IndexType,
          int RowTileSize, bool NormLoop, bool NormSquared>
__global__ void
l2NormRowMajor(Tensor<TVec, 2, true, IndexType> input,
               Tensor<T, 1, true, IndexType> output) {
  extern __shared__ char smemByte[]; // #warps * RowTileSize elements
  T* smem = (T*) smemByte;

  IndexType numWarps = utils::divUp(blockDim.x, kWarpSize);
  IndexType laneId = getLaneId();
  IndexType warpId = threadIdx.x / kWarpSize;

  bool lastRowTile = (blockIdx.x == (gridDim.x - 1));
  IndexType rowStart = RowTileSize * blockIdx.x;
  T rowNorm[RowTileSize];

  if (lastRowTile) {
    // We are handling the very end of the input matrix rows
    for (IndexType row = 0; row < input.getSize(0) - rowStart; ++row) {
      if (NormLoop) {
        rowNorm[0] = Math<T>::zero();

        for (IndexType col = threadIdx.x;
             col < input.getSize(1); col += blockDim.x) {
          TVec val = input[rowStart + row][col];
          val = Math<TVec>::mul(val, val);
          rowNorm[0] = Math<T>::add(rowNorm[0], Math<TVec>::reduceAdd(val));
        }
      } else {
        TVec val = input[rowStart + row][threadIdx.x];
        val = Math<TVec>::mul(val, val);
        rowNorm[0] = Math<TVec>::reduceAdd(val);
      }

      rowNorm[0] = warpReduceAllSum(rowNorm[0]);
      if (laneId == 0) {
        smem[row * numWarps + warpId] = rowNorm[0];
      }
    }
  } else {
    // We are guaranteed that all RowTileSize rows are available in
    // [rowStart, rowStart + RowTileSize)

    if (NormLoop) {
      // A single block of threads is not big enough to span each
      // vector
      TVec tmp[RowTileSize];

#pragma unroll
      for (int row = 0; row < RowTileSize; ++row) {
        rowNorm[row] = Math<T>::zero();
      }

      for (IndexType col = threadIdx.x;
           col < input.getSize(1); col += blockDim.x) {
#pragma unroll
        for (int row = 0; row < RowTileSize; ++row) {
          tmp[row] = input[rowStart + row][col];
        }

#pragma unroll
        for (int row = 0; row < RowTileSize; ++row) {
          tmp[row] = Math<TVec>::mul(tmp[row], tmp[row]);
        }

#pragma unroll
        for (int row = 0; row < RowTileSize; ++row) {
          rowNorm[row] = Math<T>::add(rowNorm[row],
                                      Math<TVec>::reduceAdd(tmp[row]));
        }
      }
    } else {
      TVec tmp[RowTileSize];

      // A block of threads is the exact size of the vector
#pragma unroll
      for (int row = 0; row < RowTileSize; ++row) {
        tmp[row] = input[rowStart + row][threadIdx.x];
      }

#pragma unroll
      for (int row = 0; row < RowTileSize; ++row) {
        tmp[row] = Math<TVec>::mul(tmp[row], tmp[row]);
      }

#pragma unroll
      for (int row = 0; row < RowTileSize; ++row) {
        rowNorm[row] = Math<TVec>::reduceAdd(tmp[row]);
      }
    }

    // Sum up all parts in each warp
#pragma unroll
    for (int row = 0; row < RowTileSize; ++row) {
      rowNorm[row] = warpReduceAllSum(rowNorm[row]);
    }

    if (laneId == 0) {
#pragma unroll
      for (int row = 0; row < RowTileSize; ++row) {
        smem[row * numWarps + warpId] = rowNorm[row];
      }
    }
  }

  __syncthreads();

  // Sum across warps
  if (warpId == 0) {
#pragma unroll
    for (int row = 0; row < RowTileSize; ++row) {
      rowNorm[row] = laneId < numWarps ?
                              smem[row * numWarps + laneId] : Math<T>::zero();
    }

#pragma unroll
    for (int row = 0; row < RowTileSize; ++row) {
      rowNorm[row] = warpReduceAllSum(rowNorm[row]);
    }

    // Write out answer
    if (laneId == 0) {
#pragma unroll
      for (int row = 0; row < RowTileSize; ++row) {
        int outCol = rowStart + row;

        if (lastRowTile) {
          if (outCol < output.getSize(0)) {
            output[outCol] =
              NormSquared ? rowNorm[row] :
              ConvertTo<T>::to(
                sqrtf(ConvertTo<float>::to(rowNorm[row])));
          }
        } else {
          output[outCol] =
            NormSquared ? rowNorm[row] :
            ConvertTo<T>::to(
              sqrtf(ConvertTo<float>::to(rowNorm[row])));
        }
      }
    }
  }
}

// Input: (dim x batch)
// Output: (batch norm)
// Handles the case where `input` is column major. A single thread calculates
// the norm of each vector instead of a block-wide reduction.
template <typename T, typename IndexType, bool NormSquared>
__global__ void
l2NormColMajor(Tensor<T, 2, true, IndexType> input,
               Tensor<T, 1, true, IndexType> output) {
  // grid-stride loop to handle all batch elements
  for (IndexType batch = blockIdx.x * blockDim.x + threadIdx.x;
       batch < input.getSize(1);
       batch += gridDim.x * blockDim.x) {
    float sum = 0;

    // This is still a coalesced load from the memory
    for (IndexType dim = 0; dim < input.getSize(0); ++dim) {
      // Just do the math in float32, even if the input is float16
      float v = ConvertTo<float>::to(input[dim][batch]);
      sum += v * v;
    }

    if (!NormSquared) {
      sum = sqrtf(sum);
    }

    output[batch] = ConvertTo<T>::to(sum);
  }
}

template <typename T, typename TVec, typename IndexType>
void runL2Norm(Tensor<T, 2, true, IndexType>& input,
               bool inputRowMajor,
               Tensor<T, 1, true, IndexType>& output,
               bool normSquared,
               hipStream_t stream) {
  IndexType maxThreads = (IndexType) getMaxThreadsCurrentDevice();
  constexpr int rowTileSize = 8;

#define RUN_L2_ROW_MAJOR(TYPE_T, TYPE_TVEC, INPUT)                      \
  do {                                                                  \
    if (normLoop) {                                                     \
      if (normSquared) {                                                \
        l2NormRowMajor<TYPE_T, TYPE_TVEC, IndexType, rowTileSize, true, true> \
          <<<grid, block, smem, stream>>>(INPUT, output);               \
      } else {                                                          \
        l2NormRowMajor<TYPE_T, TYPE_TVEC, IndexType, rowTileSize, true, false> \
          <<<grid, block, smem, stream>>>(INPUT, output);               \
      }                                                                 \
    } else {                                                            \
      if (normSquared) {                                                \
        l2NormRowMajor<TYPE_T, TYPE_TVEC, IndexType, rowTileSize, false, true> \
          <<<grid, block, smem, stream>>>(INPUT, output);               \
      } else {                                                          \
        l2NormRowMajor<TYPE_T, TYPE_TVEC, IndexType, rowTileSize, false, false> \
          <<<grid, block, smem, stream>>>(INPUT, output);               \
      }                                                                 \
    }                                                                   \
  } while (0)

  if (inputRowMajor) {
    //
    // Row-major kernel
    ///

    if (input.template canCastResize<TVec>()) {
      // Can load using the vectorized type
      auto inputV = input.template castResize<TVec>();

      auto dim = inputV.getSize(1);
      bool normLoop = dim > maxThreads;
      auto numThreads = min(dim, maxThreads);

      auto grid = dim3(utils::divUp(inputV.getSize(0), rowTileSize));
      auto block = dim3(numThreads);

      auto smem = sizeof(T) * rowTileSize * utils::divUp(numThreads, kWarpSize);

      RUN_L2_ROW_MAJOR(T, TVec, inputV);
    } else {
      // Can't load using the vectorized type

      auto dim = input.getSize(1);
      bool normLoop = dim > maxThreads;
      auto numThreads = min(dim, maxThreads);

      auto grid = dim3(utils::divUp(input.getSize(0), rowTileSize));
      auto block = dim3(numThreads);

      auto smem = sizeof(T) * rowTileSize * utils::divUp(numThreads, kWarpSize);

      RUN_L2_ROW_MAJOR(T, T, input);
    }
  } else {
    //
    // Column-major kernel
    //

    // Just use a fixed-sized block, since the kernel threads are fully
    // independent
    auto block = 128;

    // Cap the grid size at 2^16 since there is a grid-stride loop to handle
    // processing everything
    auto grid = (int)
      std::min(utils::divUp(input.getSize(1), (IndexType) block),
               (IndexType) 65536);

    if (normSquared) {
      l2NormColMajor<T, IndexType, true><<<grid, block, 0, stream>>>(
        input, output);
    } else {
      l2NormColMajor<T, IndexType, false><<<grid, block, 0, stream>>>(
        input, output);
    }
  }

#undef RUN_L2

  CUDA_TEST_ERROR();
}

void runL2Norm(Tensor<float, 2, true>& input,
               bool inputRowMajor,
               Tensor<float, 1, true>& output,
               bool normSquared,
               hipStream_t stream) {
  if (input.canUseIndexType<int>()) {
    runL2Norm<float, float4, int>(
      input, inputRowMajor, output, normSquared, stream);
  } else {
    auto inputCast = input.castIndexType<long>();
    auto outputCast = output.castIndexType<long>();

    runL2Norm<float, float4, long>(
      inputCast, inputRowMajor, outputCast, normSquared, stream);
  }
}

#ifdef FAISS_USE_FLOAT16
void runL2Norm(Tensor<half, 2, true>& input,
               bool inputRowMajor,
               Tensor<half, 1, true>& output,
               bool normSquared,
               hipStream_t stream) {
  if (input.canUseIndexType<int>()) {
    runL2Norm<half, half2, int>(
      input, inputRowMajor, output, normSquared, stream);
  } else {
    auto inputCast = input.castIndexType<long>();
    auto outputCast = output.castIndexType<long>();

    runL2Norm<half, half2, long>(
      inputCast, inputRowMajor, outputCast, normSquared, stream);
  }
}
#endif

} } // namespace
