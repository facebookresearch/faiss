#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include "VectorResidual.cuh"
#include "../../FaissAssert.h"
#include "../utils/ConversionOperators.cuh"
#include "../utils/DeviceUtils.h"
#include "../utils/Tensor.cuh"
#include "../utils/StaticUtils.h"
#include <hip/hip_math_constants.h> // in CUDA SDK, for HIP_NAN_F

namespace faiss { namespace gpu {

template <typename CentroidT, bool LargeDim>
__global__ void calcResidual(Tensor<float, 2, true> vecs,
                             Tensor<CentroidT, 2, true> centroids,
                             Tensor<int, 1, true> vecToCentroid,
                             Tensor<float, 2, true> residuals) {
  auto vec = vecs[blockIdx.x];
  auto residual = residuals[blockIdx.x];

  int centroidId = vecToCentroid[blockIdx.x];
  // Vector could be invalid (containing NaNs), so -1 was the
  // classified centroid
  if (centroidId == -1) {
    if (LargeDim) {
      for (int i = threadIdx.x; i < vecs.getSize(1); i += blockDim.x) {
        residual[i] = HIP_NAN_F;
      }
    } else {
      residual[threadIdx.x] = HIP_NAN_F;
    }

    return;
  }

  auto centroid = centroids[centroidId];

  if (LargeDim) {
    for (int i = threadIdx.x; i < vecs.getSize(1); i += blockDim.x) {
      residual[i] = vec[i] - ConvertTo<float>::to(centroid[i]);
    }
  } else {
    residual[threadIdx.x] = vec[threadIdx.x] -
      ConvertTo<float>::to(centroid[threadIdx.x]);
  }
}

template <typename CentroidT>
void calcResidual(Tensor<float, 2, true>& vecs,
                  Tensor<CentroidT, 2, true>& centroids,
                  Tensor<int, 1, true>& vecToCentroid,
                  Tensor<float, 2, true>& residuals,
                  hipStream_t stream) {
  FAISS_ASSERT(vecs.getSize(1) == centroids.getSize(1));
  FAISS_ASSERT(vecs.getSize(1) == residuals.getSize(1));
  FAISS_ASSERT(vecs.getSize(0) == vecToCentroid.getSize(0));
  FAISS_ASSERT(vecs.getSize(0) == residuals.getSize(0));

  dim3 grid(vecs.getSize(0));

  int maxThreads = getMaxThreadsCurrentDevice();
  bool largeDim = vecs.getSize(1) > maxThreads;
  dim3 block(std::min(vecs.getSize(1), maxThreads));

  if (largeDim) {
    calcResidual<CentroidT, true><<<grid, block, 0, stream>>>(
      vecs, centroids, vecToCentroid, residuals);
  } else {
    calcResidual<CentroidT, false><<<grid, block, 0, stream>>>(
      vecs, centroids, vecToCentroid, residuals);
  }

  CUDA_TEST_ERROR();
}

void runCalcResidual(Tensor<float, 2, true>& vecs,
                     Tensor<float, 2, true>& centroids,
                     Tensor<int, 1, true>& vecToCentroid,
                     Tensor<float, 2, true>& residuals,
                     hipStream_t stream) {
  calcResidual<float>(vecs, centroids, vecToCentroid, residuals, stream);
}

#ifdef FAISS_USE_FLOAT16
void runCalcResidual(Tensor<float, 2, true>& vecs,
                     Tensor<half, 2, true>& centroids,
                     Tensor<int, 1, true>& vecToCentroid,
                     Tensor<float, 2, true>& residuals,
                     hipStream_t stream) {
  calcResidual<half>(vecs, centroids, vecToCentroid, residuals, stream);
}
#endif

} } // namespace
