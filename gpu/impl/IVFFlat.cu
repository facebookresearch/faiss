#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */


#include <faiss/gpu/impl/IVFFlat.cuh>
#include <faiss/gpu/GpuResources.h>
#include <faiss/gpu/impl/FlatIndex.cuh>
#include <faiss/gpu/impl/IVFAppend.cuh>
#include <faiss/gpu/impl/IVFFlatScan.cuh>
#include <faiss/gpu/impl/RemapIndices.h>
#include <faiss/gpu/utils/ConversionOperators.cuh>
#include <faiss/gpu/utils/CopyUtils.cuh>
#include <faiss/gpu/utils/DeviceDefs.cuh>
#include <faiss/gpu/utils/DeviceUtils.h>
#include <faiss/gpu/utils/Float16.cuh>
#include <faiss/gpu/utils/HostTensor.cuh>
#include <faiss/gpu/utils/Transpose.cuh>
#include <limits>
#include <thrust/host_vector.h>
#include <unordered_map>

namespace faiss { namespace gpu {

IVFFlat::IVFFlat(GpuResources* resources,
                 FlatIndex* quantizer,
                 faiss::MetricType metric,
                 float metricArg,
                 bool useResidual,
                 faiss::ScalarQuantizer* scalarQ,
                 IndicesOptions indicesOptions,
                 MemorySpace space) :
    IVFBase(resources,
            metric,
            metricArg,
            quantizer,
            scalarQ ? scalarQ->code_size :
            sizeof(float) * quantizer->getDim(),
            indicesOptions,
            space),
    useResidual_(useResidual),
    scalarQ_(scalarQ ? new GpuScalarQuantizer(*scalarQ) : nullptr) {
}

IVFFlat::~IVFFlat() {
}

void
IVFFlat::addCodeVectorsFromCpu(int listId,
                               const unsigned char* vecs,
                               const long* indices,
                               size_t numVecs) {
  // This list must already exist
  FAISS_ASSERT(listId < deviceListData_.size());
  auto stream = resources_->getDefaultStreamCurrentDevice();

  // If there's nothing to add, then there's nothing we have to do
  if (numVecs == 0) {
    return;
  }

  size_t lengthInBytes = numVecs * bytesPerVector_;

  auto& listData = deviceListData_[listId];
  auto prevData = listData->data();

  // We only have int32 length representations on the GPU per each
  // list; the length is in sizeof(char)
  FAISS_ASSERT(listData->size() + lengthInBytes <=
         (size_t) std::numeric_limits<int>::max());

  listData->append(vecs,
                   lengthInBytes,
                   stream,
                   true /* exact reserved size */);

  // Handle the indices as well
  addIndicesFromCpu_(listId, indices, numVecs);

  // This list address may have changed due to vector resizing, but
  // only bother updating it on the device if it has changed
  if (prevData != listData->data()) {
    deviceListDataPointers_[listId] = listData->data();
  }

  // And our size has changed too
  int listLength = listData->size() / bytesPerVector_;
  deviceListLengths_[listId] = listLength;

  // We update this as well, since the multi-pass algorithm uses it
  maxListLength_ = std::max(maxListLength_, listLength);

  // device_vector add is potentially happening on a different stream
  // than our default stream
  if (stream != 0) {
    streamWait({stream}, {0});
  }
}

int
IVFFlat::classifyAndAddVectors(Tensor<float, 2, true>& vecs,
                               Tensor<long, 1, true>& indices) {
  FAISS_ASSERT(vecs.getSize(0) == indices.getSize(0));
  FAISS_ASSERT(vecs.getSize(1) == dim_);

  auto& mem = resources_->getMemoryManagerCurrentDevice();
  auto stream = resources_->getDefaultStreamCurrentDevice();

  // Number of valid vectors that we actually add; we return this
  int numAdded = 0;

  DeviceTensor<float, 2, true>
    listDistance2d(mem, {vecs.getSize(0), 1}, stream);

  DeviceTensor<int, 2, true>
    listIds2d(mem, {vecs.getSize(0), 1},  stream);
  auto listIds = listIds2d.view<1>({vecs.getSize(0)});

  quantizer_->query(vecs, 1, metric_, metricArg_,
                    listDistance2d, listIds2d, false);

  // Calculate residuals for these vectors, if needed
  DeviceTensor<float, 2, true>
    residuals(mem, {vecs.getSize(0), dim_}, stream);

  if (useResidual_) {
    quantizer_->computeResidual(vecs, listIds, residuals);
  }

  // Copy the lists that we wish to append to back to the CPU
  // FIXME: really this can be into pinned memory and a true async
  // copy on a different stream; we can start the copy early, but it's
  // tiny
  HostTensor<int, 1, true> listIdsHost(listIds, stream);

  // Now we add the encoded vectors to the individual lists
  // First, make sure that there is space available for adding the new
  // encoded vectors and indices

  // list id -> # being added
  std::unordered_map<int, int> assignCounts;

  // vector id -> offset in list
  // (we already have vector id -> list id in listIds)
  HostTensor<int, 1, true> listOffsetHost({listIdsHost.getSize(0)});

  for (int i = 0; i < listIds.getSize(0); ++i) {
    int listId = listIdsHost[i];

    // Add vector could be invalid (contains NaNs etc)
    if (listId < 0) {
      listOffsetHost[i] = -1;
      continue;
    }

    FAISS_ASSERT(listId < numLists_);
    ++numAdded;

    int offset = deviceListData_[listId]->size() / bytesPerVector_;

    auto it = assignCounts.find(listId);
    if (it != assignCounts.end()) {
      offset += it->second;
      it->second++;
    } else {
      assignCounts[listId] = 1;
    }

    listOffsetHost[i] = offset;
  }

  // If we didn't add anything (all invalid vectors), no need to
  // continue
  if (numAdded == 0) {
    return 0;
  }

  // We need to resize the data structures for the inverted lists on
  // the GPUs, which means that they might need reallocation, which
  // means that their base address may change. Figure out the new base
  // addresses, and update those in a batch on the device
  {
    for (auto& counts : assignCounts) {
      auto& data = deviceListData_[counts.first];
      data->resize(data->size() + counts.second * bytesPerVector_,
                   stream);
      int newNumVecs = (int) (data->size() / bytesPerVector_);

      auto& indices = deviceListIndices_[counts.first];
      if ((indicesOptions_ == INDICES_32_BIT) ||
          (indicesOptions_ == INDICES_64_BIT)) {
        size_t indexSize =
          (indicesOptions_ == INDICES_32_BIT) ? sizeof(int) : sizeof(long);

        indices->resize(indices->size() + counts.second * indexSize, stream);
      } else if (indicesOptions_ == INDICES_CPU) {
        // indices are stored on the CPU side
        FAISS_ASSERT(counts.first < listOffsetToUserIndex_.size());

        auto& userIndices = listOffsetToUserIndex_[counts.first];
        userIndices.resize(newNumVecs);
      } else {
        // indices are not stored on the GPU or CPU side
        FAISS_ASSERT(indicesOptions_ == INDICES_IVF);
      }

      // This is used by the multi-pass query to decide how much scratch
      // space to allocate for intermediate results
      maxListLength_ = std::max(maxListLength_, newNumVecs);
    }

    // Update all pointers to the lists on the device that may have
    // changed
    {
      std::vector<int> listIds(assignCounts.size());
      int i = 0;
      for (auto& counts : assignCounts) {
        listIds[i++] = counts.first;
      }

      updateDeviceListInfo_(listIds, stream);
    }
  }

  // If we're maintaining the indices on the CPU side, update our
  // map. We already resized our map above.
  if (indicesOptions_ == INDICES_CPU) {
    // We need to maintain the indices on the CPU side
    HostTensor<long, 1, true> hostIndices(indices, stream);

    for (int i = 0; i < hostIndices.getSize(0); ++i) {
      int listId = listIdsHost[i];

      // Add vector could be invalid (contains NaNs etc)
      if (listId < 0) {
        continue;
      }

      int offset = listOffsetHost[i];

      FAISS_ASSERT(listId < listOffsetToUserIndex_.size());
      auto& userIndices = listOffsetToUserIndex_[listId];

      FAISS_ASSERT(offset < userIndices.size());
      userIndices[offset] = hostIndices[i];
    }
  }

  // We similarly need to actually append the new vectors
  {
    DeviceTensor<int, 1, true> listOffset(mem, listOffsetHost, stream);

    // Now, for each list to which a vector is being assigned, write it
    runIVFFlatInvertedListAppend(listIds,
                                 listOffset,
                                 vecs,
                                 indices,
                                 useResidual_,
                                 residuals,
                                 scalarQ_.get(),
                                 deviceListDataPointers_,
                                 deviceListIndexPointers_,
                                 indicesOptions_,
                                 stream);
  }

  return numAdded;
}

void
IVFFlat::query(Tensor<float, 2, true>& queries,
               int nprobe,
               int k,
               Tensor<float, 2, true>& outDistances,
               Tensor<long, 2, true>& outIndices) {
  auto& mem = resources_->getMemoryManagerCurrentDevice();
  auto stream = resources_->getDefaultStreamCurrentDevice();

  // These are caught at a higher level
  FAISS_ASSERT(nprobe <= GPU_MAX_SELECTION_K);
  FAISS_ASSERT(k <= GPU_MAX_SELECTION_K);
  nprobe = std::min(nprobe, quantizer_->getSize());

  FAISS_ASSERT(queries.getSize(1) == dim_);

  FAISS_ASSERT(outDistances.getSize(0) == queries.getSize(0));
  FAISS_ASSERT(outIndices.getSize(0) == queries.getSize(0));

  // Reserve space for the quantized information
  DeviceTensor<float, 2, true>
    coarseDistances(mem, {queries.getSize(0), nprobe}, stream);
  DeviceTensor<int, 2, true>
    coarseIndices(mem, {queries.getSize(0), nprobe}, stream);

  // Find the `nprobe` closest lists; we can use int indices both
  // internally and externally
  quantizer_->query(queries,
                    nprobe,
                    metric_,
                    metricArg_,
                    coarseDistances,
                    coarseIndices,
                    false);

  DeviceTensor<float, 3, true>
    residualBase(mem, {queries.getSize(0), nprobe, dim_}, stream);

  if (useResidual_) {
    // Reconstruct vectors from the quantizer
    quantizer_->reconstruct(coarseIndices, residualBase);
  }

  runIVFFlatScan(queries,
                 coarseIndices,
                 deviceListDataPointers_,
                 deviceListIndexPointers_,
                 indicesOptions_,
                 deviceListLengths_,
                 maxListLength_,
                 k,
                 metric_,
                 useResidual_,
                 residualBase,
                 scalarQ_.get(),
                 outDistances,
                 outIndices,
                 resources_);

  // If the GPU isn't storing indices (they are on the CPU side), we
  // need to perform the re-mapping here
  // FIXME: we might ultimately be calling this function with inputs
  // from the CPU, these are unnecessary copies
  if (indicesOptions_ == INDICES_CPU) {
    HostTensor<long, 2, true> hostOutIndices(outIndices, stream);

    ivfOffsetToUserIndex(hostOutIndices.data(),
                         numLists_,
                         hostOutIndices.getSize(0),
                         hostOutIndices.getSize(1),
                         listOffsetToUserIndex_);

    // Copy back to GPU, since the input to this function is on the
    // GPU
    outIndices.copyFrom(hostOutIndices, stream);
  }
}

} } // namespace
