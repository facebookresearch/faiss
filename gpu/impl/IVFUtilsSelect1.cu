#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2015-present, Facebook, Inc.
 * All rights reserved.
 *
 * This source code is licensed under the BSD+Patents license found in the
 * LICENSE file in the root directory of this source tree.
 */


#include "IVFUtils.cuh"
#include "../utils/DeviceUtils.h"
#include "../utils/Limits.cuh"
#include "../utils/Select.cuh"
#include "../utils/StaticUtils.h"
#include "../utils/Tensor.cuh"

//
// This kernel is split into a separate compilation unit to cut down
// on compile time
//

namespace faiss { namespace gpu {

template <int ThreadsPerBlock, int NumWarpQ, int NumThreadQ, bool Dir>
__global__ void
pass1SelectLists(Tensor<int, 2, true> prefixSumOffsets,
                 Tensor<float, 1, true> distance,
                 int nprobe,
                 int k,
                 Tensor<float, 3, true> heapDistances,
                 Tensor<int, 3, true> heapIndices) {
  constexpr int kNumWarps = ThreadsPerBlock / kWarpSize;

  __shared__ float smemK[kNumWarps * NumWarpQ];
  __shared__ int smemV[kNumWarps * NumWarpQ];

  constexpr auto kInit = Dir ? kFloatMin : kFloatMax;
  BlockSelect<float, int, Dir, Comparator<float>,
              NumWarpQ, NumThreadQ, ThreadsPerBlock>
    heap(kInit, -1, smemK, smemV, k);

  auto queryId = blockIdx.y;
  auto sliceId = blockIdx.x;
  auto numSlices = gridDim.x;

  int sliceSize = (nprobe / numSlices);
  int sliceStart = sliceSize * sliceId;
  int sliceEnd = sliceId == (numSlices - 1) ? nprobe :
    sliceStart + sliceSize;
  auto offsets = prefixSumOffsets[queryId].data();

  // We ensure that before the array (at offset -1), there is a 0 value
  int start = *(&offsets[sliceStart] - 1);
  int end = offsets[sliceEnd - 1];

  int num = end - start;
  int limit = utils::roundDown(num, kWarpSize);

  int i = threadIdx.x;
  auto distanceStart = distance[start].data();

  // BlockSelect add cannot be used in a warp divergent circumstance; we
  // handle the remainder warp below
  for (; i < limit; i += blockDim.x) {
    heap.add(distanceStart[i], start + i);
  }

  // Handle warp divergence separately
  if (i < num) {
    heap.addThreadQ(distanceStart[i], start + i);
  }

  // Merge all final results
  heap.reduce();

  // Write out the final k-selected values; they should be all
  // together
  for (int i = threadIdx.x; i < k; i += blockDim.x) {
    heapDistances[queryId][sliceId][i] = smemK[i];
    heapIndices[queryId][sliceId][i] = smemV[i];
  }
}

void
runPass1SelectLists(Tensor<int, 2, true>& prefixSumOffsets,
                    Tensor<float, 1, true>& distance,
                    int nprobe,
                    int k,
                    bool chooseLargest,
                    Tensor<float, 3, true>& heapDistances,
                    Tensor<int, 3, true>& heapIndices,
                    hipStream_t stream) {
  constexpr auto kThreadsPerBlock = 128;

  auto grid = dim3(heapDistances.getSize(1), prefixSumOffsets.getSize(0));
  auto block = dim3(kThreadsPerBlock);

#define RUN_PASS(NUM_WARP_Q, NUM_THREAD_Q, DIR)                         \
  do {                                                                  \
    pass1SelectLists<kThreadsPerBlock, NUM_WARP_Q, NUM_THREAD_Q, DIR>   \
      <<<grid, block, 0, stream>>>(prefixSumOffsets,                    \
                                   distance,                            \
                                   nprobe,                              \
                                   k,                                   \
                                   heapDistances,                       \
                                   heapIndices);                        \
    CUDA_TEST_ERROR();                                                  \
    return; /* success */                                               \
  } while (0)

#define RUN_PASS_DIR(DIR)                            \
  do {                                               \
    if (k == 1) {                                    \
      RUN_PASS(1, 1, DIR);                           \
    } else if (k <= 32) {                            \
      RUN_PASS(32, 2, DIR);                          \
    } else if (k <= 64) {                            \
      RUN_PASS(64, 3, DIR);                          \
    } else if (k <= 128) {                           \
      RUN_PASS(128, 3, DIR);                         \
    } else if (k <= 256) {                           \
      RUN_PASS(256, 4, DIR);                         \
    } else if (k <= 512) {                           \
      RUN_PASS(512, 8, DIR);                         \
    } else if (k <= 1024) {                          \
      RUN_PASS(1024, 8, DIR);                        \
    }                                                \
  } while (0)

  if (chooseLargest) {
    RUN_PASS_DIR(true);
  } else {
    RUN_PASS_DIR(false);
  }

  // unimplemented / too many resources
  FAISS_ASSERT_FMT(false, "unimplemented k value (%d)", k);

#undef RUN_PASS_DIR
#undef RUN_PASS
}

} } // namespace
