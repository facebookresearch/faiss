/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */


#include "BinaryFlatIndex.cuh"
#include "BinaryDistance.cuh"
#include "../utils/DeviceUtils.h"
#include "../GpuResources.h"

namespace faiss { namespace gpu {

BinaryFlatIndex::BinaryFlatIndex(GpuResources* res,
                                 int dim,
                                 MemorySpace space) :
    resources_(res),
    dim_(dim),
    space_(space),
    num_(0),
    rawData_(space) {
  FAISS_ASSERT(dim % 8 == 0);
}

/// Returns the number of vectors we contain
int BinaryFlatIndex::getSize() const {
  return vectors_.getSize(0);
}

int BinaryFlatIndex::getDim() const {
  return vectors_.getSize(1) * 8;
}

void
BinaryFlatIndex::reserve(size_t numVecs, hipStream_t stream) {
  rawData_.reserve(numVecs * (dim_ / 8) * sizeof(unsigned int), stream);
}

Tensor<unsigned char, 2, true>&
BinaryFlatIndex::getVectorsRef() {
  return vectors_;
}

void
BinaryFlatIndex::query(Tensor<unsigned char, 2, true>& input,
                       int k,
                       Tensor<int, 2, true>& outDistances,
                       Tensor<int, 2, true>& outIndices) {
  auto stream = resources_->getDefaultStreamCurrentDevice();

  runBinaryDistance(vectors_,
                    input,
                    outDistances,
                    outIndices,
                    k,
                    stream);
}

void
BinaryFlatIndex::add(const unsigned char* data,
                     int numVecs,
                     hipStream_t stream) {
  if (numVecs == 0) {
    return;
  }

  rawData_.append((char*) data,
                  (size_t) (dim_ / 8) * numVecs * sizeof(unsigned char),
                  stream,
                  true /* reserve exactly */);

  num_ += numVecs;

  DeviceTensor<unsigned char, 2, true> vectors(
    (unsigned char*) rawData_.data(), {(int) num_, (dim_ / 8)}, space_);
  vectors_ = std::move(vectors);
}

void
BinaryFlatIndex::reset() {
  rawData_.clear();
  vectors_ = std::move(DeviceTensor<unsigned char, 2, true>());
  num_ = 0;
}

} }
