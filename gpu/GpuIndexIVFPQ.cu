#include "hip/hip_runtime.h"

/**
 * Copyright (c) 2015-present, Facebook, Inc.
 * All rights reserved.
 *
 * This source code is licensed under the CC-by-NC license found in the
 * LICENSE file in the root directory of this source tree.
 */

// Copyright 2004-present Facebook. All Rights Reserved.

#include "GpuIndexIVFPQ.h"
#include "../ProductQuantizer.h"
#include "GpuIndexFlat.h"
#include "GpuResources.h"
#include "impl/IVFPQ.cuh"
#include "utils/CopyUtils.cuh"
#include "utils/DeviceUtils.h"
#include "../IndexFlat.h"
#include "../IndexIVFPQ.h"

#include <limits>

namespace faiss { namespace gpu {

GpuIndexIVFPQ::GpuIndexIVFPQ(GpuResources* resources,
                             int device,
                             IndicesOptions indicesOptions,
                             bool useFloat16LookupTables,
                             const faiss::IndexIVFPQ* index) :
    GpuIndexIVF(resources,
                device,
                indicesOptions,
                false, // FIXME: float 16 coarse quantizer
                index->d,
                index->metric_type,
                index->nlist),
    useFloat16LookupTables_(useFloat16LookupTables),
    subQuantizers_(0),
    bitsPerCode_(0),
    usePrecomputed_(false),
    reserveMemoryVecs_(0),
    index_(nullptr) {
#ifndef FAISS_USE_FLOAT16
  FAISS_ASSERT(!useFloat16LookupTables_);
#endif

  copyFrom(index);
}

GpuIndexIVFPQ::GpuIndexIVFPQ(GpuResources* resources,
                             int device,
                             int dims,
                             int nlist,
                             int subQuantizers,
                             int bitsPerCode,
                             bool usePrecomputed,
                             IndicesOptions indicesOptions,
                             bool useFloat16LookupTables,
                             faiss::MetricType metric) :
    GpuIndexIVF(resources,
                device,
                indicesOptions,
                false, // FIXME: float 16 coarse quantizer
                dims,
                metric,
                nlist),
    useFloat16LookupTables_(useFloat16LookupTables),
    subQuantizers_(subQuantizers),
    bitsPerCode_(bitsPerCode),
    usePrecomputed_(usePrecomputed),
    reserveMemoryVecs_(0),
    index_(nullptr) {
#ifndef FAISS_USE_FLOAT16
  FAISS_ASSERT(!useFloat16LookupTables_);
#endif

  assertSettings_();

  // FIXME make IP work fully
  FAISS_ASSERT(this->metric_type == faiss::METRIC_L2);

  // We haven't trained ourselves, so don't construct the PQ index yet
  this->is_trained = false;
}

GpuIndexIVFPQ::~GpuIndexIVFPQ() {
  delete index_;
}

void
GpuIndexIVFPQ::copyFrom(const faiss::IndexIVFPQ* index) {
  DeviceScope scope(device_);

  // FIXME: support this
  FAISS_ASSERT(index->metric_type == faiss::METRIC_L2);
  GpuIndexIVF::copyFrom(index);

  // Clear out our old data
  delete index_;
  index_ = nullptr;

  subQuantizers_ = index->pq.M;
  bitsPerCode_ = index->pq.nbits;

  // We only support this
  FAISS_ASSERT(index->pq.byte_per_idx == 1);
  FAISS_ASSERT(index->by_residual);
  FAISS_ASSERT(index->polysemous_ht == 0);
  usePrecomputed_ = index->use_precomputed_table;

  assertSettings_();

  // The other index might not be trained
  if (!index->is_trained) {
    return;
  }

  // Otherwise, we can populate ourselves from the other index
  this->is_trained = true;

  // Copy our lists as well
  // The product quantizer must have data in it
  FAISS_ASSERT(index->pq.centroids.size() > 0);
  index_ = new IVFPQ(resources_,
                     quantizer_->getGpuData(),
                     subQuantizers_,
                     bitsPerCode_,
                     (float*) index->pq.centroids.data(),
                     indicesOptions_,
                     useFloat16LookupTables_);
  // Doesn't make sense to reserve memory here
  index_->setPrecomputedCodes(usePrecomputed_);

  // Copy database vectors, if any
  for (size_t i = 0; i < index->codes.size(); ++i) {
    auto& codes = index->codes[i];
    auto& indices = index->ids[i];

    FAISS_ASSERT(indices.size() * subQuantizers_ == codes.size());
    index_->addCodeVectorsFromCpu(i,
                                  codes.data(),
                                  indices.data(),
                                  indices.size());
  }
}

void
GpuIndexIVFPQ::copyTo(faiss::IndexIVFPQ* index) const {
  DeviceScope scope(device_);

  // We must have the indices in order to copy to ourselves
  FAISS_ASSERT(indicesOptions_ != INDICES_IVF);

  GpuIndexIVF::copyTo(index);

  //
  // IndexIVFPQ information
  //
  index->by_residual = true;
  index->use_precomputed_table = 0;
  index->code_size = subQuantizers_;
  index->pq = faiss::ProductQuantizer(this->d, subQuantizers_, bitsPerCode_);

  index->do_polysemous_training = false;
  index->polysemous_training = nullptr;

  index->scan_table_threshold = 0;
  index->max_codes = 0;
  index->polysemous_ht = 0;
  index->codes.clear();
  index->codes.resize(nlist_);
  index->precomputed_table.clear();

  if (index_) {
    // Copy the inverted lists
    for (int i = 0; i < nlist_; ++i) {
      index->ids[i] = getListIndices(i);
      index->codes[i] = getListCodes(i);
    }

    // Copy PQ centroids
    auto devPQCentroids = index_->getPQCentroids();
    index->pq.centroids.resize(devPQCentroids.numElements());

    fromDevice<float, 3>(devPQCentroids,
                         index->pq.centroids.data(),
                         resources_->getDefaultStream(device_));

    if (usePrecomputed_) {
      index->precompute_table();
    }
  }
}

void
GpuIndexIVFPQ::reserveMemory(size_t numVecs) {
  reserveMemoryVecs_ = numVecs;
  if (index_) {
    DeviceScope scope(device_);
    index_->reserveMemory(numVecs);
  }
}

void
GpuIndexIVFPQ::setPrecomputedCodes(bool enable) {
  usePrecomputed_ = enable;
  if (index_) {
    DeviceScope scope(device_);
    index_->setPrecomputedCodes(enable);
  }

  assertSettings_();
}

bool
GpuIndexIVFPQ::getPrecomputedCodes() const {
  return usePrecomputed_;
}

bool
GpuIndexIVFPQ::getFloat16LookupTables() const {
  return useFloat16LookupTables_;
}

int
GpuIndexIVFPQ::getNumSubQuantizers() const {
  return subQuantizers_;
}

int
GpuIndexIVFPQ::getBitsPerCode() const {
  return bitsPerCode_;
}

int
GpuIndexIVFPQ::getCentroidsPerSubQuantizer() const {
  return utils::pow2(bitsPerCode_);
}

size_t
GpuIndexIVFPQ::reclaimMemory() {
  if (index_) {
    DeviceScope scope(device_);
    return index_->reclaimMemory();
  }

  return 0;
}

void
GpuIndexIVFPQ::reset() {
  if (index_) {
    DeviceScope scope(device_);

    index_->reset();
    this->ntotal = 0;
  } else {
    FAISS_ASSERT(this->ntotal == 0);
  }
}

void
GpuIndexIVFPQ::trainResidualQuantizer_(Index::idx_t n, const float* x) {
  // Code largely copied from faiss::IndexIVFPQ
  // FIXME: GPUize more of this
  n = std::min(n, (Index::idx_t) (1 << bitsPerCode_) * 64);

  if (this->verbose) {
    printf("computing residuals\n");
  }

  std::vector<Index::idx_t> assign(n);
  quantizer_->assign (n, x, assign.data());

  std::vector<float> residuals(n * d);

  for (idx_t i = 0; i < n; i++) {
    quantizer_->compute_residual(x + i * d, &residuals[i * d], assign[i]);
  }

  if (this->verbose) {
    printf("training %d x %d product quantizer on %ld vectors in %dD\n",
           subQuantizers_, getCentroidsPerSubQuantizer(), n, this->d);
  }

  // Just use the CPU product quantizer to determine sub-centroids
  faiss::ProductQuantizer pq(this->d, subQuantizers_, bitsPerCode_);
  pq.verbose = this->verbose;
  pq.train(n, residuals.data());

  index_ = new IVFPQ(resources_,
                     quantizer_->getGpuData(),
                     subQuantizers_,
                     bitsPerCode_,
                     pq.centroids.data(),
                     indicesOptions_,
                     useFloat16LookupTables_);
  if (reserveMemoryVecs_) {
    index_->reserveMemory(reserveMemoryVecs_);
  }

  index_->setPrecomputedCodes(usePrecomputed_);
}

void
GpuIndexIVFPQ::train(Index::idx_t n, const float* x) {
  DeviceScope scope(device_);

  if (this->is_trained) {
    FAISS_ASSERT(quantizer_->is_trained);
    FAISS_ASSERT(quantizer_->ntotal == nlist_);
    FAISS_ASSERT(index_);
    return;
  }

  FAISS_ASSERT(!index_);

  trainQuantizer_(n, x);
  trainResidualQuantizer_(n, x);

  this->is_trained = true;
}

void
GpuIndexIVFPQ::add_with_ids(Index::idx_t n,
                            const float* x,
                            const Index::idx_t* xids) {
  FAISS_ASSERT(this->is_trained);
  FAISS_ASSERT(index_);

  if (n == 0) {
    return;
  }

  DeviceScope scope(device_);
  auto stream = resources_->getDefaultStreamCurrentDevice();

  auto deviceVecs =
    toDevice<float, 2>(resources_,
                       device_,
                       const_cast<float*>(x),
                       stream,
                       {(int) n, index_->getDim()});

  auto deviceIndices =
    toDevice<Index::idx_t, 1>(resources_,
                              device_,
                              const_cast<Index::idx_t*>(xids),
                              stream,
                              {(int) n});

  // Not all vectors may be able to be added (some may contain NaNs
  // etc)
  ntotal += index_->classifyAndAddVectors(deviceVecs, deviceIndices);
}

void
GpuIndexIVFPQ::search(faiss::Index::idx_t n,
                      const float* x,
                      faiss::Index::idx_t k,
                      float* distances,
                      faiss::Index::idx_t* labels) const {
  FAISS_ASSERT(this->is_trained);
  FAISS_ASSERT(index_);

  if (n == 0) {
    return;
  }

  DeviceScope scope(device_);

  // Make sure arguments are on the device we desire; use temporary
  // memory allocations to move it if necessary
  auto devX =
    toDevice<float, 2>(resources_,
                       device_,
                       const_cast<float*>(x),
                       resources_->getDefaultStream(device_),
                       {(int) n, index_->getDim()});
  auto devDistances =
    toDevice<float, 2>(resources_,
                       device_,
                       distances,
                       resources_->getDefaultStream(device_),
                       {(int) n, (int) k});
  auto devLabels =
    toDevice<faiss::Index::idx_t, 2>(resources_,
                                     device_,
                                     labels,
                                     resources_->getDefaultStream(device_),
                                     {(int) n, (int) k});

  index_->query(devX,
                nprobe_,
                (int) k,
                devDistances,
                devLabels);

  // Copy back if necessary
  fromDevice<float, 2>(
    devDistances, distances, resources_->getDefaultStream(device_));
  fromDevice<faiss::Index::idx_t, 2>(
    devLabels, labels, resources_->getDefaultStream(device_));
}

void
GpuIndexIVFPQ::set_typename() {
  // FIXME: implement
  FAISS_ASSERT(false);
}

int
GpuIndexIVFPQ::getListLength(int listId) const {
  FAISS_ASSERT(index_);
  return index_->getListLength(listId);
}

std::vector<unsigned char>
GpuIndexIVFPQ::getListCodes(int listId) const {
  FAISS_ASSERT(index_);
  DeviceScope scope(device_);

  return index_->getListCodes(listId);
}

std::vector<long>
GpuIndexIVFPQ::getListIndices(int listId) const {
  FAISS_ASSERT(index_);
  DeviceScope scope(device_);

  return index_->getListIndices(listId);
}

void
GpuIndexIVFPQ::assertSettings_() const {
  // Our implementation has these restrictions:

  // Must have some number of lists
  FAISS_ASSERT(nlist_ > 0);

  // up to a single byte per code
  FAISS_ASSERT(bitsPerCode_ <= 8);

  // Sub-quantizers must evenly divide dimensions available
  FAISS_ASSERT(this->d % subQuantizers_ == 0);

  // The number of bytes per encoded vector must be one we support
  FAISS_ASSERT(IVFPQ::isSupportedPQCodeLength(subQuantizers_));

  // We must have enough shared memory on the current device to store
  // our lookup distances
  int lookupTableSize = sizeof(float);
#ifdef FAISS_USE_FLOAT16
  if (useFloat16LookupTables_) {
    lookupTableSize = sizeof(half);
  }
#endif

  // 64 bytes per code is only supported with usage of float16, at 2^8
  // codes per subquantizer
  FAISS_ASSERT(lookupTableSize * subQuantizers_ * utils::pow2(bitsPerCode_)
               <= getMaxSharedMemPerBlock(device_));

  // If precomputed codes are disabled, we have an extra limitation in
  // terms of the number of dimensions per subquantizer
  FAISS_ASSERT(usePrecomputed_ ||
               IVFPQ::isSupportedNoPrecomputedSubDimSize(
                 this->d / subQuantizers_));

  // TODO: fully implement METRIC_INNER_PRODUCT
  FAISS_ASSERT(this->metric_type == faiss::METRIC_L2);
}

} } // namespace
