#include "hip/hip_runtime.h"

/**
 * Copyright (c) 2015-present, Facebook, Inc.
 * All rights reserved.
 *
 * This source code is licensed under the CC-by-NC license found in the
 * LICENSE file in the root directory of this source tree.
 */

// Copyright 2004-present Facebook. All Rights Reserved.

#include "GpuIndexFlat.h"
#include "../IndexFlat.h"
#include "GpuResources.h"
#include "impl/FlatIndex.cuh"
#include "utils/CopyUtils.cuh"
#include "utils/DeviceUtils.h"
#include "utils/StaticUtils.h"

#include <thrust/execution_policy.h>
#include <thrust/transform.h>
#include <limits>

namespace faiss { namespace gpu {

/// Default CPU search size for which we use paged copies
constexpr size_t kMinPagedQuerySize = (size_t) 256 * 1024 * 1024;

/// Size above which we page copies from the CPU to GPU (non-paged
/// memory usage)
constexpr size_t kNonPinnedPageSize = (size_t) 256 * 1024 * 1024;

GpuIndexFlat::GpuIndexFlat(GpuResources* resources,
                           int device,
                           bool useFloat16,
                           const faiss::IndexFlat* index) :
    GpuIndex(resources, device, index->d, index->metric_type),
    minPagedSize_(kMinPagedQuerySize),
    useFloat16_(useFloat16),
    data_(nullptr) {
  copyFrom(index);
}

GpuIndexFlat::GpuIndexFlat(GpuResources* resources,
                           int device,
                           int dims,
                           bool useFloat16,
                           faiss::MetricType metric) :
    GpuIndex(resources, device, dims, metric),
    minPagedSize_(kMinPagedQuerySize),
    useFloat16_(useFloat16),
    data_(nullptr) {
  DeviceScope scope(device_);

  data_ = new FlatIndex(resources,
                        dims,
                        metric == faiss::METRIC_L2,
                        useFloat16);
}

GpuIndexFlat::~GpuIndexFlat() {
  delete data_;
}

void
GpuIndexFlat::setMinPagingSize(size_t size) {
  minPagedSize_ = size;
}

size_t
GpuIndexFlat::getMinPagingSize() const {
  return minPagedSize_;
}

bool
GpuIndexFlat::getUseFloat16() const {
  return useFloat16_;
}

void
GpuIndexFlat::copyFrom(const faiss::IndexFlat* index) {
  DeviceScope scope(device_);

  this->d = index->d;
  this->metric_type = index->metric_type;

  // GPU code has 32 bit indices
  FAISS_ASSERT(index->ntotal <=
               (faiss::Index::idx_t) std::numeric_limits<int>::max());
  this->ntotal = index->ntotal;

  delete data_;
  data_ = new FlatIndex(resources_,
                        this->d,
                        index->metric_type == faiss::METRIC_L2,
                        useFloat16_);

  // The index could be empty
  if (index->ntotal > 0) {
    data_->add(index->xb.data(),
               index->ntotal,
               resources_->getDefaultStream(device_));
  }
}

void
GpuIndexFlat::copyTo(faiss::IndexFlat* index) const {
  DeviceScope scope(device_);

  index->d = this->d;
  index->ntotal = this->ntotal;
  index->metric_type = this->metric_type;

  FAISS_ASSERT(data_->getSize() == this->ntotal);
  index->xb.resize(this->ntotal * this->d);

  auto stream = resources_->getDefaultStream(device_);

  if (this->ntotal > 0) {
    if (useFloat16_) {
      auto vecFloat32 = data_->getVectorsFloat32Copy(stream);
      fromDevice(vecFloat32, index->xb.data(), stream);
    } else {
      fromDevice(data_->getVectorsFloat32Ref(), index->xb.data(), stream);
    }
  }
}

size_t
GpuIndexFlat::getNumVecs() const {
  return this->ntotal;
}

void
GpuIndexFlat::reset() {
  DeviceScope scope(device_);

  // Free the underlying memory
  data_->reset();
  this->ntotal = 0;
}

void
GpuIndexFlat::train(Index::idx_t n, const float* x) {
  // nothing to do
}

void
GpuIndexFlat::add(Index::idx_t n, const float* x) {
  // Due to GPU indexing in int32, we can't store more than this
  // number of vectors on a GPU
  FAISS_ASSERT(this->ntotal + n <=
               (faiss::Index::idx_t) std::numeric_limits<int>::max());

  if (n > 0) {
    DeviceScope scope(device_);
    data_->add(x, n, resources_->getDefaultStream(device_));
    this->ntotal += n;
  }
}

struct IntToLong {
  __device__ long operator()(int v) const { return (long) v; }
};

void
GpuIndexFlat::search(faiss::Index::idx_t n,
                     const float* x,
                     faiss::Index::idx_t k,
                     float* distances,
                     faiss::Index::idx_t* labels) const {
  if (n == 0) {
    return;
  }

  // For now, only support <= max int results
  // TODO: handle tiling over arbitrary n to keep within 32 bit bounds
  FAISS_ASSERT(n <= (faiss::Index::idx_t) std::numeric_limits<int>::max());
  FAISS_ASSERT(k <= 1024); // select limitation

  DeviceScope scope(device_);
  auto stream = resources_->getDefaultStream(device_);

  // The input vectors may be too large for the GPU, but we still
  // assume that the output distances and labels are not.
  // Go ahead and make space for output distances and labels on the
  // GPU.
  // If we reach a point where all inputs are too big, we can add
  // another level of tiling.
  auto outDistances = toDevice<float, 2>(resources_,
                                         device_,
                                         distances,
                                         stream,
                                         {(int) n, (int) k});

  // FlatIndex only supports an interface returning int indices
  DeviceTensor<int, 2, true> outIntIndices(
    resources_->getMemoryManagerCurrentDevice(),
    {(int) n, (int) k}, stream);

  bool usePaged = false;

  if (getDeviceForAddress(x) == -1) {
    // It is possible that the user is querying for a vector set size
    // `x` that won't fit on the GPU.
    // In this case, we will have to handle paging of the data from CPU
    // -> GPU.
    // Currently, we don't handle the case where the output data won't
    // fit on the GPU (e.g., n * k is too large for the GPU memory).
    size_t dataSize = (size_t) n * this->d * sizeof(float);

    if (dataSize >= minPagedSize_) {
      searchFromCpuPaged_(n, x, k,
                          outDistances.data(),
                          outIntIndices.data());
      usePaged = true;
    }
  }

  if (!usePaged) {
    searchNonPaged_(n, x, k,
                    outDistances.data(),
                    outIntIndices.data());
  }

  // Convert and copy int indices out
  auto outIndices = toDevice<faiss::Index::idx_t, 2>(resources_,
                                                     device_,
                                                     labels,
                                                     stream,
                                                     {(int) n, (int) k});

  // Convert int to long
  thrust::transform(thrust::cuda::par.on(stream),
                    outIntIndices.data(),
                    outIntIndices.end(),
                    outIndices.data(),
                    IntToLong());

  // Copy back if necessary
  fromDevice<float, 2>(outDistances, distances, stream);
  fromDevice<faiss::Index::idx_t, 2>(outIndices, labels, stream);
}

void
GpuIndexFlat::searchNonPaged_(int n,
                              const float* x,
                              int k,
                              float* outDistancesData,
                              int* outIndicesData) const {
  Tensor<float, 2, true> outDistances(outDistancesData, {n, k});
  Tensor<int, 2, true> outIndices(outIndicesData, {n, k});

  auto stream = resources_->getDefaultStream(device_);

  // Make sure arguments are on the device we desire; use temporary
  // memory allocations to move it if necessary
  auto vecs = toDevice<float, 2>(resources_,
                                 device_,
                                 const_cast<float*>(x),
                                 stream,
                                 {n, (int) this->d});

  data_->query(vecs, k, outDistances, outIndices, true);
}

void
GpuIndexFlat::searchFromCpuPaged_(int n,
                                  const float* x,
                                  int k,
                                  float* outDistancesData,
                                  int* outIndicesData) const {
  Tensor<float, 2, true> outDistances(outDistancesData, {n, k});
  Tensor<int, 2, true> outIndices(outIndicesData, {n, k});

  // Is pinned memory available?
  auto pinnedAlloc = resources_->getPinnedMemory();
  int pageSizeInVecs =
    (int) ((pinnedAlloc.second / 2) / (sizeof(float) * this->d));

  if (!pinnedAlloc.first || pageSizeInVecs < 1) {
    // Just page without overlapping copy with compute
    int batchSize = utils::nextHighestPowerOf2(
      (int) ((size_t) kNonPinnedPageSize /
             (sizeof(float) * this->d)));

    for (int cur = 0; cur < n; cur += batchSize) {
      int num = std::min(batchSize, n - cur);

      auto outDistancesSlice = outDistances.narrowOutermost(cur, num);
      auto outIndicesSlice = outIndices.narrowOutermost(cur, num);

      searchNonPaged_(num,
                      x + (size_t) cur * this->d,
                      k,
                      outDistancesSlice.data(),
                      outIndicesSlice.data());
    }

    return;
  }

  //
  // Pinned memory is available, so we can overlap copy with compute.
  // We use two pinned memory buffers, and triple-buffer the
  // procedure:
  //
  // 1 CPU copy -> pinned
  // 2 pinned copy -> GPU
  // 3 GPU compute
  //
  // 1 2 3 1 2 3 ...   (pinned buf A)
  //   1 2 3 1 2 ...   (pinned buf B)
  //     1 2 3 1 ...   (pinned buf A)
  // time ->
  //
  auto defaultStream = resources_->getDefaultStream(device_);
  auto copyStream = resources_->getAsyncCopyStream(device_);

  FAISS_ASSERT((size_t) pageSizeInVecs * this->d <=
               (size_t) std::numeric_limits<int>::max());

  float* bufPinnedA = (float*) pinnedAlloc.first;
  float* bufPinnedB = bufPinnedA + (size_t) pageSizeInVecs * this->d;
  float* bufPinned[2] = {bufPinnedA, bufPinnedB};

  // Reserve space on the GPU for the destination of the pinned buffer
  // copy
  DeviceTensor<float, 2, true> bufGpuA(
    resources_->getMemoryManagerCurrentDevice(),
    {(int) pageSizeInVecs, (int) this->d},
    defaultStream);
  DeviceTensor<float, 2, true> bufGpuB(
    resources_->getMemoryManagerCurrentDevice(),
    {(int) pageSizeInVecs, (int) this->d},
    defaultStream);
  DeviceTensor<float, 2, true>* bufGpus[2] = {&bufGpuA, &bufGpuB};

  // Copy completion events for the pinned buffers
  std::unique_ptr<CudaEvent> eventPinnedCopyDone[2];

  // Execute completion events for the GPU buffers
  std::unique_ptr<CudaEvent> eventGpuExecuteDone[2];

  // All offsets are in terms of number of vectors; they remain within
  // int bounds (as this function only handles max in vectors)

  // Current start offset for buffer 1
  int cur1 = 0;
  int cur1BufIndex = 0;

  // Current start offset for buffer 2
  int cur2 = -1;
  int cur2BufIndex = 0;

  // Current start offset for buffer 3
  int cur3 = -1;
  int cur3BufIndex = 0;

  while (cur3 < n) {
    // Start async pinned -> GPU copy first (buf 2)
    if (cur2 != -1 && cur2 < n) {
      // Copy pinned to GPU
      int numToCopy = std::min(pageSizeInVecs, n - cur2);

      // Make sure any previous execution has completed before continuing
      auto& eventPrev = eventGpuExecuteDone[cur2BufIndex];
      if (eventPrev.get()) {
        eventPrev->streamWaitOnEvent(copyStream);
      }

      CUDA_VERIFY(hipMemcpyAsync(bufGpus[cur2BufIndex]->data(),
                                  bufPinned[cur2BufIndex],
                                  (size_t) numToCopy * this->d * sizeof(float),
                                  hipMemcpyHostToDevice,
                                  copyStream));

      // Mark a completion event in this stream
      eventPinnedCopyDone[cur2BufIndex] =
        std::move(std::unique_ptr<CudaEvent>(new CudaEvent(copyStream)));

      // We pick up from here
      cur3 = cur2;
      cur2 += numToCopy;
      cur2BufIndex = (cur2BufIndex == 0) ? 1 : 0;
    }

    if (cur3 != -1 && cur3 < n) {
      // Process on GPU
      int numToProcess = std::min(pageSizeInVecs, n - cur3);

      // Make sure the previous copy has completed before continuing
      auto& eventPrev = eventPinnedCopyDone[cur3BufIndex];
      FAISS_ASSERT(eventPrev.get());

      eventPrev->streamWaitOnEvent(defaultStream);

      // Create tensor wrappers
      DeviceTensor<float, 2, true> input(bufGpus[cur3BufIndex]->data(),
                                         {numToProcess, this->d});
      auto outDistancesSlice = outDistances.narrowOutermost(cur3, numToProcess);
      auto outIndicesSlice = outIndices.narrowOutermost(cur3, numToProcess);

      data_->query(input, k,
                   outDistancesSlice,
                   outIndicesSlice, true);

      // Create completion event
      eventGpuExecuteDone[cur3BufIndex] =
        std::move(std::unique_ptr<CudaEvent>(new CudaEvent(defaultStream)));

      // We pick up from here
      cur3BufIndex = (cur3BufIndex == 0) ? 1 : 0;
      cur3 += numToProcess;
    }

    if (cur1 < n) {
      // Copy CPU mem to CPU pinned
      int numToCopy = std::min(pageSizeInVecs, n - cur1);

      // Make sure any previous copy has completed before continuing
      auto& eventPrev = eventPinnedCopyDone[cur1BufIndex];
      if (eventPrev.get()) {
        eventPrev->cpuWaitOnEvent();
      }

      memcpy(bufPinned[cur1BufIndex],
             x + (size_t) cur1 * this->d,
             (size_t) numToCopy * this->d * sizeof(float));

      // We pick up from here
      cur2 = cur1;
      cur1 += numToCopy;
      cur1BufIndex = (cur1BufIndex == 0) ? 1 : 0;
    }
  }
}

void
GpuIndexFlat::reconstruct(faiss::Index::idx_t key,
                          float* out) const {
  DeviceScope scope(device_);

  FAISS_ASSERT(key < this->ntotal);
  auto stream = resources_->getDefaultStream(device_);

  if (useFloat16_) {
    auto vec = data_->getVectorsFloat32Copy(key, 1, stream);
    fromDevice(vec.data(), out, this->d, stream);
  } else {
    auto vec = data_->getVectorsFloat32Ref()[key];
    fromDevice(vec.data(), out, this->d, stream);
  }
}

void
GpuIndexFlat::reconstruct_n(faiss::Index::idx_t i0,
                            faiss::Index::idx_t num,
                            float* out) const {
  DeviceScope scope(device_);

  FAISS_ASSERT(i0 < this->ntotal);
  FAISS_ASSERT(i0 + num - 1 < this->ntotal);
  auto stream = resources_->getDefaultStream(device_);

  if (useFloat16_) {
    auto vec = data_->getVectorsFloat32Copy(i0, num, stream);
    fromDevice(vec.data(), out, num * this->d, stream);
  } else {
    auto vec = data_->getVectorsFloat32Ref()[i0];
    fromDevice(vec.data(), out, this->d * num, stream);
  }
}

void
GpuIndexFlat::set_typename() {
  if (this->metric_type == faiss::METRIC_L2) {
    this->index_typename = "GpuL2";
  } else {
    this->index_typename = "GpuIP";
  }
}

//
// GpuIndexFlatL2
//

GpuIndexFlatL2::GpuIndexFlatL2(GpuResources* resources,
                               int device,
                               bool useFloat16,
                               faiss::IndexFlatL2* index) :
    GpuIndexFlat(resources, device, useFloat16, index) {
}

GpuIndexFlatL2::GpuIndexFlatL2(GpuResources* resources,
                               int device,
                               int dims,
                               bool useFloat16) :
    GpuIndexFlat(resources, device, dims, useFloat16, faiss::METRIC_L2) {
}

void
GpuIndexFlatL2::copyFrom(faiss::IndexFlatL2* index) {
  GpuIndexFlat::copyFrom(index);
}

void
GpuIndexFlatL2::copyTo(faiss::IndexFlatL2* index) {
  GpuIndexFlat::copyTo(index);
}

//
// GpuIndexFlatIP
//

GpuIndexFlatIP::GpuIndexFlatIP(GpuResources* resources,
                               int device,
                               bool useFloat16,
                               faiss::IndexFlatIP* index) :
    GpuIndexFlat(resources, device, useFloat16, index) {
}

GpuIndexFlatIP::GpuIndexFlatIP(GpuResources* resources,
                               int device,
                               int dims,
                               bool useFloat16) :
    GpuIndexFlat(resources, device, dims, useFloat16,
                 faiss::METRIC_INNER_PRODUCT) {
}

void
GpuIndexFlatIP::copyFrom(faiss::IndexFlatIP* index) {
  GpuIndexFlat::copyFrom(index);
}

void
GpuIndexFlatIP::copyTo(faiss::IndexFlatIP* index) {
  GpuIndexFlat::copyTo(index);
}

} } // namespace
