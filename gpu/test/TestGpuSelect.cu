#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */


#include "../test/TestUtils.h"
#include "../utils/BlockSelectKernel.cuh"
#include "../utils/DeviceDefs.cuh"
#include "../utils/DeviceTensor.cuh"
#include "../utils/DeviceUtils.h"
#include "../utils/HostTensor.cuh"
#include "../utils/WarpSelectKernel.cuh"
#include <algorithm>
#include <gtest/gtest.h>
#include <sstream>
#include <unordered_map>
#include <vector>

void testForSize(int rows, int cols, int k, bool dir, bool warp) {
  std::vector<float> v = faiss::gpu::randVecs(rows, cols);
  faiss::gpu::HostTensor<float, 2, true> hostVal({rows, cols});

  for (int r = 0; r < rows; ++r) {
    for (int c = 0; c < cols; ++c) {
      hostVal[r][c] = v[r * cols + c];
    }
  }

  // row -> (val -> idx)
  std::unordered_map<int, std::vector<std::pair<int, float>>> hostOutValAndInd;
  for (int r = 0; r < rows; ++r) {
    std::vector<std::pair<int, float>> closest;

    for (int c = 0; c < cols; ++c) {
      closest.emplace_back(c, (float) hostVal[r][c]);
    }

    auto dirFalseFn =
      [](std::pair<int, float>& a, std::pair<int, float>& b) {
      return a.second < b.second;
    };
    auto dirTrueFn =
      [](std::pair<int, float>& a, std::pair<int, float>& b) {
      return a.second > b.second;
    };

    std::sort(closest.begin(), closest.end(), dir ? dirTrueFn : dirFalseFn);
    hostOutValAndInd.emplace(r, closest);
  }

  // Select top-k on GPU
  faiss::gpu::DeviceTensor<float, 2, true> gpuVal(hostVal, 0);
  faiss::gpu::DeviceTensor<float, 2, true> gpuOutVal({rows, k});
  faiss::gpu::DeviceTensor<int, 2, true> gpuOutInd({rows, k});

  if (warp) {
    faiss::gpu::runWarpSelect(gpuVal, gpuOutVal, gpuOutInd, dir, k, 0);
  } else {
    faiss::gpu::runBlockSelect(gpuVal, gpuOutVal, gpuOutInd, dir, k, 0);
  }

  // Copy back to CPU
  faiss::gpu::HostTensor<float, 2, true> outVal(gpuOutVal, 0);
  faiss::gpu::HostTensor<int, 2, true> outInd(gpuOutInd, 0);

  for (int r = 0; r < rows; ++r) {
    std::unordered_map<int, int> seenIndices;

    for (int i = 0; i < k; ++i) {
      float gpuV = outVal[r][i];
      float cpuV = hostOutValAndInd[r][i].second;

      EXPECT_EQ(gpuV, cpuV) <<
        "rows " << rows << " cols " << cols << " k " << k << " dir " << dir
                << " row " << r << " ind " << i;

      // If there are identical elements in a row that should be
      // within the top-k, then it is possible that the index can
      // differ, because the order in which the GPU will see the
      // equivalent values is different than the CPU (and will remain
      // unspecified, since this is affected by the choice of
      // k-selection algorithm that we use)
      int gpuInd = outInd[r][i];
      int cpuInd = hostOutValAndInd[r][i].first;

      // We should never see duplicate indices, however
      auto itSeenIndex = seenIndices.find(gpuInd);

      EXPECT_EQ(itSeenIndex, seenIndices.end()) <<
        "Row " << r << " user index " << gpuInd << " was seen at both " <<
        itSeenIndex->second << " and " << i;

      seenIndices[gpuInd] = i;

      if (gpuInd != cpuInd) {
        // Gather the values from the original data via index; the
        // values should be the same
        float gpuGatherV = hostVal[r][gpuInd];
        float cpuGatherV = hostVal[r][cpuInd];

        EXPECT_EQ(gpuGatherV, cpuGatherV) <<
          "rows " << rows << " cols " << cols << " k " << k << " dir " << dir
                  << " row " << r << " ind " << i << " source ind "
                  << gpuInd << " " << cpuInd;
      }
    }
  }
}

// General test
TEST(TestGpuSelect, test) {
  for (int i = 0; i < 10; ++i) {
    int rows = faiss::gpu::randVal(10, 100);
    int cols = faiss::gpu::randVal(1, 30000);
    int k = std::min(cols, faiss::gpu::randVal(1, GPU_MAX_SELECTION_K));
    bool dir = faiss::gpu::randBool();

    testForSize(rows, cols, k, dir, false);
  }
}

// Test for k = 1
TEST(TestGpuSelect, test1) {
  for (int i = 0; i < 5; ++i) {
    int rows = faiss::gpu::randVal(10, 100);
    int cols = faiss::gpu::randVal(1, 30000);
    bool dir = faiss::gpu::randBool();

    testForSize(rows, cols, 1, dir, false);
  }
}

// Test for where k = #cols exactly (we are returning all the values,
// just sorted)
TEST(TestGpuSelect, testExact) {
  for (int i = 0; i < 5; ++i) {
    int rows = faiss::gpu::randVal(10, 100);
    int cols = faiss::gpu::randVal(1, GPU_MAX_SELECTION_K);
    bool dir = faiss::gpu::randBool();

    testForSize(rows, cols, cols, dir, false);
  }
}

// General test
TEST(TestGpuSelect, testWarp) {
  for (int i = 0; i < 10; ++i) {
    int rows = faiss::gpu::randVal(10, 100);
    int cols = faiss::gpu::randVal(1, 30000);
    int k = std::min(cols, faiss::gpu::randVal(1, GPU_MAX_SELECTION_K));
    bool dir = faiss::gpu::randBool();

    testForSize(rows, cols, k, dir, true);
  }
}

// Test for k = 1
TEST(TestGpuSelect, test1Warp) {
  for (int i = 0; i < 5; ++i) {
    int rows = faiss::gpu::randVal(10, 100);
    int cols = faiss::gpu::randVal(1, 30000);
    bool dir = faiss::gpu::randBool();

    testForSize(rows, cols, 1, dir, true);
  }
}

// Test for where k = #cols exactly (we are returning all the values,
// just sorted)
TEST(TestGpuSelect, testExactWarp) {
  for (int i = 0; i < 5; ++i) {
    int rows = faiss::gpu::randVal(10, 100);
    int cols = faiss::gpu::randVal(1, GPU_MAX_SELECTION_K);
    bool dir = faiss::gpu::randBool();

    testForSize(rows, cols, cols, dir, true);
  }
}

int main(int argc, char** argv) {
  testing::InitGoogleTest(&argc, argv);

  // just run with a fixed test seed
  faiss::gpu::setTestSeed(100);

  return RUN_ALL_TESTS();
}
