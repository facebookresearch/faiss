/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */


#include "../../IndexFlat.h"
#include "../../utils.h"
#include "../GpuIndexFlat.h"
#include "IndexWrapper.h"
#include "../test/TestUtils.h"
#include "../utils/DeviceTensor.cuh"
#include "../utils/DeviceUtils.h"
#include "../utils/HostTensor.cuh"
#include "../utils/Timer.h"
#include <gflags/gflags.h>
#include <map>
#include <memory>
#include <vector>

#include <hip/hip_runtime_api.h>

DEFINE_bool(l2, true, "L2 or inner product");
DEFINE_int32(k, 3, "final number of closest results returned");
DEFINE_int32(num, 128, "# of vecs");
DEFINE_int32(dim, 128, "# of dimensions");
DEFINE_int32(num_queries, 3, "number of query vectors");
DEFINE_bool(diff, true, "show exact distance + index output discrepancies");
DEFINE_bool(use_float16, false, "use encodings in float16");
DEFINE_bool(use_float16_math, false, "perform math in float16");
DEFINE_bool(transposed, false, "store vectors transposed");
DEFINE_int64(seed, -1, "specify random seed");
DEFINE_int32(num_gpus, 1, "number of gpus to use");
DEFINE_int64(pinned_mem, 0, "pinned memory allocation to use");
DEFINE_bool(cpu, true, "run the CPU code for timing and comparison");
DEFINE_bool(use_unified_mem, false, "use Pascal unified memory for the index");

using namespace faiss::gpu;

int main(int argc, char** argv) {
  gflags::ParseCommandLineFlags(&argc, &argv, true);

  hipProfilerStop();

  auto seed = FLAGS_seed != -1L ? FLAGS_seed : time(nullptr);
  printf("using seed %ld\n", seed);

  auto numQueries = FLAGS_num_queries;

  auto index = std::unique_ptr<faiss::IndexFlat>(
    new faiss::IndexFlat(FLAGS_dim, FLAGS_l2 ?
                         faiss::METRIC_L2 : faiss::METRIC_INNER_PRODUCT));

  HostTensor<float, 2, true> vecs({FLAGS_num, FLAGS_dim});
  faiss::float_rand(vecs.data(), vecs.numElements(), seed);

  index->add(FLAGS_num, vecs.data());

  printf("Database: dim %d num vecs %d\n", FLAGS_dim, FLAGS_num);
  printf("%s lookup: %d queries, total k %d\n",
         FLAGS_l2 ? "L2" : "IP",
         numQueries, FLAGS_k);
  printf("float16 encoding %s\n", FLAGS_use_float16 ? "enabled" : "disabled");
  printf("transposed storage %s\n", FLAGS_transposed ? "enabled" : "disabled");

  // Convert to GPU index
  printf("Copying index to %d GPU(s)...\n", FLAGS_num_gpus);

  auto initFn = [&index](faiss::gpu::GpuResources* res, int dev) ->
    std::unique_ptr<faiss::gpu::GpuIndexFlat> {
    ((faiss::gpu::StandardGpuResources*) res)->setPinnedMemory(
      FLAGS_pinned_mem);

    GpuIndexFlatConfig config;
    config.device = dev;
    config.useFloat16 = FLAGS_use_float16;
    config.useFloat16Accumulator = FLAGS_use_float16_math;
    config.storeTransposed = FLAGS_transposed;
    config.memorySpace = FLAGS_use_unified_mem ?
    MemorySpace::Unified : MemorySpace::Device;

    auto p = std::unique_ptr<faiss::gpu::GpuIndexFlat>(
      new faiss::gpu::GpuIndexFlat(res, index.get(), config));
    return p;
  };

  IndexWrapper<faiss::gpu::GpuIndexFlat> gpuIndex(FLAGS_num_gpus, initFn);
  printf("copy done\n");

  // Build query vectors
  HostTensor<float, 2, true> cpuQuery({numQueries, FLAGS_dim});
  faiss::float_rand(cpuQuery.data(), cpuQuery.numElements(), seed);

  // Time faiss CPU
  HostTensor<float, 2, true> cpuDistances({numQueries, FLAGS_k});
  HostTensor<faiss::Index::idx_t, 2, true> cpuIndices({numQueries, FLAGS_k});

  if (FLAGS_cpu) {
    float cpuTime = 0.0f;

    CpuTimer timer;
    index->search(numQueries,
                  cpuQuery.data(),
                  FLAGS_k,
                  cpuDistances.data(),
                  cpuIndices.data());

    cpuTime = timer.elapsedMilliseconds();
    printf("CPU time %.3f ms\n", cpuTime);
  }

  HostTensor<float, 2, true> gpuDistances({numQueries, FLAGS_k});
  HostTensor<faiss::Index::idx_t, 2, true> gpuIndices({numQueries, FLAGS_k});

  CUDA_VERIFY(hipProfilerStart());
  faiss::gpu::synchronizeAllDevices();

  float gpuTime = 0.0f;

  // Time GPU
  {
    CpuTimer timer;

    gpuIndex.getIndex()->search(cpuQuery.getSize(0),
                                cpuQuery.data(),
                                FLAGS_k,
                                gpuDistances.data(),
                                gpuIndices.data());

    // There is a device -> host copy above, so no need to time
    // additional synchronization with the GPU
    gpuTime = timer.elapsedMilliseconds();
  }

  CUDA_VERIFY(hipProfilerStop());
  printf("GPU time %.3f ms\n", gpuTime);

  if (FLAGS_cpu) {
    compareLists(cpuDistances.data(), cpuIndices.data(),
                 gpuDistances.data(), gpuIndices.data(),
                 numQueries, FLAGS_k,
                 "", true, FLAGS_diff, false);
  }

  CUDA_VERIFY(hipDeviceSynchronize());

  return 0;
}
