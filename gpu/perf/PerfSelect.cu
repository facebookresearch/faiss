#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */


#include "../utils/DeviceDefs.cuh"
#include "../utils/DeviceUtils.h"
#include "../utils/BlockSelectKernel.cuh"
#include "../utils/WarpSelectKernel.cuh"
#include "../utils/HostTensor.cuh"
#include "../utils/DeviceTensor.cuh"
#include "../test/TestUtils.h"
#include <algorithm>
#include <gflags/gflags.h>
#include <gtest/gtest.h>
#include <sstream>
#include <unordered_map>
#include <vector>

DEFINE_int32(rows, 10000, "rows in matrix");
DEFINE_int32(cols, 40000, "cols in matrix");
DEFINE_int32(k, 100, "k");
DEFINE_bool(dir, false, "direction of sort");
DEFINE_bool(warp, false, "warp select");
DEFINE_int32(iter, 5, "iterations to run");
DEFINE_bool(k_powers, false, "test k powers of 2 from 1 -> max k");

int main(int argc, char** argv) {
  gflags::ParseCommandLineFlags(&argc, &argv, true);

  std::vector<float> v = faiss::gpu::randVecs(FLAGS_rows, FLAGS_cols);
  faiss::gpu::HostTensor<float, 2, true> hostVal({FLAGS_rows, FLAGS_cols});

  for (int r = 0; r < FLAGS_rows; ++r) {
    for (int c = 0; c < FLAGS_cols; ++c) {
      hostVal[r][c] = v[r * FLAGS_cols + c];
    }
  }

  // Select top-k on GPU
  faiss::gpu::DeviceTensor<float, 2, true> gpuVal(hostVal, 0);

  int startK = FLAGS_k;
  int limitK = FLAGS_k;

  if (FLAGS_k_powers) {
    startK = 1;
    limitK = GPU_MAX_SELECTION_K;
  }

  for (int k = startK; k <= limitK; k *= 2) {
    faiss::gpu::DeviceTensor<float, 2, true> gpuOutVal({FLAGS_rows, k});
    faiss::gpu::DeviceTensor<int, 2, true> gpuOutInd({FLAGS_rows, k});

    for (int i = 0; i < FLAGS_iter; ++i) {
      if (FLAGS_warp) {
        faiss::gpu::runWarpSelect(gpuVal, gpuOutVal, gpuOutInd,
                                  FLAGS_dir, k, 0);
      } else {
        faiss::gpu::runBlockSelect(gpuVal, gpuOutVal, gpuOutInd,
                                   FLAGS_dir, k, 0);
      }
    }
  }

  hipDeviceSynchronize();
}
