/**
 * Copyright (c) 2015-present, Facebook, Inc.
 * All rights reserved.
 *
 * This source code is licensed under the BSD+Patents license found in the
 * LICENSE file in the root directory of this source tree.
 */


#include "../../IndexIVFPQ.h"
#include "../../index_io.h"
#include "../../utils.h"

#include "../GpuIndexIVFPQ.h"
#include "IndexWrapper.h"
#include "../test/TestUtils.h"
#include "../utils/DeviceTensor.cuh"
#include "../utils/DeviceUtils.h"
#include "../utils/HostTensor.cuh"
#include "../utils/Timer.h"
#include <gflags/gflags.h>
#include <memory>
#include <vector>

DEFINE_int32(k, 10, "final number of closest results returned");
DEFINE_string(in, "/home/jhj/local/ivfpq_index.out", "index file for input");
DEFINE_bool(use_precomputed, true, "enable or disable precomputed codes");
DEFINE_bool(float16_lookup, false, "use float16 residual distance tables");
DEFINE_int32(num_gpus, 1, "number of gpus to use");
DEFINE_int32(index, 2, "0 = no indices on GPU; 1 = 32 bit, 2 = 64 bit on GPU");

using namespace faiss::gpu;

int main(int argc, char** argv) {
  gflags::ParseCommandLineFlags(&argc, &argv, true);

  auto seed = time(nullptr);
  auto k = FLAGS_k;

  auto index = std::unique_ptr<faiss::IndexIVFPQ>(
    dynamic_cast<faiss::IndexIVFPQ*>(faiss::read_index(FLAGS_in.c_str())));
  FAISS_ASSERT((bool) index);

  auto dim = index->d;
  auto codes = index->pq.M;
  auto bitsPerCode = index->pq.nbits;

  printf("Database: dim %d num vecs %ld\n", dim, index->ntotal);
  printf("Coarse centroids: %ld\n", index->quantizer->ntotal);
  printf("PQ centroids: codes %ld bits per code %ld\n", codes, bitsPerCode);
  printf("L2 lookup: total k %d, precomputed codes %d\n\n",
         k, FLAGS_use_precomputed);

  // Convert to GPU index
  printf("Copying index to %d GPU(s)...\n", FLAGS_num_gpus);

  bool precomp = FLAGS_use_precomputed;
  auto indicesOpt = (faiss::gpu::IndicesOptions) FLAGS_index;
  auto useFloat16Lookup = FLAGS_float16_lookup;

  auto initFn = [precomp, indicesOpt, useFloat16Lookup, &index]
    (faiss::gpu::GpuResources* res, int dev) ->
    std::unique_ptr<faiss::gpu::GpuIndexIVFPQ> {

    faiss::gpu::GpuIndexIVFPQConfig config;
    config.device = dev;
    config.usePrecomputedTables = precomp;
    config.indicesOptions = indicesOpt;
    config.useFloat16LookupTables = useFloat16Lookup;

    auto p = std::unique_ptr<faiss::gpu::GpuIndexIVFPQ>(
      new faiss::gpu::GpuIndexIVFPQ(res, index.get(), config));

    return p;
  };

  IndexWrapper<faiss::gpu::GpuIndexIVFPQ> gpuIndex(FLAGS_num_gpus, initFn);
  printf("copy done\n");

  auto querySizes = std::vector<int>{1, 4, 16, 64, 256, 1024, 4096, 16384};
  auto nprobeSizes = std::vector<int>{1, 4, 8, 16, 32, 64, 128, 256};

  HostTensor<float, 2, true> cpuTimePerVector(
    {(int) querySizes.size(), (int) nprobeSizes.size()});
  HostTensor<float, 2, true> gpuTimePerVector(
    {(int) querySizes.size(), (int) nprobeSizes.size()});

  printf("GPU relative speedup over CPU (x):\n");

  for (auto q = 0; q < querySizes.size(); ++q) {
    auto numQueries = querySizes[q];
    bool first = true;

    for (auto p = 0; p < nprobeSizes.size(); ++p) {
      auto nprobe = nprobeSizes[q];

      HostTensor<float, 2, true> cpuQuery{numQueries, dim};
      faiss::float_rand(cpuQuery.data(), cpuQuery.numElements(), seed);

      HostTensor<faiss::Index::idx_t, 2, true> resultIndices{numQueries, k};
      HostTensor<float, 2, true> resultDistances{numQueries, k};

      index->nprobe = nprobe;

      float cpuTime = 0.0f;
      {
        CpuTimer timer;
        if (!FLAGS_use_precomputed) {
          index->use_precomputed_table = 0;
        }
        index->search(numQueries, cpuQuery.data(),
                      k, resultDistances.data(), resultIndices.data());
        cpuTime = timer.elapsedMilliseconds();
        cpuTimePerVector[q][p] = cpuTime / (float) numQueries;
      }

      gpuIndex.setNumProbes(nprobe);

      HostTensor<float, 2, true>
        gpuHostDistances({numQueries, k});
      HostTensor<faiss::Index::idx_t, 2, true>
        gpuHostIndices({numQueries, k});

      DeviceTensor<float, 2, true> gpuQuery(cpuQuery, 0);
      DeviceTensor<float, 2, true> gpuDistances({numQueries, k});
      DeviceTensor<long, 2, true> gpuIndices({numQueries, k});
      CUDA_VERIFY(hipDeviceSynchronize());

      float gpuTime = 0.0f;

      {
        CpuTimer timer;

        gpuIndex.getIndex()->search(cpuQuery.getSize(0),
                                    cpuQuery.data(),
                                    FLAGS_k,
                                    gpuHostDistances.data(),
                                    gpuHostIndices.data());

        CUDA_VERIFY(hipDeviceSynchronize());
        gpuTime = timer.elapsedMilliseconds();
      }

      gpuTimePerVector[q][p] = gpuTime / (float) numQueries;

      if (!first) {
        printf(", ");
      }
      first = false;

      printf("%.2f", cpuTime / gpuTime);
    }

    printf("\n");
  }

  printf("\n");
  printf("CPU time per query vector (us):\n");

  for (int q = 0; q < cpuTimePerVector.getSize(0); ++q) {
    bool first = true;

    for (int p = 0; p < cpuTimePerVector.getSize(1); ++p) {
      if (!first) {
        printf(", ");
      }
      first = false;

      printf("%.1f", (float) cpuTimePerVector[q][p] * 1000.0f);
    }

    printf("\n");
  }

  printf("\n");
  printf("GPU time per query vector (us):\n");

  for (int q = 0; q < gpuTimePerVector.getSize(0); ++q) {
    bool first = true;

    for (int p = 0; p < gpuTimePerVector.getSize(1); ++p) {
      if (!first) {
        printf(", ");
      }
      first = false;

      printf("%.1f", (float) gpuTimePerVector[q][p] * 1000.0f);
    }

    printf("\n");
  }

  return 0;
}
