#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */


#include "GpuIndexIVF.h"
#include "../FaissAssert.h"
#include "../IndexFlat.h"
#include "../IndexIVF.h"
#include "GpuIndexFlat.h"
#include "utils/DeviceUtils.h"
#include "utils/Float16.cuh"

namespace faiss { namespace gpu {

GpuIndexIVF::GpuIndexIVF(GpuResources* resources,
                         int dims,
                         faiss::MetricType metric,
                         int nlist,
                         GpuIndexIVFConfig config) :
    GpuIndex(resources, dims, metric, config),
    ivfConfig_(std::move(config)),
    nlist_(nlist),
    nprobe_(1),
    quantizer_(nullptr) {
#ifndef FAISS_USE_FLOAT16
  FAISS_THROW_IF_NOT_MSG(!ivfConfig_.flatConfig.useFloat16 &&
                         !ivfConfig_.flatConfig.useFloat16Accumulator,
                         "float16 unsupported; need CUDA SDK >= 7.5");
#endif

  init_();
}

void
GpuIndexIVF::init_() {
  FAISS_ASSERT(nlist_ > 0);

  // Spherical by default if the metric is inner_product
  if (this->metric_type == faiss::METRIC_INNER_PRODUCT) {
    this->cp.spherical = true;
  }

  // here we set a low # iterations because this is typically used
  // for large clusterings
  this->cp.niter = 10;
  this->cp.verbose = this->verbose;

  if (!quantizer_) {
    // Construct an empty quantizer
    GpuIndexFlatConfig config = ivfConfig_.flatConfig;
    // FIXME: inherit our same device
    config.device = device_;

    if (this->metric_type == faiss::METRIC_L2) {
      quantizer_ = new GpuIndexFlatL2(resources_, this->d, config);
    } else if (this->metric_type == faiss::METRIC_INNER_PRODUCT) {
      quantizer_ = new GpuIndexFlatIP(resources_, this->d, config);
    } else {
      // unknown metric type
      FAISS_ASSERT_MSG(false, "unknown metric type");
    }
  }
}

GpuIndexIVF::~GpuIndexIVF() {
  delete quantizer_;
}

GpuIndexFlat*
GpuIndexIVF::getQuantizer() {
  return quantizer_;
}

void
GpuIndexIVF::copyFrom(const faiss::IndexIVF* index) {
  DeviceScope scope(device_);

  this->d = index->d;
  this->metric_type = index->metric_type;

  FAISS_ASSERT(index->nlist > 0);
  FAISS_THROW_IF_NOT_FMT(index->nlist <=
                     (faiss::Index::idx_t) std::numeric_limits<int>::max(),
                     "GPU index only supports %zu inverted lists",
                     (size_t) std::numeric_limits<int>::max());
  nlist_ = index->nlist;

  FAISS_THROW_IF_NOT_FMT(index->nprobe > 0 &&
                         index->nprobe <= getMaxKSelection(),
                         "GPU index only supports nprobe <= %zu; passed %zu",
                         (size_t) getMaxKSelection(),
                         index->nprobe);
  nprobe_ = index->nprobe;

  // The metric type may have changed as well, so we might have to
  // change our quantizer
  delete quantizer_;
  quantizer_ = nullptr;

  // Construct an empty quantizer
  GpuIndexFlatConfig config = ivfConfig_.flatConfig;
  // FIXME: inherit our same device
  config.device = device_;

  if (index->metric_type == faiss::METRIC_L2) {
    // FIXME: 2 different float16 options?
    quantizer_ = new GpuIndexFlatL2(resources_, this->d, config);
  } else if (index->metric_type == faiss::METRIC_INNER_PRODUCT) {
    // FIXME: 2 different float16 options?
    quantizer_ = new GpuIndexFlatIP(resources_, this->d, config);
  } else {
    // unknown metric type
    FAISS_ASSERT(false);
  }

  if (!index->is_trained) {
    this->is_trained = false;
    this->ntotal = 0;
    return;
  }

  // Otherwise, we can populate ourselves from the other index
  this->is_trained = true;

  // ntotal can exceed max int, but the number of vectors per inverted
  // list cannot exceed this. We check this in the subclasses.
  this->ntotal = index->ntotal;

  // Since we're trained, the quantizer must have data
  FAISS_ASSERT(index->quantizer->ntotal > 0);

  if (index->metric_type == faiss::METRIC_L2) {
    auto q = dynamic_cast<faiss::IndexFlatL2*>(index->quantizer);
    FAISS_ASSERT(q);

    quantizer_->copyFrom(q);
  } else if (index->metric_type == faiss::METRIC_INNER_PRODUCT) {
    auto q = dynamic_cast<faiss::IndexFlatIP*>(index->quantizer);
    FAISS_ASSERT(q);

    quantizer_->copyFrom(q);
  } else {
    // unknown metric type
    FAISS_ASSERT(false);
  }
}

void
GpuIndexIVF::copyTo(faiss::IndexIVF* index) const {
  DeviceScope scope(device_);

  //
  // Index information
  //
  index->ntotal = this->ntotal;
  index->d = this->d;
  index->metric_type = this->metric_type;
  index->is_trained = this->is_trained;

  //
  // IndexIVF information
  //
  index->nlist = nlist_;
  index->nprobe = nprobe_;

  // Construct and copy the appropriate quantizer
  faiss::IndexFlat* q = nullptr;

  if (this->metric_type == faiss::METRIC_L2) {
    q = new faiss::IndexFlatL2(this->d);

  } else if (this->metric_type == faiss::METRIC_INNER_PRODUCT) {
    q = new faiss::IndexFlatIP(this->d);

  } else {
    // unknown metric type
    FAISS_ASSERT(false);
  }

  FAISS_ASSERT(quantizer_);
  quantizer_->copyTo(q);

  if (index->own_fields) {
    delete index->quantizer;
  }

  index->quantizer = q;
  index->quantizer_trains_alone = 0;
  index->own_fields = true;
  index->cp = this->cp;
  index->maintain_direct_map = false;
  index->direct_map.clear();
}

int
GpuIndexIVF::getNumLists() const {
  return nlist_;
}

void
GpuIndexIVF::setNumProbes(int nprobe) {
  FAISS_THROW_IF_NOT_FMT(nprobe > 0 && nprobe <= getMaxKSelection(),
                         "GPU index only supports nprobe <= %d; passed %d",
                         getMaxKSelection(),
                         nprobe);
  nprobe_ = nprobe;
}

int
GpuIndexIVF::getNumProbes() const {
  return nprobe_;
}

bool
GpuIndexIVF::addImplRequiresIDs_() const {
  // All IVF indices have storage for IDs
  return true;
}

void
GpuIndexIVF::trainQuantizer_(faiss::Index::idx_t n, const float* x) {
  if (n == 0) {
    // nothing to do
    return;
  }

  if (quantizer_->is_trained && (quantizer_->ntotal == nlist_)) {
    if (this->verbose) {
      printf ("IVF quantizer does not need training.\n");
    }

    return;
  }

  if (this->verbose) {
    printf ("Training IVF quantizer on %ld vectors in %dD\n", n, d);
  }

  DeviceScope scope(device_);

  // leverage the CPU-side k-means code, which works for the GPU
  // flat index as well
  quantizer_->reset();
  Clustering clus(this->d, nlist_, this->cp);
  clus.verbose = verbose;
  clus.train(n, x, *quantizer_);
  quantizer_->is_trained = true;

  FAISS_ASSERT(quantizer_->ntotal == nlist_);
}

} } // namespace
