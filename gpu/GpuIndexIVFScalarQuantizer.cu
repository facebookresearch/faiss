#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */


#include <faiss/gpu/GpuIndexIVFScalarQuantizer.h>
#include <faiss/gpu/GpuIndexFlat.h>
#include <faiss/gpu/GpuResources.h>
#include <faiss/gpu/impl/GpuScalarQuantizer.cuh>
#include <faiss/gpu/impl/IVFFlat.cuh>
#include <faiss/gpu/utils/CopyUtils.cuh>
#include <faiss/gpu/utils/DeviceUtils.h>
#include <limits>

namespace faiss { namespace gpu {

GpuIndexIVFScalarQuantizer::GpuIndexIVFScalarQuantizer(
  GpuResources* resources,
  const faiss::IndexIVFScalarQuantizer* index,
  GpuIndexIVFScalarQuantizerConfig config) :
    GpuIndexIVF(resources,
                index->d,
                index->metric_type,
                index->metric_arg,
                index->nlist,
                config),
    ivfSQConfig_(config),
    sq(index->sq),
    by_residual(index->by_residual),
    reserveMemoryVecs_(0),
    index_(nullptr) {
  copyFrom(index);

  FAISS_THROW_IF_NOT_MSG(isSQSupported(sq.qtype),
                         "Unsupported QuantizerType on GPU");
}

GpuIndexIVFScalarQuantizer::GpuIndexIVFScalarQuantizer(
  GpuResources* resources,
  int dims,
  int nlist,
  faiss::ScalarQuantizer::QuantizerType qtype,
  faiss::MetricType metric,
  bool encodeResidual,
  GpuIndexIVFScalarQuantizerConfig config) :
    GpuIndexIVF(resources, dims, metric, 0, nlist, config),
    ivfSQConfig_(config),
    sq(dims, qtype),
    by_residual(encodeResidual),
    reserveMemoryVecs_(0),
    index_(nullptr) {

  // faiss::Index params
  this->is_trained = false;

  // We haven't trained ourselves, so don't construct the IVFFlat
  // index yet
  FAISS_THROW_IF_NOT_MSG(isSQSupported(sq.qtype),
                         "Unsupported QuantizerType on GPU");
}

GpuIndexIVFScalarQuantizer::~GpuIndexIVFScalarQuantizer() {
  delete index_;
}

void
GpuIndexIVFScalarQuantizer::reserveMemory(size_t numVecs) {
  reserveMemoryVecs_ = numVecs;
  if (index_) {
    DeviceScope scope(device_);
    index_->reserveMemory(numVecs);
  }
}

void
GpuIndexIVFScalarQuantizer::copyFrom(
  const faiss::IndexIVFScalarQuantizer* index) {
  DeviceScope scope(device_);

  // Clear out our old data
  delete index_;
  index_ = nullptr;

  // Copy what we need from the CPU index
  GpuIndexIVF::copyFrom(index);
  sq = index->sq;
  by_residual = index->by_residual;

  // The other index might not be trained, in which case we don't need to copy
  // over the lists
  if (!index->is_trained) {
    return;
  }

  // Otherwise, we can populate ourselves from the other index
  this->is_trained = true;

  // Copy our lists as well
  index_ = new IVFFlat(resources_,
                       quantizer->getGpuData(),
                       index->metric_type,
                       index->metric_arg,
                       by_residual,
                       &sq,
                       ivfSQConfig_.indicesOptions,
                       memorySpace_);

  InvertedLists* ivf = index->invlists;

  for (size_t i = 0; i < ivf->nlist; ++i) {
    auto numVecs = ivf->list_size(i);

    // GPU index can only support max int entries per list
    FAISS_THROW_IF_NOT_FMT(numVecs <=
                           (size_t) std::numeric_limits<int>::max(),
                           "GPU inverted list can only support "
                           "%zu entries; %zu found",
                           (size_t) std::numeric_limits<int>::max(),
                           numVecs);

    index_->addCodeVectorsFromCpu(
      i,
      (const unsigned char*) ivf->get_codes(i),
      ivf->get_ids(i),
      numVecs);
  }
}

void
GpuIndexIVFScalarQuantizer::copyTo(
  faiss::IndexIVFScalarQuantizer* index) const {
  DeviceScope scope(device_);

  // We must have the indices in order to copy to ourselves
  FAISS_THROW_IF_NOT_MSG(
    ivfSQConfig_.indicesOptions != INDICES_IVF,
    "Cannot copy to CPU as GPU index doesn't retain "
    "indices (INDICES_IVF)");

  GpuIndexIVF::copyTo(index);
  index->sq = sq;
  index->code_size = sq.code_size;
  index->by_residual = by_residual;

  InvertedLists* ivf = new ArrayInvertedLists(nlist, index->code_size);
  index->replace_invlists(ivf, true);

  // Copy the inverted lists
  if (index_) {
    for (int i = 0; i < nlist; ++i) {
      auto listIndices = index_->getListIndices(i);
      auto listData = index_->getListVectors(i);

      ivf->add_entries(i,
                       listIndices.size(),
                       listIndices.data(),
                       (const uint8_t*) listData.data());
    }
  }
}

size_t
GpuIndexIVFScalarQuantizer::reclaimMemory() {
  if (index_) {
    DeviceScope scope(device_);

    return index_->reclaimMemory();
  }

  return 0;
}

void
GpuIndexIVFScalarQuantizer::reset() {
  if (index_) {
    DeviceScope scope(device_);

    index_->reset();
    this->ntotal = 0;
  } else {
    FAISS_ASSERT(this->ntotal == 0);
  }
}

void
GpuIndexIVFScalarQuantizer::trainResiduals_(Index::idx_t n, const float* x) {
  // The input is already guaranteed to be on the CPU
  sq.train_residual(n, x, quantizer, by_residual, verbose);
}

void
GpuIndexIVFScalarQuantizer::train(Index::idx_t n, const float* x) {
  DeviceScope scope(device_);

  if (this->is_trained) {
    FAISS_ASSERT(quantizer->is_trained);
    FAISS_ASSERT(quantizer->ntotal == nlist);
    FAISS_ASSERT(index_);
    return;
  }

  FAISS_ASSERT(!index_);

  // FIXME: GPUize more of this
  // First, make sure that the data is resident on the CPU, if it is not on the
  // CPU, as we depend upon parts of the CPU code
  auto hostData = toHost<float, 2>((float*) x,
                                   resources_->getDefaultStream(device_),
                                   {(int) n, (int) this->d});

  trainQuantizer_(n, hostData.data());
  trainResiduals_(n, hostData.data());

  // The quantizer is now trained; construct the IVF index
  index_ = new IVFFlat(resources_,
                       quantizer->getGpuData(),
                       this->metric_type,
                       this->metric_arg,
                       by_residual,
                       &sq,
                       ivfSQConfig_.indicesOptions,
                       memorySpace_);

  if (reserveMemoryVecs_) {
    index_->reserveMemory(reserveMemoryVecs_);
  }

  this->is_trained = true;
}

void
GpuIndexIVFScalarQuantizer::addImpl_(int n,
                                     const float* x,
                                     const Index::idx_t* xids) {
  // Device is already set in GpuIndex::add
  FAISS_ASSERT(index_);
  FAISS_ASSERT(n > 0);

  // Data is already resident on the GPU
  Tensor<float, 2, true> data(const_cast<float*>(x), {n, (int) this->d});

  static_assert(sizeof(long) == sizeof(Index::idx_t), "size mismatch");
  Tensor<long, 1, true> labels(const_cast<long*>(xids), {n});

  // Not all vectors may be able to be added (some may contain NaNs etc)
  index_->classifyAndAddVectors(data, labels);

  // but keep the ntotal based on the total number of vectors that we attempted
  // to add
  ntotal += n;
}

void
GpuIndexIVFScalarQuantizer::searchImpl_(int n,
                                        const float* x,
                                        int k,
                                        float* distances,
                                        Index::idx_t* labels) const {
  // Device is already set in GpuIndex::search
  FAISS_ASSERT(index_);
  FAISS_ASSERT(n > 0);

  // Data is already resident on the GPU
  Tensor<float, 2, true> queries(const_cast<float*>(x), {n, (int) this->d});
  Tensor<float, 2, true> outDistances(distances, {n, k});

  static_assert(sizeof(long) == sizeof(Index::idx_t), "size mismatch");
  Tensor<long, 2, true> outLabels(const_cast<long*>(labels), {n, k});

  index_->query(queries, nprobe, k, outDistances, outLabels);
}

} } // namespace
